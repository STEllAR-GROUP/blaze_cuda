#include "hip/hip_runtime.h"
//=================================================================================================
/*!
//  \file src/mathtest/sparsematrix/GeneralTest.cpp
//  \brief Source file for the general SparseMatrix operation test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/sparse/SparseMatrix.h>
#include <blaze/math/CompressedMatrix.h>
#include <blazetest/mathtest/IsEqual.h>
#include <blazetest/mathtest/sparsematrix/GeneralTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace sparsematrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the GeneralTest class test.
//
// \exception std::runtime_error Operation error detected.
*/
GeneralTest::GeneralTest()
{
   testIsNan();
   testIsSquare();
   testIsSymmetric();
   testIsHermitian();
   testIsUniform();
   testIsZero();
   testIsLower();
   testIsUniLower();
   testIsStrictlyLower();
   testIsUpper();
   testIsUniUpper();
   testIsStrictlyUpper();
   testIsDiagonal();
   testIsIdentity();
   testMinimum();
   testMaximum();
   testL1Norm();
   testL2Norm();
   testL3Norm();
   testL4Norm();
   testLpNorm();
   testTrace();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the \c isnan() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isnan() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsNan()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isnan()";

      // isnan with 0x0 matrix
      {
         blaze::CompressedMatrix<float,blaze::rowMajor> mat;

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isnan with empty 3x5 matrix
      {
         blaze::CompressedMatrix<float,blaze::rowMajor> mat( 3UL, 5UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 0UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isnan with filled 4x2 matrix
      {
         blaze::CompressedMatrix<float,blaze::rowMajor> mat( 4UL, 2UL );
         mat(1,1) =  1.0F;
         mat(2,0) = -2.0F;
         mat(2,1) =  3.0F;
         mat(3,0) =  4.0F;

         checkRows    ( mat, 4UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 4UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isnan()";

      // isnan with 0x0 matrix
      {
         blaze::CompressedMatrix<float,blaze::columnMajor> mat;

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isnan with empty 3x5 matrix
      {
         blaze::CompressedMatrix<float,blaze::columnMajor> mat( 3UL, 5UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 0UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isnan with filled 4x2 matrix
      {
         blaze::CompressedMatrix<float,blaze::columnMajor> mat( 4UL, 2UL );
         mat(1,1) =  1.0F;
         mat(2,0) = -2.0F;
         mat(2,1) =  3.0F;
         mat(3,0) =  4.0F;

         checkRows    ( mat, 4UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 4UL );

         if( blaze::isnan( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isnan evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isSquare() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isSquare() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsSquare()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isSquare()";

      // Square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows   ( mat, 3UL );
         checkColumns( mat, 3UL );

         if( isSquare( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSquare evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL, 0 );

         checkRows   ( mat, 2UL );
         checkColumns( mat, 3UL );

         if( isSquare( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSquare evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isSquare()";

      // Square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );

         checkRows   ( mat, 3UL );
         checkColumns( mat, 3UL );

         if( isSquare( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSquare evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 2UL, 0 );

         checkRows   ( mat, 3UL );
         checkColumns( mat, 2UL );

         if( isSquare( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSquare evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isSymmetric() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isSymmetric() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsSymmetric()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isSymmetric()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-symmetric matrix (additional element in the lower part)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-symmetric matrix (additional element in the upper part)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Symmetric matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isSymmetric()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-symmetric matrix (additional element in the lower part)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-symmetric matrix (additional element in the upper part)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Symmetric matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isSymmetric( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isSymmetric evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isHermitian() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isHermitian() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsHermitian()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isHermitian()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isHermitian( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-real diagonal element
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 1UL );
         mat(1,1).imag( 1 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 1UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (additional element in the lower part)
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0).real( 1 );
         mat(1,1).real( 2 );
         mat(2,0).real( 4 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (additional element in the upper part)
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0).real( 1 );
         mat(0,2).real( 4 );
         mat(1,1).real( 2 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (invalid pair of elements)
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0).real( 1 );
         mat(0,2).imag( 4 );
         mat(1,1).real( 2 );
         mat(2,0).imag( 4 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Hermitian matrix
      {
         blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
         mat(0,0).real(  1 );
         mat(0,2).imag(  4 );
         mat(1,1).real(  2 );
         mat(2,0).imag( -4 );
         mat(2,2).real(  3 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 7UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isHermitian()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isHermitian( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-real diagonal element
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 1UL );
         mat(1,1).imag( 1 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 1UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (additional element in the lower part)
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0).real( 1 );
         mat(1,1).real( 2 );
         mat(2,0).real( 4 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (additional element in the upper part)
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0).real( 1 );
         mat(0,2).real( 4 );
         mat(1,1).real( 2 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-Hermitian matrix (invalid pair of elements)
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0).real( 1 );
         mat(0,2).imag( 4 );
         mat(1,1).real( 2 );
         mat(2,0).imag( 4 );
         mat(2,2).real( 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Hermitian matrix
      {
         blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0).real(  1 );
         mat(0,2).imag(  4 );
         mat(1,1).real(  2 );
         mat(2,0).imag( -4 );
         mat(2,2).real(  3 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 7UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isHermitian( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isHermitian evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isUniform() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isUniform() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsUniform()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isUniform()";

      // Uniform matrix (0x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 3UL );

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x0)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 0UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 0UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (1x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 1UL, 3UL, 3UL );
         mat(0,0) = 5;
         mat(0,1) = 5;
         mat(0,2) = 5;

         checkRows    ( mat, 1UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 3UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x1)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 1UL, 3UL );
         mat(0,0) = 5;
         mat(1,0) = 5;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 1UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x5)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 5UL, 3UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (5x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 3UL, 5UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );
         mat.insert( 3UL, 1UL, 0 );
         mat.insert( 4UL, 2UL, 0 );

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );
         checkNonZeros( mat, 3UL, 1UL );
         checkNonZeros( mat, 4UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-uniform matrix (3x3, 3 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 2UL, 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-uniform matrix (3x3, 9 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 9UL );
         for( size_t i=0UL; i<3UL; ++i )
            for( size_t j=0UL; j<3UL; ++j )
               mat.insert( i, j, 0UL );
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 9UL );
         checkNonZeros( mat, 9UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 3UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUniform( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isUniform()";

      // Uniform matrix (0x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 3UL );

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x0)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 0UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 0UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (1x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 1UL, 3UL, 3UL );
         mat(0,0) = 5;
         mat(0,1) = 5;
         mat(0,2) = 5;

         checkRows    ( mat, 1UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x1)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 1UL, 3UL );
         mat(0,0) = 5;
         mat(1,0) = 5;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 1UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 3UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (3x5)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 5UL, 5UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 2UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 3UL, 0 );
         mat.insert( 0UL, 4UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );
         checkNonZeros( mat, 3UL, 1UL );
         checkNonZeros( mat, 4UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Uniform matrix (5x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 3UL, 3UL );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 1UL, 0 );
         mat.insert( 0UL, 2UL, 0 );

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-uniform matrix (3x3, 3 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 2UL, 2UL, 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniform( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-uniform matrix (3x3, 9 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 9UL );
         for( size_t i=0UL; i<3UL; ++i )
            for( size_t j=0UL; j<3UL; ++j )
               mat.insert( i, j, 0UL );
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 9UL );
         checkNonZeros( mat, 9UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 3UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUniform( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniform evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isZero() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isZero() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsZero()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isZero()";

      // Zero matrix (0x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 3UL );

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x0)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 0UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 0UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (1x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 1UL, 3UL, 1UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 0UL, 2UL, 0 );

         checkRows    ( mat, 1UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 3UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x1)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 1UL, 1UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 1UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x5)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 5UL, 3UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (5x3)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 3UL, 5UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );
         mat.insert( 3UL, 1UL, 0 );
         mat.insert( 4UL, 2UL, 0 );

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );
         checkNonZeros( mat, 3UL, 1UL );
         checkNonZeros( mat, 4UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-zero matrix (3x3, 3 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 2UL, 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-zero matrix (3x3, 9 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 9UL );
         for( size_t i=0UL; i<3UL; ++i )
            for( size_t j=0UL; j<3UL; ++j )
               mat.insert( i, j, 0UL );
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 9UL );
         checkNonZeros( mat, 9UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 3UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isZero( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isZero()";

      // Zero matrix (0x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 3UL );

         checkRows    ( mat, 0UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x0)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 0UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 0UL );
         checkCapacity( mat, 0UL );
         checkNonZeros( mat, 0UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (1x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 1UL, 3UL, 3UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 0UL, 2UL, 0 );

         checkRows    ( mat, 1UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x1)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 1UL, 3UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 1UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 3UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (3x5)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 5UL, 5UL );
         mat.insert( 0UL, 0UL, 0 );
         mat.insert( 2UL, 1UL, 0 );
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 3UL, 0 );
         mat.insert( 0UL, 4UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );
         checkNonZeros( mat, 3UL, 1UL );
         checkNonZeros( mat, 4UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Zero matrix (5x3)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 3UL, 3UL );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 2UL, 1UL, 0 );
         mat.insert( 0UL, 2UL, 0 );

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-zero matrix (3x3, 3 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat.insert( 1UL, 0UL, 0 );
         mat.insert( 0UL, 1UL, 0 );
         mat.insert( 2UL, 2UL, 3 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isZero( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-zero matrix (3x3, 9 non-zero elements)
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 9UL );
         for( size_t i=0UL; i<3UL; ++i )
            for( size_t j=0UL; j<3UL; ++j )
               mat.insert( i, j, 0UL );
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 9UL );
         checkNonZeros( mat, 9UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 3UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isZero( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isZero evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isLower() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isLower() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsLower()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(1,1) = 4;
         mat(2,2) = 5;
         mat(2,0) = 6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 2UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-lower triangle matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(1,1) = 4;
         mat(2,2) = 5;
         mat(2,0) = 6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isUniLower() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isUniLower() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsUniLower()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isUniLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat(2,0) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-lower unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isUniLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat(2,0) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-lower unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isStrictlyLower() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isStrictlyLower() function for sparse matrices. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsStrictlyLower()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isStrictlyLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Strictly lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 2UL );
         mat(1,0) = 2;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-strictly lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isStrictlyLower()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Strictly lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 2UL );
         mat(1,0) = 2;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyLower( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,0) = 2;
         mat(1,1) = 3;
         mat(2,2) = 4;
         mat(2,0) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 3UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-strictly lower triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat(0,2) = 2;
         mat(1,0) = 3;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyLower( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyLower evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isUpper() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isUpper() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsUpper()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-upper triangle matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,0) = 5;
         mat(2,2) = 6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 2UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-upper triangle matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,0) = 5;
         mat(2,2) = 6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isUniUpper() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isUniUpper() function for sparse matrices. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsUniUpper()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isUniUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(1,2) = 3;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(1,2) = 3;
         mat(2,0) = 4;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isUniUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isUniUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper unitriangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(1,2) = 3;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUniUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(1,2) = 3;
         mat(2,0) = 4;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 6UL );
         checkNonZeros( mat, 6UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isUniUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isUniUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isStrictlyUpper() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isStrictlyUpper() function for sparse matrices. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsStrictlyUpper()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isStrictlyUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Strictly upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 2UL );
         mat(0,2) = 2;
         mat(1,2) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-strictly upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,2) = 2;
         mat(1,2) = 3;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isStrictlyUpper()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isStrictlyUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Strictly upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 2UL );
         mat(0,2) = 2;
         mat(1,2) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyUpper( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(1,2) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 3UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Non-strictly upper triangular matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat(0,2) = 2;
         mat(1,2) = 3;
         mat(2,0) = 4;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isStrictlyUpper( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isStrictlyUpper evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isDiagonal() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isDiagonal() function for sparse matrices. In case
// an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsDiagonal()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isDiagonal()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isDiagonal( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isDiagonal( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isDiagonal()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isDiagonal( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isDiagonal( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,0) = 4;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(0,2) = 4;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isDiagonal( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDiagonal evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isIdentity() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isIdentity() function for sparse matrices. In case
// an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testIsIdentity()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isIdentity()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 2UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isIdentity( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Incomplete identity matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 2UL );
         mat(0,0) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,0) = 2;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isIdentity()";

      // Non-square matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Default initialized matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 0UL );
         checkNonZeros( mat, 0UL, 0UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 0UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Identity matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,2) = 1;
         mat.insert( 1UL, 2UL, 0 );
         mat.insert( 2UL, 0UL, 0 );

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 5UL );
         checkNonZeros( mat, 5UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isIdentity( mat ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Incomplete identity matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 2UL );
         mat(0,0) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 2UL );
         checkNonZeros( mat, 2UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 0UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Diagonal matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 3UL );
         mat(0,0) = 1;
         mat(1,1) = 2;
         mat(2,2) = 3;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 3UL );
         checkNonZeros( mat, 3UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Lower matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(1,1) = 1;
         mat(2,0) = 2;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 2UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 1UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Upper matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 1;
         mat(2,2) = 1;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkCapacity( mat, 4UL );
         checkNonZeros( mat, 4UL );
         checkNonZeros( mat, 0UL, 1UL );
         checkNonZeros( mat, 1UL, 1UL );
         checkNonZeros( mat, 2UL, 2UL );

         if( isIdentity( mat ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isIdentity evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << mat << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c min() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c min() function for sparse matrices template. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testMinimum()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major min()";

      // Attempt to find the minimum at the beginning in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 2UL, 5UL );
         mat(0,0) = -1;
         mat(0,1) =  2;
         mat(1,0) =  3;
         mat(1,1) =  4;
         mat(2,0) =  5;
         mat(2,1) =  6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 6UL );

         const int minimum = min( mat );

         if( minimum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the end in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL, 5UL );
         mat(0,0) =  1;
         mat(0,1) =  2;
         mat(0,2) =  3;
         mat(1,0) =  4;
         mat(1,1) =  5;
         mat(1,2) = -6;

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 6UL );

         const int minimum = min( mat );

         if( minimum != -6 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -6\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the beginning in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,2) =  2;
         mat(2,1) =  3;
         mat(4,0) =  4;
         mat(4,2) =  5;

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Third computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the end in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 5UL, 5UL );
         mat(0,0) =  1;
         mat(0,4) =  2;
         mat(1,2) =  3;
         mat(2,0) =  4;
         mat(2,4) = -5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -5 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fourth computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -5\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to detect 0 as the minimum value
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(2,0) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fifth computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major min()";

      // Attempt to find the minimum at the beginning in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,2) =  2;
         mat(2,1) =  3;
         mat(4,0) =  4;
         mat(4,2) =  5;

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the end in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 5UL, 5UL );
         mat(0,0) =  1;
         mat(0,4) =  2;
         mat(1,2) =  3;
         mat(2,0) =  4;
         mat(2,4) = -5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -5 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -5\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the beginning in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,2) =  2;
         mat(2,1) =  3;
         mat(4,0) =  4;
         mat(4,2) =  5;

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Third computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the minimum at the end in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 5UL, 5UL );
         mat(0,0) =  1;
         mat(0,4) =  2;
         mat(1,2) =  3;
         mat(2,0) =  4;
         mat(2,4) = -5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != -5 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fourth computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: -5\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to detect 0 as the minimum value
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = 1;
         mat(0,2) = 2;
         mat(1,1) = 3;
         mat(2,0) = 4;
         mat(2,2) = 5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int minimum = min( mat );

         if( minimum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fifth computation failed\n"
                << " Details:\n"
                << "   Result: " << minimum << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c max() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c max() function for sparse matrices template. In case an
// error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testMaximum()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major max()";

      // Attempt to find the maximum at the beginning in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 2UL, 5UL );
         mat(0,0) =  1;
         mat(0,1) = -2;
         mat(1,0) = -3;
         mat(1,1) = -4;
         mat(2,0) = -5;
         mat(2,1) = -6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 6UL );

         const int maximum = max( mat );

         if( maximum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the end in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,1) = -2;
         mat(0,2) = -3;
         mat(1,0) = -4;
         mat(1,1) = -5;
         mat(1,2) =  6;

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 6UL );

         const int maximum = max( mat );

         if( maximum != 6 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 6\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the beginning in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 3UL, 5UL );
         mat(0,0) =  1;
         mat(0,2) = -2;
         mat(2,1) = -3;
         mat(4,0) = -4;
         mat(4,2) = -5;

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Third computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the end in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 5UL, 5UL );
         mat(0,0) = -1;
         mat(0,4) = -2;
         mat(1,2) = -3;
         mat(2,0) = -4;
         mat(2,4) =  5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != 5 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fourth computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 5\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to detect 0 as the maximum value
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,2) = -2;
         mat(1,1) = -3;
         mat(2,0) = -4;
         mat(2,2) = -5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fifth computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major max()";

      // Attempt to find the maximum at the beginning in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 2UL, 5UL );
         mat(0,0) =  1;
         mat(0,1) = -2;
         mat(1,0) = -3;
         mat(1,1) = -4;
         mat(2,0) = -5;
         mat(2,1) = -6;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 2UL );
         checkNonZeros( mat, 6UL );

         const int maximum = max( mat );

         if( maximum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the end in a fully filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,1) = -2;
         mat(0,2) = -3;
         mat(1,0) = -4;
         mat(1,1) = -5;
         mat(1,2) =  6;

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 6UL );

         const int maximum = max( mat );

         if( maximum != 6 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 6\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the beginning in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 3UL, 5UL );
         mat(0,0) =  1;
         mat(0,2) = -2;
         mat(2,1) = -3;
         mat(4,0) = -4;
         mat(4,2) = -5;

         checkRows    ( mat, 5UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Third computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to find the maximum at the end in a partially filled matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 5UL, 5UL );
         mat(0,0) = -1;
         mat(0,4) = -2;
         mat(1,2) = -3;
         mat(2,0) = -4;
         mat(2,4) =  5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 5UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != 5 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fourth computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 5\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Attempt to detect 0 as the maximum value
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
         mat(0,0) = -1;
         mat(0,2) = -2;
         mat(1,1) = -3;
         mat(2,0) = -4;
         mat(2,2) = -5;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 5UL );

         const int maximum = max( mat );

         if( maximum != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Fifth computation failed\n"
                << " Details:\n"
                << "   Result: " << maximum << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c trace() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c trace() function for sparse matrices template. In case
// an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testTrace()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major trace()";

      // Determining the trace of a 0x0 matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         checkRows   ( mat, 0UL );
         checkColumns( mat, 0UL );

         const int trace = blaze::trace( mat );

         if( trace != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << trace << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the trace of a 3x3 matrix
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = -1;
         mat(0,2) = -3;
         mat(1,1) = -5;
         mat(1,2) =  6;
         mat(2,0) =  7;
         mat(2,2) = -9;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 6UL );

         const int trace = blaze::trace( mat );

         if( trace != -15 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << trace << "\n"
                << "   Expected result: -15\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the trace of a non-square matrix
      try
      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 2UL, 3UL );

         checkRows   ( mat, 2UL );
         checkColumns( mat, 3UL );

         const int trace = blaze::trace( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Trace computation on a non-square matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << trace << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major trace()";

      // Determining the trace of a 0x0 matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         checkRows   ( mat, 0UL );
         checkColumns( mat, 0UL );

         const int trace = blaze::trace( mat );

         if( trace != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: First computation failed\n"
                << " Details:\n"
                << "   Result: " << trace << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the trace of a 3x3 matrix
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
         mat(0,0) = -1;
         mat(0,2) = -3;
         mat(1,1) = -5;
         mat(1,2) =  6;
         mat(2,0) =  7;
         mat(2,2) = -9;

         checkRows    ( mat, 3UL );
         checkColumns ( mat, 3UL );
         checkNonZeros( mat, 6UL );

         const int trace = blaze::trace( mat );

         if( trace != -15 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Second computation failed\n"
                << " Details:\n"
                << "   Result: " << trace << "\n"
                << "   Expected result: -15\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the trace of a non-square matrix
      try
      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 2UL, 3UL );

         checkRows    ( mat, 2UL );
         checkColumns ( mat, 3UL );

         const int trace = blaze::trace( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Trace computation on a non-square matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << trace << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c l1Norm() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c l1Norm() function for sparse matrices template. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testL1Norm()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "l1Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 7UL );

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 0,  0,  1,  0,  1,  0,  0 },
                                                           { 0, -2,  0,  0,  0, -1,  0 },
                                                           { 0,  0,  0,  2,  0,  0,  0 } };

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 7 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 7\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "l1Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 7UL, 0 );

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat{ { 0,  0,  0 },
                                                              { 0, -2,  0 },
                                                              { 1,  0,  0 },
                                                              { 0,  0,  2 },
                                                              { 1,  0,  0 },
                                                              { 0, -1,  0 },
                                                              { 0,  0,  0 } };

         const int norm = blaze::l1Norm( mat );

         if( !isEqual( norm, 7 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L1 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 7\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c l2Norm() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c l2Norm() function for sparse matrices template. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testL2Norm()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "l2Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 7UL );

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 0,  0,  1,  0,  1, -2,  0 },
                                                           { 0, -2,  0,  0,  0, -1,  0 },
                                                           { 0,  1,  0,  2,  0,  0,  0 } };

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 4.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 4\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "l2Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 7UL );

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0,  0,  0 },
                                                              {  0, -2,  1 },
                                                              {  1,  0,  0 },
                                                              {  0,  0,  2 },
                                                              {  1,  0,  0 },
                                                              { -2, -1,  0 },
                                                              {  0,  0,  0 } };

         const double norm = blaze::l2Norm( mat );

         if( !isEqual( norm, 4.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L2 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 4\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c l3Norm() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c l3Norm() function for sparse matrices template. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testL3Norm()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "l3Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 7UL );

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 0,  0,  1,  0,  1, -2,  0 },
                                                           { 0, -2,  0,  0,  0, -1,  0 },
                                                           { 0,  0,  0,  2,  0,  0,  0 } };

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 3.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "l3Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 7UL );

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0,  0,  0 },
                                                              {  0, -2,  0 },
                                                              {  1,  0,  0 },
                                                              {  0,  0,  2 },
                                                              {  1,  0,  0 },
                                                              { -2, -1,  0 },
                                                              {  0,  0,  0 } };

         const double norm = blaze::l3Norm( mat );

         if( !isEqual( norm, 3.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L3 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c l4Norm() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c l4Norm() function for sparse matrices template. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testL4Norm()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "l4Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 7UL );

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 0,  0,  2,  0,  2, -2,  0 },
                                                           { 0, -2,  0,  0,  0, -1,  0 },
                                                           { 0,  0,  0,  2,  0,  0,  0 } };

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 3.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "l4Norm() function";

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 7UL );

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat{ { 0,  0,  2,  0,  2, -2,  0 },
                                                              { 0, -2,  0,  0,  0, -1,  0 },
                                                              { 0,  0,  0,  2,  0,  0,  0 } };

         const double norm = blaze::l4Norm( mat );

         if( !isEqual( norm, 3.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: L4 norm computation failed\n"
                << " Details:\n"
                << "   Result: " << norm << "\n"
                << "   Expected result: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c lpNorm() function for sparse matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c lpNorm() function for sparse matrices template. In
// case an error is detected, a \a std::runtime_error exception is thrown.
*/
void GeneralTest::testLpNorm()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "lpNorm() function";

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat;

         const double norm1 = blaze::lpNorm( mat, 2 );
         const double norm2 = blaze::lpNorm<2UL>( mat );

         if( !isEqual( norm1, 0.0 ) || !isEqual( norm2, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 7UL );

         const double norm1 = blaze::lpNorm( mat, 2 );
         const double norm2 = blaze::lpNorm<2UL>( mat );

         if( !isEqual( norm1, 0.0 ) || !isEqual( norm2, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const int norm1( blaze::lpNorm( mat, 1 ) );
         const int norm2( blaze::lpNorm<1UL>( mat ) );
         const int norm3( blaze::l1Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<1>(): " << norm1 << "\n"
                << "   lpNorm(1): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 2 ) );
         const double norm2( blaze::lpNorm<2UL>( mat ) );
         const double norm3( blaze::l2Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 3 ) );
         const double norm2( blaze::lpNorm<3UL>( mat ) );
         const double norm3( blaze::l3Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<4>(): " << norm1 << "\n"
                << "   lpNorm(4): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::rowMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 4 ) );
         const double norm2( blaze::lpNorm<4UL>( mat ) );
         const double norm3( blaze::l4Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<4>(): " << norm1 << "\n"
                << "   lpNorm(4): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "lpNorm() function";

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat;

         const double norm1 = blaze::lpNorm( mat, 2 );
         const double norm2 = blaze::lpNorm<2UL>( mat );

         if( !isEqual( norm1, 0.0 ) || !isEqual( norm2, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 7UL );

         const double norm1 = blaze::lpNorm( mat, 2 );
         const double norm2 = blaze::lpNorm<2UL>( mat );

         if( !isEqual( norm1, 0.0 ) || !isEqual( norm2, 0.0 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const int norm1( blaze::lpNorm( mat, 1 ) );
         const int norm2( blaze::lpNorm<1UL>( mat ) );
         const int norm3( blaze::l1Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<1>(): " << norm1 << "\n"
                << "   lpNorm(1): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 2 ) );
         const double norm2( blaze::lpNorm<2UL>( mat ) );
         const double norm3( blaze::l2Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<2>(): " << norm1 << "\n"
                << "   lpNorm(2): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 3 ) );
         const double norm2( blaze::lpNorm<3UL>( mat ) );
         const double norm3( blaze::l3Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<3>(): " << norm1 << "\n"
                << "   lpNorm(3): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 10UL );
         randomize( mat, 20UL, -5, 5 );

         const double norm1( blaze::lpNorm( mat, 4 ) );
         const double norm2( blaze::lpNorm<4UL>( mat ) );
         const double norm3( blaze::l4Norm( mat ) );

         if( !isEqual( norm1, norm3 ) || !isEqual( norm2, norm3 ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lp norm computation failed\n"
                << " Details:\n"
                << "   lpNorm<4>(): " << norm1 << "\n"
                << "   lpNorm(4): " << norm2 << "\n"
                << "   Expected result: " << norm3 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************

} // namespace sparsematrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running general SparseMatrix operation test..." << std::endl;

   try
   {
      RUN_SPARSEMATRIX_GENERAL_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during general SparseMatrix operation test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
