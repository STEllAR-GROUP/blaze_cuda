//=================================================================================================
/*!
//  \file src/mathtest/initializermatrix/ClassTest.cpp
//  \brief Source file for the InitializerMatrix class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/initializermatrix/ClassTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace initializermatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the InitializerMatrix class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testConstructors();
   testFunctionCall();
   testAt();
   testIterator();
   testNonZeros();
   testSwap();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the InitializerMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the InitializerMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   using blaze::initializer_list;


   //=====================================================================================
   // Single argument constructor
   //=====================================================================================

   {
      test_ = "InitializerMatrix single argument constructor (0x0)";

      initializer_list< initializer_list<int> > list = {};

      blaze::InitializerMatrix<int> mat( list );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "InitializerMatrix single argument constructor (3x4)";

      initializer_list< initializer_list<int> > list = { { 1, 0, 3, 4 },
                                                         { 0 },
                                                         { 2, 0, 5 } };

      blaze::InitializerMatrix<int> mat( list );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 5UL );
   }


   //=====================================================================================
   // Two argument constructor
   //=====================================================================================

   {
      test_ = "InitializerMatrix two argument constructor (3x0)";

      initializer_list< initializer_list<int> > list = { {},
                                                         {},
                                                         {} };

      blaze::InitializerMatrix<int> mat( list, 0UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "InitializerMatrix two argument constructor (3x4)";

      initializer_list< initializer_list<int> > list = { { 1, 0, 3, 4 },
                                                         { 0 },
                                                         { 2, 0, 5 } };

      blaze::InitializerMatrix<int> mat( list, 4UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 5UL );
   }

   {
      test_ = "InitializerMatrix two argument constructor (3x6)";

      initializer_list< initializer_list<int> > list = { { 1, 0, 3, 4 },
                                                         { 0 },
                                                         { 2, 0, 5 } };

      blaze::InitializerMatrix<int> mat( list, 6UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 6UL );
      checkNonZeros( mat, 5UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the InitializerMatrix function call operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the function call operator
// of the InitializerMatrix class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testFunctionCall()
{
   using blaze::initializer_list;


   test_ = "InitializerMatrix::operator()";

   initializer_list< initializer_list<int> > list = { { 1, 0, 3, 4 },
                                                      { 0 },
                                                      { 2, 0, 5 } };

   blaze::InitializerMatrix<int> mat( list, 6UL );

   // Access to the element (0,2)
   if( mat(0,2) != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Function call operator failed\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Access to the element (1,2)
   if( mat(1,2) != 0 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Function call operator failed\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c at() member function of the InitializerMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the \c at() member function
// of the InitializerMatrix class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testAt()
{
   using blaze::initializer_list;


   test_ = "InitializerMatrix::operator()";

   initializer_list< initializer_list<int> > list = { { 1, 0, 3, 4 },
                                                      { 0 },
                                                      { 2, 0, 5 } };

   blaze::InitializerMatrix<int> mat( list, 6UL );

   // Access to the element (0,2)
   if( mat.at(0,2) != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Function call operator failed\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Access to the element (1,2)
   if( mat.at(1,2) != 0 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Function call operator failed\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Attempt to access the element (3,0)
   try {
      mat.at(3,0);

      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Out-of-bound access succeeded\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }
   catch( std::out_of_range& ) {}

   // Attempt to access the element (2,6)
   try {
      mat.at(2,6);

      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Out-of-bound access succeeded\n"
          << " Details:\n"
          << "   Result:\n" << mat << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 )\n( 0 0 0 0 0 0 )\n( 2 0 5 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }
   catch( std::out_of_range& ) {}
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the InitializerMatrix iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the InitializerMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testIterator()
{
   using blaze::initializer_list;


   using MatrixType    = blaze::InitializerMatrix<int>;
   using Iterator      = MatrixType::Iterator;
   using ConstIterator = MatrixType::ConstIterator;

   initializer_list< initializer_list<int> > list = { {  0,  1 },
                                                      { -2,  0, -3 },
                                                      {  0,  4,  5 } };

   MatrixType mat( list, 4UL );

   // Testing the Iterator default constructor
   {
      test_ = "Iterator default constructor";

      Iterator it{};

      if( it != Iterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing the ConstIterator default constructor
   {
      test_ = "ConstIterator default constructor";

      ConstIterator it{};

      if( it != ConstIterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing conversion from Iterator to ConstIterator
   {
      test_ = "Iterator/ConstIterator conversion";

      ConstIterator it( begin( mat, 1UL ) );

      if( it == end( mat, 1UL ) || *it != -2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator conversion detected\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements in 0th row via Iterator (end-begin)
   {
      test_ = "Iterator subtraction (end-begin)";

      const ptrdiff_t number( end( mat, 0UL ) - begin( mat, 0UL ) );

      if( number != 4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements in 0th row via Iterator (begin-end)
   {
      test_ = "Iterator subtraction (begin-end)";

      const ptrdiff_t number( begin( mat, 0UL ) - end( mat, 0UL ) );

      if( number != -4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements in 1st row via ConstIterator (end-begin)
   {
      test_ = "ConstIterator subtraction (end-begin)";

      const ptrdiff_t number( cend( mat, 1UL ) - cbegin( mat, 1UL ) );

      if( number != 4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements in 1st row via ConstIterator (begin-end)
   {
      test_ = "ConstIterator subtraction (begin-end)";

      const ptrdiff_t number( cbegin( mat, 1UL ) - cend( mat, 1UL ) );

      if( number != -4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing read-only access via ConstIterator
   {
      test_ = "Read-only access via ConstIterator";

      ConstIterator it ( cbegin( mat, 2UL ) );
      ConstIterator end( cend( mat, 2UL ) );

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid initial iterator detected\n";
         throw std::runtime_error( oss.str() );
      }

      ++it;

      if( it == end || *it != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      --it;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it++;

      if( it == end || *it != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it--;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it += 2UL;

      if( it == end || *it != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator addition assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it -= 2UL;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator subtraction assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it + 2UL;

      if( it == end || *it != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar addition failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it - 2UL;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar subtraction failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = 4UL + it;

      if( it != end ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scalar/iterator addition failed\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the InitializerMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the InitializerMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testNonZeros()
{
   using blaze::initializer_list;


   test_ = "InitializerMatrix::nonZeros()";

   {
      initializer_list< initializer_list<int> > list = { { 0, 0, 0 },
                                                         { 0, 0, 0 } };

      blaze::InitializerMatrix<int> mat( list );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );

      if( mat(0,0) != 0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 0 || mat(1,1) != 0 || mat(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      initializer_list< initializer_list<int> > list = { { 0, 1, 2 },
                                                         { 0, 3, 0 } };

      blaze::InitializerMatrix<int> mat( list );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != 0 || mat(0,1) != 1 || mat(0,2) != 2 ||
          mat(1,0) != 0 || mat(1,1) != 3 || mat(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 1 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      initializer_list< initializer_list<int> > list = { { 0, 1, 2 },
                                                         { 0, 3, 0 } };

      blaze::InitializerMatrix<int> mat( list, 4UL );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 8UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != 0 || mat(0,1) != 1 || mat(0,2) != 2 || mat(0,3) != 0 ||
          mat(1,0) != 0 || mat(1,1) != 3 || mat(1,2) != 0 || mat(1,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 1 2 0 )\n( 0 3 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the InitializerMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the InitializerMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSwap()
{
   using blaze::initializer_list;


   test_ = "InitializerMatrix swap";

   initializer_list< initializer_list<int> > list1{ { 1, 2 },
                                                    { 0, 3 },
                                                    { 4 } };
   initializer_list< initializer_list<int> > list2{ { 6, 5, 4 },
                                                    { 3, 2, 1 }  };

   blaze::InitializerMatrix<int> mat1( list1 );
   blaze::InitializerMatrix<int> mat2( list2, 4UL );

   swap( mat1, mat2 );

   checkRows    ( mat1, 2UL );
   checkColumns ( mat1, 4UL );
   checkCapacity( mat1, 8UL );
   checkNonZeros( mat1, 6UL );
   checkNonZeros( mat1, 0UL, 3UL );
   checkNonZeros( mat1, 1UL, 3UL );

   if( mat1(0,0) != 6 || mat1(0,1) != 5 || mat1(0,2) != 4 ||
       mat1(1,0) != 3 || mat1(1,1) != 2 || mat1(1,2) != 1 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the first matrix failed\n"
          << " Details:\n"
          << "   Result:\n" << mat1 << "\n"
          << "   Expected result:\n( 6 5 4 )\n( 3 2 1 )\n";
      throw std::runtime_error( oss.str() );
   }

   checkRows    ( mat2, 3UL );
   checkColumns ( mat2, 2UL );
   checkCapacity( mat2, 6UL );
   checkNonZeros( mat2, 4UL );
   checkNonZeros( mat2, 0UL, 2UL );
   checkNonZeros( mat2, 1UL, 1UL );
   checkNonZeros( mat2, 2UL, 1UL );

   if( mat2(0,0) != 1 || mat2(0,1) != 2 ||
       mat2(1,0) != 0 || mat2(1,1) != 3 ||
       mat2(2,0) != 4 || mat2(2,1) != 0 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the second matrix failed\n"
          << " Details:\n"
          << "   Result:\n" << mat2 << "\n"
          << "   Expected result:\n( 1 2 )\n( 0 3 )\n( 4, 0 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************

} // namespace initializermatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running InitializerMatrix class test..." << std::endl;

   try
   {
      RUN_INITIALIZERMATRIX_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during InitializerMatrix class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
