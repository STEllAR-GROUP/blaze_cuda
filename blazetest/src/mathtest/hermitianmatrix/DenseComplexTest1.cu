//=================================================================================================
/*!
//  \file src/mathtest/hermitianmatrix/DenseComplexTest1.cpp
//  \brief Source file for the HermitianMatrix dense complex test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/CompressedMatrix.h>
#include <blaze/math/CustomMatrix.h>
#include <blaze/math/HybridMatrix.h>
#include <blaze/math/StaticMatrix.h>
#include <blaze/util/policies/ArrayDelete.h>
#include <blazetest/mathtest/hermitianmatrix/DenseComplexTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace hermitianmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the HermitianMatrix dense test.
//
// \exception std::runtime_error Operation error detected.
*/
DenseComplexTest::DenseComplexTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
   testSchurAssign();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the HermitianMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the HermitianMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseComplexTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   // Default constructor (StaticMatrix)
   {
      test_ = "Row-major HermitianMatrix default constructor (StaticMatrix)";

      const blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 0UL );
   }

   // Default constructor (HybridMatrix)
   {
      test_ = "Row-major HermitianMatrix default constructor (HybridMatrix)";

      const blaze::HermitianMatrix< blaze::HybridMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Default constructor (DynamicMatrix)
   {
      test_ = "Row-major HermitianMatrix default constructor (DynamicMatrix)";

      const HT herm;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   // Size constructor (HybridMatrix)
   {
      test_ = "Row-major HermitianMatrix size constructor (HybridMatrix)";

      const blaze::HermitianMatrix< blaze::HybridMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm( 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 0UL );
   }

   // Size constructor (DynamicMatrix)
   {
      test_ = "Row-major HermitianMatrix size constructor (DynamicMatrix)";

      const HT herm( 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 0UL );
   }


   //=====================================================================================
   // Row-major list initialization
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Row-major HermitianMatrix initializer list constructor (complete list)";

      const HT herm{ { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                     { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                     { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Row-major HermitianMatrix initializer list constructor (incomplete list)";

      const HT herm{ { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                     { cplx(2,-2), cplx(4, 0) },
                     { cplx(3, 3), cplx(0, 0), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 0) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0, 0) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 0) )\n"
                                     "( (3, 3) (0, 0) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major array initialization
   //=====================================================================================

   // Dynamic array initialization constructor
   {
      test_ = "Row-major HermitianMatrix dynamic array initialization constructor";

      std::unique_ptr<cplx[]> array( new cplx[9] );
      array[0] = cplx(1, 0);
      array[1] = cplx(2, 2);
      array[2] = cplx(3,-3);
      array[3] = cplx(2,-2);
      array[4] = cplx(4, 0);
      array[5] = cplx(0, 5);
      array[6] = cplx(3, 3);
      array[7] = cplx(0,-5);
      array[8] = cplx(6, 0);
      const HT herm( 3UL, array.get() );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Static array initialization constructor
   {
      test_ = "Row-major HermitianMatrix static array initialization constructor";

      const cplx array[3][3] = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                                 { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                                 { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };
      const HT herm( array );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major custom matrix constructors
   //=====================================================================================

   // Custom matrix constructor (ElementType*, size_t)
   {
      test_ = "Row-major HermitianMatrix custom matrix constructor (ElementType*, size_t)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<cplx,unaligned,unpadded,rowMajor>;
      std::unique_ptr<cplx[]> memory( new cplx[5UL] );
      memory[1] = cplx(1, 0);
      memory[2] = cplx(2, 1);
      memory[3] = cplx(2,-1);
      memory[4] = cplx(3, 0);
      const blaze::HermitianMatrix<UnalignedUnpadded> herm( memory.get()+1UL, 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 4UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2,1) ||
          herm(1,0) != cplx(2,-1) || herm(1,1) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2,1) )\n( (2,-1) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Custom matrix constructor (ElementType*, size_t, size_t)
   {
      test_ = "Row-major HermitianMatrix custom matrix constructor (ElementType*, size_t, size_t)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<cplx,unaligned,unpadded,rowMajor>;
      std::unique_ptr<cplx[]> memory( new cplx[11UL] );
      memory[1] = cplx(1, 0);
      memory[2] = cplx(2, 1);
      memory[6] = cplx(2,-1);
      memory[7] = cplx(3, 0);
      const blaze::HermitianMatrix<UnalignedUnpadded> herm( memory.get()+1UL, 2UL, 5UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 4UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2,1) ||
          herm(1,0) != cplx(2,-1) || herm(1,1) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2,1) )\n( (2,-1) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Row-major HermitianMatrix copy constructor (0x0)";

      const HT herm1;
      const HT herm2( herm1 );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Row-major HermitianMatrix copy constructor (3x3)";

      HT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      const HT herm2( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Row-major HermitianMatrix move constructor (0x0)";

      HT herm1;
      HT herm2( std::move( herm1 ) );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Row-major HermitianMatrix move constructor (3x3)";

      HT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2( std::move( herm1 ) );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Row-major HermitianMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<cplx,blaze::rowMajor> mat;
      const HT herm( mat );

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Conversion constructor (Hermitian)
   {
      test_ = "Row-major HermitianMatrix conversion constructor (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      const HT herm( mat );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (invalid diagonal)
   {
      test_ = "Row-major HermitianMatrix conversion constructor (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         const HT herm( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-Hermitian HermitianMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (invalid pair)
   {
      test_ = "Row-major HermitianMatrix conversion constructor (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         const HT herm( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-Hermitian HermitianMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (HermitianMatrix)
   {
      test_ = "Row-major HermitianMatrix conversion constructor (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      const HT herm2( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   // Default constructor (StaticMatrix)
   {
      test_ = "Column-major HermitianMatrix default constructor (StaticMatrix)";

      const blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 0UL );
   }

   // Default constructor (HybridMatrix)
   {
      test_ = "Column-major HermitianMatrix default constructor (HybridMatrix)";

      const blaze::HermitianMatrix< blaze::HybridMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Default constructor (DynamicMatrix)
   {
      test_ = "Column-major HermitianMatrix default constructor (DynamicMatrix)";

      const OHT herm;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   // Size constructor (HybridMatrix)
   {
      test_ = "Column-major HermitianMatrix size constructor (HybridMatrix)";

      const blaze::HermitianMatrix< blaze::HybridMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm( 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 0UL );
   }

   // Size constructor (DynamicMatrix)
   {
      test_ = "Column-major HermitianMatrix size constructor (DynamicMatrix)";

      const OHT herm( 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 0UL );
   }


   //=====================================================================================
   // Column-major list initialization
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Column-major HermitianMatrix initializer list constructor (complete list)";

      const OHT herm{ { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                      { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                      { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Column-major HermitianMatrix initializer list constructor (incomplete list)";

      const OHT herm{ { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                      { cplx(2,-2), cplx(4, 0) },
                      { cplx(3, 3), cplx(0, 0), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 0) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0, 0) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 0) )\n"
                                     "( (3, 3) (0, 0) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major array initialization
   //=====================================================================================

   // Dynamic array initialization constructor
   {
      test_ = "Column-major HermitianMatrix dynamic array initialization constructor";

      std::unique_ptr<cplx[]> array( new cplx[9] );
      array[0] = cplx(1, 0);
      array[1] = cplx(2,-2);
      array[2] = cplx(3, 3);
      array[3] = cplx(2, 2);
      array[4] = cplx(4, 0);
      array[5] = cplx(0,-5);
      array[6] = cplx(3,-3);
      array[7] = cplx(0, 5);
      array[8] = cplx(6, 0);
      const OHT herm( 3UL, array.get() );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Static array initialization constructor
   {
      test_ = "Column-major HermitianMatrix static array initialization constructor";

      const cplx array[3][3] = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                                 { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                                 { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };
      const OHT herm( array );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major custom matrix constructors
   //=====================================================================================

   // Custom matrix constructor (ElementType*, size_t)
   {
      test_ = "Column-major HermitianMatrix custom matrix constructor (ElementType*, size_t)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<cplx,unaligned,unpadded,columnMajor>;
      std::unique_ptr<cplx[]> memory( new cplx[5UL] );
      memory[1] = cplx(1, 0);
      memory[2] = cplx(2,-1);
      memory[3] = cplx(2, 1);
      memory[4] = cplx(3, 0);
      const blaze::HermitianMatrix<UnalignedUnpadded> herm( memory.get()+1UL, 2UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 4UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2,1) ||
          herm(1,0) != cplx(2,-1) || herm(1,1) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2,1) )\n( (2,-1) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Custom matrix constructor (ElementType*, size_t, size_t)
   {
      test_ = "Column-major HermitianMatrix custom matrix constructor (ElementType*, size_t, size_t)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<cplx,unaligned,unpadded,columnMajor>;
      std::unique_ptr<cplx[]> memory( new cplx[11UL] );
      memory[1] = cplx(1, 0);
      memory[2] = cplx(2,-1);
      memory[6] = cplx(2, 1);
      memory[7] = cplx(3, 0);
      const blaze::HermitianMatrix<UnalignedUnpadded> herm( memory.get()+1UL, 2UL, 5UL );

      checkRows    ( herm, 2UL );
      checkColumns ( herm, 2UL );
      checkCapacity( herm, 4UL );
      checkNonZeros( herm, 4UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2,1) ||
          herm(1,0) != cplx(2,-1) || herm(1,1) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2,1) )\n( (2,-1) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Column-major HermitianMatrix copy constructor (0x0)";

      const OHT herm1;
      const OHT herm2( herm1 );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Column-major HermitianMatrix copy constructor (3x3)";

      OHT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      const OHT herm2( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Column-major HermitianMatrix move constructor (0x0)";

      OHT herm1;
      OHT herm2( std::move( herm1 ) );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Column-major HermitianMatrix move constructor (3x3)";

      OHT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2( std::move( herm1 ) );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Column-major HermitianMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<cplx,blaze::columnMajor> mat;
      const OHT herm( mat );

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Conversion constructor (Hermitian)
   {
      test_ = "Column-major HermitianMatrix conversion constructor (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      const OHT herm( mat );

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (invalid diagonal)
   {
      test_ = "Column-major HermitianMatrix conversion constructor (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         const OHT herm( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-Hermitian HermitianMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (invalid pair)
   {
      test_ = "Column-major HermitianMatrix conversion constructor (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         const OHT herm( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-Hermitian HermitianMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (HermitianMatrix)
   {
      test_ = "Column-major HermitianMatrix conversion constructor (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      const OHT herm2( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the HermitianMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the HermitianMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseComplexTest::testAssignment()
{
   //=====================================================================================
   // Row-major list assignment
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Row-major HermitianMatrix initializer list assignment (complete list)";

      HT herm;
      herm = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
               { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
               { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 3UL );
      checkNonZeros( herm, 2UL, 3UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Row-major HermitianMatrix initializer list assignment (incomplete list)";

      HT herm;
      herm = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
               { cplx(2,-2), cplx(4, 0) },
               { cplx(3, 3), cplx(0, 0), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 0) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0, 0) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 0) )\n"
                                     "( (3, 3) (0, 0) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major array assignment
   //=====================================================================================

   // Array assignment
   {
      test_ = "Row-major HermitianMatrix array assignment";

      const cplx array[3][3] = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                                 { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                                 { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };
      HT herm;
      herm = array;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 3UL );
      checkNonZeros( herm, 2UL, 3UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Row-major HermitianMatrix copy assignment (0x0)";

      HT herm1, herm2;

      herm2 = herm1;

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Row-major HermitianMatrix copy assignment (3x3)";

      HT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Row-major HermitianMatrix move assignment (0x0)";

      HT herm1, herm2;

      herm2 = std::move( herm1 );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Row-major HermitianMatrix move assignment (3x3)";

      HT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = std::move( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major HermitianMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<cplx,blaze::rowMajor> mat;

      HT herm;
      herm = mat;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Row-major/row-major dense matrix assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      HT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      HT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major HermitianMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<cplx,blaze::rowMajor> mat;

      HT herm;
      herm = mat;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Row-major/row-major sparse matrix assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,2) = cplx( 3, 0);

      try {
         HT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 7UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 7UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      HT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major list assignment
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Column-major HermitianMatrix initializer list assignment (complete list)";

      OHT herm;
      herm = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
               { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
               { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 3UL );
      checkNonZeros( herm, 2UL, 3UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Column-major HermitianMatrix initializer list assignment (incomplete list)";

      OHT herm;
      herm = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
               { cplx(2,-2), cplx(4, 0) },
               { cplx(3, 3), cplx(0, 0), cplx(6, 0) } };

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 0) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0, 0) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 0) )\n"
                                     "( (3, 3) (0, 0) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major array assignment
   //=====================================================================================

   // Array assignment
   {
      test_ = "Column-major HermitianMatrix array assignment";

      const cplx array[3][3] = { { cplx(1, 0), cplx(2, 2), cplx(3,-3) },
                                 { cplx(2,-2), cplx(4, 0), cplx(0, 5) },
                                 { cplx(3, 3), cplx(0,-5), cplx(6, 0) } };
      OHT herm;
      herm = array;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 9UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 3UL );
      checkNonZeros( herm, 2UL, 3UL );

      if( herm(0,0) != cplx(1, 0) || herm(0,1) != cplx(2, 2) || herm(0,2) != cplx(3,-3) ||
          herm(1,0) != cplx(2,-2) || herm(1,1) != cplx(4, 0) || herm(1,2) != cplx(0, 5) ||
          herm(2,0) != cplx(3, 3) || herm(2,1) != cplx(0,-5) || herm(2,2) != cplx(6, 0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( (1, 0) (2, 2) (3,-3) )\n"
                                     "( (2,-2) (4, 0) (0, 5) )\n"
                                     "( (3, 3) (0,-5) (6, 0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Column-major HermitianMatrix copy assignment (0x0)";

      OHT herm1, herm2;

      herm2 = herm1;

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Column-major HermitianMatrix copy assignment (3x3)";

      OHT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Column-major HermitianMatrix move assignment (0x0)";

      OHT herm1, herm2;

      herm2 = std::move( herm1 );

      checkRows    ( herm2, 0UL );
      checkColumns ( herm2, 0UL );
      checkNonZeros( herm2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Column-major HermitianMatrix move assignment (3x3)";

      OHT herm1( 3UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = std::move( herm1 );

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major HermitianMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<cplx,blaze::rowMajor> mat;

      OHT herm;
      herm = mat;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Column-major/row-major dense matrix assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      OHT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      OHT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(1,2) = cplx( 0, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,1) = cplx( 0, 0);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::rowMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::StaticMatrix<cplx,3UL,3UL,blaze::columnMajor> > herm1;
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major HermitianMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<cplx,blaze::rowMajor> mat;

      OHT herm;
      herm = mat;

      checkRows    ( herm, 0UL );
      checkColumns ( herm, 0UL );
      checkNonZeros( herm, 0UL );
   }

   // Column-major/row-major sparse matrix assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm;
      herm = mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkNonZeros( herm, 7UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-4,-1) || herm(0,2) != cplx(7,3) ||
          herm(1,0) != cplx(-4, 1) || herm(1,1) != cplx( 2, 0) || herm(1,2) != cplx(0,0) ||
          herm(2,0) != cplx( 7,-3) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 1);
      mat(2,0) = cplx( 7,-3);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = cplx( 1, 0);
      mat(0,1) = cplx(-4,-1);
      mat(0,2) = cplx( 7, 3);
      mat(1,0) = cplx(-4, 1);
      mat(1,1) = cplx( 2, 0);
      mat(2,0) = cplx( 7, 3);
      mat(2,2) = cplx( 3, 0);

      try {
         OHT herm;
         herm = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 7UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 7UL );
      herm1(0,0) = cplx( 1, 0);
      herm1(0,1) = cplx(-4,-1);
      herm1(0,2) = cplx( 7, 3);
      herm1(1,1) = cplx( 2, 0);
      herm1(2,2) = cplx( 3, 0);

      OHT herm2;
      herm2 = herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkNonZeros( herm2, 7UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-4,-1) || herm2(0,2) != cplx(7,3) ||
          herm2(1,0) != cplx(-4, 1) || herm2(1,1) != cplx( 2, 0) || herm2(1,2) != cplx(0,0) ||
          herm2(2,0) != cplx( 7,-3) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx(3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-4,-1) (7,3) )\n"
                                     "( (-4, 1) ( 2, 0) (0,0) )\n"
                                     "( ( 7,-3) ( 0, 0) (3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the HermitianMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the HermitianMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseComplexTest::testAddAssign()
{
   //=====================================================================================
   // Row-major dense matrix addition assignment
   //=====================================================================================

   // Row-major/row-major dense matrix addition assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix addition assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix addition assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix addition assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix addition assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix addition assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix addition assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix addition assignment
   //=====================================================================================

   // Column-major/row-major dense matrix addition assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix addition assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix addition assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix addition assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix addition assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix addition assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix addition assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm += mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix addition assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix addition assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix addition assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix addition assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 += herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the HermitianMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the HermitianMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseComplexTest::testSubAssign()
{
   //=====================================================================================
   // Row-major dense matrix subtraction assignment
   //=====================================================================================

   // Row-major/row-major dense matrix subtraction assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix subtraction assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix subtraction assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix subtraction assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix subtraction assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix subtraction assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix subtraction assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix subtraction assignment
   //=====================================================================================

   // Column-major/row-major dense matrix subtraction assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix subtraction assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix subtraction assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix subtraction assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix subtraction assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix subtraction assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix subtraction assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm -= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 7UL );
      checkNonZeros( herm, 0UL, 3UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 2UL );

      if( herm(0,0) != cplx( 1, 0) || herm(0,1) != cplx(-6,-2) || herm(0,2) != cplx(13,1) ||
          herm(1,0) != cplx(-6, 2) || herm(1,1) != cplx( 5, 0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(13,-1) || herm(2,1) != cplx( 0, 0) || herm(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix subtraction assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 2);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3,-9);
      mat(2,0) = cplx(-6,-2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix subtraction assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx( 2, 1);
      mat(0,2) = cplx(-6, 8);
      mat(1,0) = cplx( 2,-1);
      mat(1,1) = cplx(-3, 0);
      mat(2,0) = cplx(-6,-9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix subtraction assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix subtraction assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx( 2,1);
      herm1(0,2) = cplx(-6,2);
      herm1(1,1) = cplx(-3,0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 -= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 7UL );
      checkNonZeros( herm2, 0UL, 3UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 2UL );

      if( herm2(0,0) != cplx( 1, 0) || herm2(0,1) != cplx(-6,-2) || herm2(0,2) != cplx(13,1) ||
          herm2(1,0) != cplx(-6, 2) || herm2(1,1) != cplx( 5, 0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(13,-1) || herm2(2,1) != cplx( 0, 0) || herm2(2,2) != cplx( 3,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 1, 0) (-6,-2) (13,1) )\n"
                                     "( (-6, 2) ( 5, 0) ( 0,0) )\n"
                                     "( (13,-1) ( 0, 0) ( 3,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the HermitianMatrix Schur product assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the Schur product assignment operators of the HermitianMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseComplexTest::testSchurAssign()
{
   //=====================================================================================
   // Row-major dense matrix Schur product assignment
   //=====================================================================================

   // Row-major/row-major dense matrix Schur product assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix Schur product assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix Schur product assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix Schur product assignment (Hermitian)
   {
      test_ = "Row-major/row-major HermitianMatrix sparse matrix Schur product assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix Schur product assignment (Hermitian)
   {
      test_ = "Row-major/column-major HermitianMatrix sparse matrix Schur product assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      HT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Row-major/row-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Row-major/column-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      HT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix Schur product assignment
   //=====================================================================================

   // Column-major/row-major dense matrix Schur product assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix Schur product assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (Hermitian)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::DynamicMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, cplx(0,0) );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      HT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      OHT herm1( 3UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix Schur product assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix Schur product assignment (Hermitian)
   {
      test_ = "Column-major/row-major HermitianMatrix sparse matrix Schur product assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix Schur product assignment (Hermitian)
   {
      test_ = "Column-major/column-major HermitianMatrix sparse matrix Schur product assignment (Hermitian)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 2);
      mat.insert( 1UL, 2UL, cplx(0,0) );

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      herm %= mat;

      checkRows    ( herm, 3UL );
      checkColumns ( herm, 3UL );
      checkCapacity( herm, 9UL );
      checkNonZeros( herm, 5UL );
      checkNonZeros( herm, 0UL, 2UL );
      checkNonZeros( herm, 1UL, 2UL );
      checkNonZeros( herm, 2UL, 1UL );

      if( herm(0,0) != cplx( 0, 0) || herm(0,1) != cplx(7,6) || herm(0,2) != cplx(48,4) ||
          herm(1,0) != cplx( 7,-6) || herm(1,1) != cplx(6,0) || herm(1,2) != cplx( 0,0) ||
          herm(2,0) != cplx(48,-4) || herm(2,1) != cplx(0,0) || herm(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix Schur product assignment (invalid diagonal)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid diagonal)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-2);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 9);
      mat(2,0) = cplx( 6, 2);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::rowMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix Schur product assignment (invalid pair)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (invalid pair)";

      blaze::CompressedMatrix<cplx,blaze::columnMajor> mat( 3UL, 3UL, 5UL );
      mat(0,1) = cplx(-2,-1);
      mat(0,2) = cplx( 6,-8);
      mat(1,0) = cplx(-2, 1);
      mat(1,1) = cplx( 3, 0);
      mat(2,0) = cplx( 6, 9);

      OHT herm( 3UL );
      herm(0,0) = cplx( 1, 0);
      herm(0,1) = cplx(-4,-1);
      herm(0,2) = cplx( 7, 3);
      herm(1,1) = cplx( 2, 0);
      herm(2,2) = cplx( 3, 0);

      try {
         herm %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-Hermitian row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << herm << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Column-major/row-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::rowMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix Schur product assignment (HermitianMatrix)
   {
      test_ = "Column-major/column-major HermitianMatrix dense matrix Schur product assignment (HermitianMatrix)";

      blaze::HermitianMatrix< blaze::CompressedMatrix<cplx,blaze::columnMajor> > herm1( 3UL, 5UL );
      herm1(0,1) = cplx(-2,-1);
      herm1(0,2) = cplx( 6,-2);
      herm1(1,1) = cplx( 3, 0);

      OHT herm2( 3UL );
      herm2(0,0) = cplx( 1, 0);
      herm2(0,1) = cplx(-4,-1);
      herm2(0,2) = cplx( 7, 3);
      herm2(1,1) = cplx( 2, 0);
      herm2(2,2) = cplx( 3, 0);

      herm2 %= herm1;

      checkRows    ( herm2, 3UL );
      checkColumns ( herm2, 3UL );
      checkCapacity( herm2, 9UL );
      checkNonZeros( herm2, 5UL );
      checkNonZeros( herm2, 0UL, 2UL );
      checkNonZeros( herm2, 1UL, 2UL );
      checkNonZeros( herm2, 2UL, 1UL );

      if( herm2(0,0) != cplx( 0, 0) || herm2(0,1) != cplx(7,6) || herm2(0,2) != cplx(48,4) ||
          herm2(1,0) != cplx( 7,-6) || herm2(1,1) != cplx(6,0) || herm2(1,2) != cplx( 0,0) ||
          herm2(2,0) != cplx(48,-4) || herm2(2,1) != cplx(0,0) || herm2(2,2) != cplx( 0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << herm2 << "\n"
             << "   Expected result:\n( ( 0, 0) (7,6) (48,4) )\n"
                                     "( ( 7,-6) (6,0) ( 0,0) )\n"
                                     "( (48,-4) (0,0) ( 0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace hermitianmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running HermitianMatrix dense complex test (part 1)..." << std::endl;

   try
   {
      RUN_HERMITIANMATRIX_DENSECOMPLEX_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during HermitianMatrix dense complex test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
