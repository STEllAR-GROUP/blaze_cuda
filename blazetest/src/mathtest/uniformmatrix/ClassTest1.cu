#include "hip/hip_runtime.h"
//=================================================================================================
/*!
//  \file src/mathtest/uniformmatrix/ClassTest1.cpp
//  \brief Source file for the UniformMatrix class test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/CompressedMatrix.h>
#include <blaze/math/CustomMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/util/Memory.h>
#include <blaze/util/policies/Deallocate.h>
#include <blaze/util/Random.h>
#include <blazetest/mathtest/uniformmatrix/ClassTest.h>
#include <blazetest/mathtest/RandomMaximum.h>
#include <blazetest/mathtest/RandomMinimum.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace uniformmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the UniformMatrix class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the UniformMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the UniformMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   // Default constructor
   {
      test_ = "Row-major UniformMatrix default constructor";

      blaze::UniformMatrix<int,blaze::rowMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix size constructor (0x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 0UL, 0UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix size constructor (0x4)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 0UL, 4UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix size constructor (3x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 3UL, 0UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix size constructor (3x4)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 3UL, 4UL );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
   }


   //=====================================================================================
   // Row-major homogeneous initialization
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix homogeneous initialization constructor (0x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 0UL, 0UL, 2 );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix homogeneous initialization constructor (0x4)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 0UL, 4UL, 2 );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix homogeneous initialization constructor (3x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 3UL, 0UL, 2 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix homogeneous initialization constructor (3x4)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 3UL, 4UL, 2 );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 4UL );
      checkNonZeros( mat,  1UL, 4UL );
      checkNonZeros( mat,  2UL, 4UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix copy constructor (0x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 0UL, 0UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix copy constructor (0x3)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 0UL, 3UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix copy constructor (2x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 0UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix copy constructor (2x3)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix move constructor (0x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 0UL, 0UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix move constructor (0x3)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 0UL, 3UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix move constructor (2x0)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 0UL );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major UniformMatrix copy constructor (2x3)";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix constructor (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix constructor (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Row-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix constructor (uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix constructor (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix constructor (uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix constructor (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix default constructor";

      blaze::UniformMatrix<int,blaze::columnMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix size constructor (0x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 0UL, 0UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix size constructor (0x4)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 0UL, 4UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix size constructor (3x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 3UL, 0UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix size constructor (3x4)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 3UL, 4UL );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
   }


   //=====================================================================================
   // Column-major homogeneous initialization
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix homogeneous initialization constructor (0x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 0UL, 0UL, 2 );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix homogeneous initialization constructor (0x4)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 0UL, 4UL, 2 );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix homogeneous initialization constructor (3x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 3UL, 0UL, 2 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix homogeneous initialization constructor (3x4)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 3UL, 4UL, 2 );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 3UL );
      checkNonZeros( mat,  1UL, 3UL );
      checkNonZeros( mat,  2UL, 3UL );
      checkNonZeros( mat,  3UL, 3UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix copy constructor (0x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 0UL, 0UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix copy constructor (0x3)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 0UL, 3UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix copy constructor (2x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 0UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix copy constructor (2x3)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix move constructor (0x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 0UL, 0UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix move constructor (0x3)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 0UL, 3UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix move constructor (2x0)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 0UL );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major UniformMatrix move constructor (2x3)";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix constructor (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix constructor (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix constructor (uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix constructor (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix constructor (uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix constructor (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( mat1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-uniform UniformMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the UniformMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the UniformMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAssignment()
{
   //=====================================================================================
   // Row-major homogeneous assignment
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix homogeneous assignment";

      blaze::UniformMatrix<int,blaze::rowMajor> mat( 3UL, 4UL );
      mat = 2;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 4UL );
      checkNonZeros( mat,  1UL, 4UL );
      checkNonZeros( mat,  2UL, 4UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix copy assignment";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major UniformMatrix copy assignment stress test";

      using RandomMatrixType = blaze::UniformMatrix<int,blaze::rowMajor>;

      blaze::UniformMatrix<int,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   {
      test_ = "Row-major UniformMatrix move assignment";

      blaze::UniformMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL,  2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 4UL, 1UL, 11 );

      mat2 = std::move( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix assignment (uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix assignment (uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major homogeneous assignment
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix homogeneous assigment";

      blaze::UniformMatrix<int,blaze::columnMajor> mat( 3UL, 4UL );
      mat = 2;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 3UL );
      checkNonZeros( mat,  1UL, 3UL );
      checkNonZeros( mat,  2UL, 3UL );
      checkNonZeros( mat,  3UL, 3UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix copy assignment";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major UniformMatrix copy assignment stress test";

      using RandomMatrixType = blaze::UniformMatrix<int,blaze::columnMajor>;

      blaze::UniformMatrix<int,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   {
      test_ = "Column-major UniformMatrix move assignment";

      blaze::UniformMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL,  2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 4UL, 1UL, 11 );

      mat2 = std::move( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );
      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 2;
      mat1(0,1) = 2;
      mat1(0,2) = 2;
      mat1(1,0) = 2;
      mat1(1,1) = 2;
      mat1(1,2) = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix assignment (uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix assignment (uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };
      blaze::UniformMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 2 || mat2(0,1) != 2 || mat2(0,2) != 2 ||
          mat2(1,0) != 2 || mat2(1,1) != 2 || mat2(1,2) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 2 2 2 )\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2;
         mat2 = mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the UniformMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the UniformMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAddAssign()
{
   //=====================================================================================
   // Row-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix addition assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix addition assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix addition assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix addition assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Row-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix addition assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix addition assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix addition assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix addition assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix addition assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix addition assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix addition assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 3 || mat2(0,1) != 3 || mat2(0,2) != 3 ||
          mat2(1,0) != 3 || mat2(1,1) != 3 || mat2(1,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 3 3 3 )\n( 3 3 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix addition assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 += mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the UniformMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the UniformMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSubAssign()
{
   //=====================================================================================
   // Row-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix subtraction assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix dense matrix subtraction assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix subtraction assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix dense matrix subtraction assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Row-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major UniformMatrix sparse matrix subtraction assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major UniformMatrix sparse matrix subtraction assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix subtraction assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::rowMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix dense matrix subtraction assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix subtraction assignment (mixed type)";

      blaze::UniformMatrix<short,blaze::columnMajor> mat1( 2UL, 3UL, 2 );

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 2;

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix dense matrix subtraction assignment (non-uniform)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform dense matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major UniformMatrix sparse matrix subtraction assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 2, 2 } };

      blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != -1 || mat2(0,1) != -1 || mat2(0,2) != -1 ||
          mat2(1,0) != -1 || mat2(1,1) != -1 || mat2(1,2) != -1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( -1 -1 -1 )\n( -1 -1 -1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major UniformMatrix sparse matrix subtraction assignment (non-uniform)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 2, 2, 2 }, { 2, 0, 2 } };

      try {
         blaze::UniformMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL, 1 );
         mat2 -= mat1;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-uniform sparse matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************

} // namespace uniformmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running UniformMatrix class test (part 1)..." << std::endl;

   try
   {
      RUN_UNIFORMMATRIX_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during UniformMatrix class test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
