//=================================================================================================
/*!
//  \file src/mathtest/ql/DenseTest.cpp
//  \brief Source file for the dense matrix QL test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/DiagonalMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/HermitianMatrix.h>
#include <blaze/math/LowerMatrix.h>
#include <blaze/math/SymmetricMatrix.h>
#include <blaze/math/UniLowerMatrix.h>
#include <blaze/math/UniUpperMatrix.h>
#include <blaze/math/UpperMatrix.h>
#include <blazetest/mathtest/ql/DenseTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace ql {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the DenseTest test.
//
// \exception std::runtime_error Error during QL decomposition detected.
*/
DenseTest::DenseTest()
{
   testGeneral();
   testSymmetric();
   testHermitian();
   testLower();
   testUniLower();
   testUpper();
   testUniUpper();
   testDiagonal();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for general matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for general matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testGeneral()
{
   //testRandom< blaze::DynamicMatrix<float,blaze::rowMajor   > >();
   //testRandom< blaze::DynamicMatrix<float,blaze::columnMajor> >();

   testRandom< blaze::DynamicMatrix<double,blaze::rowMajor   > >();
   testRandom< blaze::DynamicMatrix<double,blaze::columnMajor> >();

   //testRandom< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > >();
   //testRandom< blaze::DynamicMatrix<cfloat,blaze::columnMajor> >();

   testRandom< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > >();
   testRandom< blaze::DynamicMatrix<cdouble,blaze::columnMajor> >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for symmetric matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for symmetric matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSymmetric()
{
   //testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::SymmetricMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for Hermitian matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for Hermitian matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testHermitian()
{
   //testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::HermitianMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for lower matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for lower matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testLower()
{
   //testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::LowerMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for unilower matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for unilower matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testUniLower()
{
   //testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::UniLowerMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for upper matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for upper matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testUpper()
{
   //testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::UpperMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for uniupper matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for uniupper matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testUniUpper()
{
   //testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::UniUpperMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the QL decomposition functionality for diagonal matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function tests the dense matrix QL decomposition for diagonal matrices. In case an error
// is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testDiagonal()
{
   //testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<float,blaze::rowMajor   > > >();
   //testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<float,blaze::columnMajor> > >();

   testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<double,blaze::rowMajor   > > >();
   testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > >();

   //testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<cfloat,blaze::rowMajor   > > >();
   //testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<cfloat,blaze::columnMajor> > >();

   testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<cdouble,blaze::rowMajor   > > >();
   testRandom< blaze::DiagonalMatrix< blaze::DynamicMatrix<cdouble,blaze::columnMajor> > >();
}
//*************************************************************************************************

} // namespace ql

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running dense matrix QL decomposition test..." << std::endl;

   try
   {
      RUN_DENSE_QL_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during dense matrix QL decomposition test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
