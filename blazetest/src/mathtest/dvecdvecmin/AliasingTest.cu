#include "hip/hip_runtime.h"
//=================================================================================================
/*!
//  \file src/mathtest/dvecdvecmin/AliasingTest.cpp
//  \brief Source file for the dense vector/dense vector minimum aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/Subvector.h>
#include <blazetest/mathtest/dvecdvecmin/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace dvecdvecmin {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : da4_   ( 4UL )
   , db3_   ( 3UL )
   , dc3_   ( 3UL )
   , sa4_   ( 4UL )
   , sb3_   ( 3UL )
   , dA3x4_ ( 3UL, 4UL )
   , dB3x3_ ( 3UL, 3UL )
   , result_()
   , test_  ()
{
   testDVecDVecMin();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the dense vector/dense vector minimum.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense vector/dense vector minimum.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testDVecDVecMin()
{
   using blaze::subvector;


   //=====================================================================================
   // Minimum
   //=====================================================================================

   // Assignment to left-hand side operand (1)
   {
      test_ = "DVecDVecMin - Assignment to left-hand side operand (1)";

      initialize();

      result_ = min( db3_, dc3_ );
      db3_    = min( db3_, dc3_ );

      checkResult( db3_, result_ );
   }

   // Assignment to left-hand side operand (2)
   {
      test_ = "DVecDVecMin - Assignment to left-hand side operand (2)";

      initialize();

      result_ = min( db3_, eval( dc3_ )  );
      db3_    = min( db3_, eval( dc3_ ) );

      checkResult( db3_, result_ );
   }

   // Assignment to left-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Assignment to left-hand side dense compound operand";

      initialize();

      result_ = min( ( dA3x4_ * da4_ ), dc3_ );
      da4_    = min( ( dA3x4_ * da4_ ), dc3_ );

      checkResult( da4_, result_ );
   }

   // Assignment to left-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Assignment to left-hand side sparse compound operand";

      initialize();

      result_ = min( ( dA3x4_ * sa4_ ), dc3_ );
      sa4_    = min( ( dA3x4_ * sa4_ ), dc3_ );

      checkResult( sa4_, result_ );
   }

   // Assignment to left-hand side subvector operand
   {
      test_ = "DVecDVecMin - Assignment to left-hand side subvector operand";

      initialize();

      result_ = min( subvector( da4_, 1UL, 3UL ), db3_ );
      da4_    = min( subvector( da4_, 1UL, 3UL ), db3_ );

      checkResult( da4_, result_ );
   }

   // Assignment to right-hand side operand (1)
   {
      test_ = "DVecDVecMin - Assignment to right-hand side operand (1)";

      initialize();

      result_ = min( db3_, dc3_ );
      dc3_    = min( db3_, dc3_ );

      checkResult( dc3_, result_ );
   }

   // Assignment to right-hand side operand (2)
   {
      test_ = "DVecDVecMin - Assignment to right-hand side operand (2)";

      initialize();

      result_ = min( eval( db3_ ), dc3_ );
      dc3_    = min( eval( db3_ ), dc3_ );

      checkResult( dc3_, result_ );
   }

   // Assignment to right-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Assignment to right-hand side dense compound operand";

      initialize();

      result_ = min( db3_, ( dA3x4_ * da4_ ) );
      da4_    = min( db3_, ( dA3x4_ * da4_ ) );

      checkResult( da4_, result_ );
   }

   // Assignment to right-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Assignment to right-hand side sparse compound operand";

      initialize();

      result_ = min( db3_, ( dA3x4_ * sa4_ ) );
      sa4_    = min( db3_, ( dA3x4_ * sa4_ ) );

      checkResult( sa4_, result_ );
   }

   // Assignment to right-hand side subvector operand
   {
      test_ = "DVecDVecMin - Assignment to right-hand side subvector operand";

      initialize();

      result_ = min( db3_, subvector( da4_, 1UL, 3UL ) );
      da4_    = min( db3_, subvector( da4_, 1UL, 3UL ) );

      checkResult( da4_, result_ );
   }

   // Complex operation: a = min( ( 2*a ), ( A * b ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a = min( ( 2*a ), ( A * b ) );";

      initialize();

      result_ = min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );
      db3_    = min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a = min( ( A * b ), ( 2*a ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a = min( ( A * b ), ( 2*a ) );";

      initialize();

      result_ = min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );
      db3_    = min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a = min( b, ( a + A * c ) );
   {
      test_ = "DVecDVecMin - Complex operation: a = min( b, ( a + A * c ) );";

      initialize();

      result_ = min( db3_, ( dc3_ + dA3x4_ * da4_ ) );
      dc3_    = min( db3_, ( dc3_ + dA3x4_ * da4_ ) );

      checkResult( dc3_, result_ );
   }

   // Complex operation: a = min( ( A * b + a ), c );
   {
      test_ = "DVecDVecMin - Complex operation: a = min( ( A * b + a ), c );";

      initialize();

      result_ = min( ( dA3x4_ * da4_ + db3_ ), dc3_ );
      db3_    = min( ( dA3x4_ * da4_ + db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Minimum with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand (1)
   {
      test_ = "DVecDVecMin - Addition assignment to left-hand side operand (1)";

      initialize();

      result_ =  db3_;
      result_ += min( db3_, dc3_ );
      db3_    += min( db3_, dc3_ );

      checkResult( db3_, result_ );
   }

   // Addition assignment to left-hand side operand (2)
   {
      test_ = "DVecDVecMin - Addition assignment to left-hand side operand (2)";

      initialize();

      result_ =  db3_;
      result_ += min( db3_, eval( dc3_ ) );
      db3_    += min( db3_, eval( dc3_ ) );

      checkResult( db3_, result_ );
   }

   // Addition assignment to left-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Addition assignment to left-hand side dense compound operand";

      initialize();

      result_ =  db3_;
      result_ += min( ( dB3x3_ * db3_ ), dc3_ );
      db3_    += min( ( dB3x3_ * db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }

   // Addition assignment to left-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Addition assignment to left-hand side sparse compound operand";

      initialize();

      result_ =  sb3_;
      result_ += min( ( dB3x3_ * sb3_ ), dc3_ );
      sb3_    += min( ( dB3x3_ * sb3_ ), dc3_ );

      checkResult( sb3_, result_ );
   }

   // Addition assignment to right-hand side operand (1)
   {
      test_ = "DVecDVecMin - Addition assignment to right-hand side operand (1)";

      initialize();

      result_ =  dc3_;
      result_ += min( db3_, dc3_ );
      dc3_    += min( db3_, dc3_ );

      checkResult( dc3_, result_ );
   }

   // Addition assignment to right-hand side operand (2)
   {
      test_ = "DVecDVecMin - Addition assignment to right-hand side operand (2)";

      initialize();

      result_ =  dc3_;
      result_ += min( eval( db3_ ), dc3_ );
      dc3_    += min( eval( db3_ ), dc3_ );

      checkResult( dc3_, result_ );
   }

   // Addition assignment to right-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Addition assignment to right-hand side dense compound operand";

      initialize();

      result_ =  dc3_;
      result_ += min( db3_, ( dB3x3_ * dc3_ ) );
      dc3_    += min( db3_, ( dB3x3_ * dc3_ ) );

      checkResult( dc3_, result_ );
   }

   // Addition assignment to right-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Addition assignment to right-hand side dense compound operand";

      initialize();

      result_ =  sb3_;
      result_ += min( db3_, ( dB3x3_ * sb3_ ) );
      sb3_    += min( db3_, ( dB3x3_ * sb3_ ) );

      checkResult( sb3_, result_ );
   }

   // Complex operation: a += min( ( 2*a ), ( A * b ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a += min( ( 2*a ), ( A * b ) );";

      initialize();

      result_ =  db3_;
      result_ += min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );
      db3_    += min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a += min( ( A * b ), ( 2*a ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a += min( ( A * b ), ( 2*a ) );";

      initialize();

      result_ =  db3_;
      result_ += min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );
      db3_    += min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a += min( b, ( a + A * c ) );
   {
      test_ = "DVecDVecMin - Complex operation: a += min( b, ( a + A * c ) );";

      initialize();

      result_ =  dc3_;
      result_ += min( db3_, ( dc3_ + dA3x4_ * da4_ ) );
      dc3_    += min( db3_, ( dc3_ + dA3x4_ * da4_ ) );

      checkResult( dc3_, result_ );
   }

   // Complex operation: a += min( ( A * b + a ), c );
   {
      test_ = "DVecDVecMin - Complex operation: a += min( ( A * b + a ), c );";

      initialize();

      result_ =  db3_;
      result_ += min( ( dA3x4_ * da4_ + db3_ ), dc3_ );
      db3_    += min( ( dA3x4_ * da4_ + db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Minimum with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand (1)
   {
      test_ = "DVecDVecMin - Subtraction assignment to left-hand side operand (1)";

      initialize();

      result_ =  db3_;
      result_ -= min( db3_, dc3_ );
      db3_    -= min( db3_, dc3_ );

      checkResult( db3_, result_ );
   }

   // Subtraction assignment to left-hand side operand (2)
   {
      test_ = "DVecDVecMin - Subtraction assignment to left-hand side operand (2)";

      initialize();

      result_ =  db3_;
      result_ -= min( db3_, eval( dc3_ ) );
      db3_    -= min( db3_, eval( dc3_ ) );

      checkResult( db3_, result_ );
   }

   // Subtraction assignment to left-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Subtraction assignment to left-hand side dense compound operand";

      initialize();

      result_ =  db3_;
      result_ -= min( ( dB3x3_ * db3_ ), dc3_ );
      db3_    -= min( ( dB3x3_ * db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }

   // Subtraction assignment to left-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Subtraction assignment to left-hand side sparse compound operand";

      initialize();

      result_ =  sb3_;
      result_ -= min( ( dB3x3_ * sb3_ ), dc3_ );
      sb3_    -= min( ( dB3x3_ * sb3_ ), dc3_ );

      checkResult( sb3_, result_ );
   }

   // Subtraction assignment to right-hand side operand (1)
   {
      test_ = "DVecDVecMin - Subtraction assignment to right-hand side operand (1)";

      initialize();

      result_ =  dc3_;
      result_ -= min( db3_, dc3_ );
      dc3_    -= min( db3_, dc3_ );

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to right-hand side operand (2)
   {
      test_ = "DVecDVecMin - Subtraction assignment to right-hand side operand (2)";

      initialize();

      result_ =  dc3_;
      result_ -= min( eval( db3_ ), dc3_ );
      dc3_    -= min( eval( db3_ ), dc3_ );

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to right-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Subtraction assignment to right-hand side dense compound operand";

      initialize();

      result_ =  dc3_;
      result_ -= min( db3_, ( dB3x3_ * dc3_ ) );
      dc3_    -= min( db3_, ( dB3x3_ * dc3_ ) );

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to right-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Subtraction assignment to right-hand side sparse compound operand";

      initialize();

      result_ =  sb3_;
      result_ -= min( db3_, ( dB3x3_ * sb3_ ) );
      sb3_    -= min( db3_, ( dB3x3_ * sb3_ ) );

      checkResult( sb3_, result_ );
   }

   // Complex operation: a -= min( ( 2*a ), ( A * b ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a -= min( ( 2*a ), ( A * b ) );";

      initialize();

      result_ =  db3_;
      result_ -= min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );
      db3_    -= min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a -= min( ( A * b ), ( 2*a ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a -= min( ( A * b ), ( 2*a ) );";

      initialize();

      result_ =  db3_;
      result_ -= min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );
      db3_    -= min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a -= min( b, ( a + A * c ) );
   {
      test_ = "DVecDVecMin - Complex operation: a -= min( b, ( a + A * c ) );";

      initialize();

      result_ =  dc3_;
      result_ -= min( db3_, ( dc3_ + dA3x4_ * da4_ ) );
      dc3_    -= min( db3_, ( dc3_ + dA3x4_ * da4_ ) );

      checkResult( dc3_, result_ );
   }

   // Complex operation: a -= min( ( A * b + a ), c );
   {
      test_ = "DVecDVecMin - Complex operation: a -= min( ( A * b + a ), c );";

      initialize();

      result_ =  db3_;
      result_ -= min( ( dA3x4_ * da4_ + db3_ ), dc3_ );
      db3_    -= min( ( dA3x4_ * da4_ + db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Minimum with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand (1)
   {
      test_ = "DVecDVecMin - Multiplication assignment to left-hand side operand (1)";

      initialize();

      result_ =  db3_;
      result_ *= min( db3_, dc3_ );
      db3_    *= min( db3_, dc3_ );

      checkResult( db3_, result_ );
   }

   // Multiplication assignment to left-hand side operand (2)
   {
      test_ = "DVecDVecMin - Multiplication assignment to left-hand side operand (2)";

      initialize();

      result_ =  db3_;
      result_ *= min( db3_, eval( dc3_ ) );
      db3_    *= min( db3_, eval( dc3_ ) );

      checkResult( db3_, result_ );
   }

   // Multiplication assignment to left-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Multiplication assignment to left-hand side dense compound operand";

      initialize();

      result_ =  db3_;
      result_ *= min( ( dB3x3_ * db3_ ), dc3_ );
      db3_    *= min( ( dB3x3_ * db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }

   // Multiplication assignment to left-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Multiplication assignment to left-hand side sparse compound operand";

      initialize();

      result_ =  sb3_;
      result_ *= min( ( dB3x3_ * sb3_ ), dc3_ );
      sb3_    *= min( ( dB3x3_ * sb3_ ), dc3_ );

      checkResult( sb3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (1)
   {
      test_ = "DVecDVecMin - Multiplication assignment to right-hand side operand (1)";

      initialize();

      result_ =  dc3_;
      result_ *= min( db3_, dc3_ );
      dc3_    *= min( db3_, dc3_ );

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (2)
   {
      test_ = "DVecDVecMin - Multiplication assignment to right-hand side operand (2)";

      initialize();

      result_ =  dc3_;
      result_ *= min( eval( db3_ ), dc3_ );
      dc3_    *= min( eval( db3_ ), dc3_ );

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to right-hand side dense compound operand
   {
      test_ = "DVecDVecMin - Multiplication assignment to right-hand side dense compound operand";

      initialize();

      result_ =  dc3_;
      result_ *= min( db3_, ( dB3x3_ * dc3_ ) );
      dc3_    *= min( db3_, ( dB3x3_ * dc3_ ) );

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to right-hand side sparse compound operand
   {
      test_ = "DVecDVecMin - Multiplication assignment to right-hand side sparse compound operand";

      initialize();

      result_ =  sb3_;
      result_ *= min( db3_, ( dB3x3_ * sb3_ ) );
      sb3_    *= min( db3_, ( dB3x3_ * sb3_ ) );

      checkResult( sb3_, result_ );
   }

   // Complex operation: a *= min( ( 2*a ), ( A * b ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a *= min( ( 2*a ), ( A * b ) );";

      initialize();

      result_ =  db3_;
      result_ *= min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );
      db3_    *= min( ( 2*db3_ ), ( dA3x4_ * da4_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a *= min( ( A * b ), ( 2*a ) );
   {
      test_ = "DVecDVecAdd - Complex operation: a *= min( ( A * b ), ( 2*a ) );";

      initialize();

      result_ =  db3_;
      result_ *= min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );
      db3_    *= min( ( dA3x4_ * da4_ ), ( 2*db3_ ) );

      checkResult( db3_, result_ );
   }

   // Complex operation: a *= min( b, ( a + A * c ) );
   {
      test_ = "DVecDVecMin - Complex operation: a *= min( b, ( a + A * c ) );";

      initialize();

      result_ =  dc3_;
      result_ *= min( db3_, ( dc3_ + dA3x4_ * da4_ ) );
      dc3_    *= min( db3_, ( dc3_ + dA3x4_ * da4_ ) );

      checkResult( dc3_, result_ );
   }

   // Complex operation: a *= min( ( A * b + a ), c );
   {
      test_ = "DVecDVecMin - Complex operation: a *= min( ( A * b + a ), c );";

      initialize();

      result_ =  db3_;
      result_ *= min( ( dA3x4_ * da4_ + db3_ ), dc3_ );
      db3_    *= min( ( dA3x4_ * da4_ + db3_ ), dc3_ );

      checkResult( db3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense column vector
   da4_.resize( 4UL, false );
   da4_[0] = -1;
   da4_[1] =  0;
   da4_[2] = -3;
   da4_[3] =  2;

   // Initializing the second dense column vector
   db3_.resize( 3UL, false );
   db3_[0] = 1;
   db3_[1] = 2;
   db3_[2] = 3;

   // Initializing the third dense column vector
   dc3_.resize( 3UL, false );
   dc3_[0] = 0;
   dc3_[1] = 2;
   dc3_[2] = 1;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse column vector
   sa4_.resize( 4UL, false );
   sa4_.reset();
   sa4_[0] = -1;
   sa4_[2] = -3;
   sa4_[3] =  2;

   // Initializing the second sparse column vector
   sb3_.resize( 3UL, false );
   sb3_.reset();
   sb3_[0] = 1;
   sb3_[1] = 2;
   sb3_[2] = 3;


   //=====================================================================================
   // Initialization of the dense matrices
   //=====================================================================================

   // Initializing the first row-major dense matrix
   dA3x4_(0,0) = -1;
   dA3x4_(0,1) =  0;
   dA3x4_(0,2) = -2;
   dA3x4_(0,3) =  0;
   dA3x4_(1,0) =  0;
   dA3x4_(1,1) =  2;
   dA3x4_(1,2) = -3;
   dA3x4_(1,3) =  1;
   dA3x4_(2,0) =  0;
   dA3x4_(2,1) =  1;
   dA3x4_(2,2) =  2;
   dA3x4_(2,3) =  2;

   // Initializing the second row-major dense matrix
   dB3x3_(0,0) =  0;
   dB3x3_(0,1) = -1;
   dB3x3_(0,2) =  0;
   dB3x3_(1,0) =  1;
   dB3x3_(1,1) = -2;
   dB3x3_(1,2) =  2;
   dB3x3_(2,0) =  0;
   dB3x3_(2,1) =  0;
   dB3x3_(2,2) = -3;
}
//*************************************************************************************************

} // namespace dvecdvecmin

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_DVECDVECMIN_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
