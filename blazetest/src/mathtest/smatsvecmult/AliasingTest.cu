//=================================================================================================
/*!
//  \file src/mathtest/smatsvecmult/AliasingTest.cpp
//  \brief Source file for the sparse matrix/sparse vector multiplication aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/smatsvecmult/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace smatsvecmult {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : sA3x4_ ( 3UL, 4UL )
   , sB3x3_ ( 3UL, 3UL )
   , tsA3x4_( 3UL, 4UL )
   , tsB3x3_( 3UL, 3UL )
   , sa4_   ( 4UL )
   , sb4_   ( 4UL )
   , sc3_   ( 3UL )
   , sd3_   ( 3UL )
   , se3_   ( 3UL )
   , da4_   ( 4UL )
   , db3_   ( 3UL )
   , result_()
   , test_  ()
{
   testSMatSVecMult ();
   testTSMatSVecMult();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the sparse matrix/sparse vector multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the sparse matrix/sparse vector multiplication.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testSMatSVecMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "SMatSVecMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = sA3x4_ * sa4_;
      sa4_    = sA3x4_ * sa4_;

      checkResult( sa4_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( sc3_ * trans( sa4_ ) ) * sb4_;
      sc3_    = ( sc3_ * trans( sa4_ ) ) * sb4_;

      checkResult( sc3_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Assignment to second operand of left-hand side compound";

      initialize();

      result_ = ( sc3_ * trans( sa4_ ) ) * sb4_;
      sa4_    = ( sc3_ * trans( sa4_ ) ) * sb4_;

      checkResult( sa4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = sA3x4_ * ( sa4_ * da4_ );
      sa4_    = sA3x4_ * ( sa4_ * da4_ );

      checkResult( sa4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = sA3x4_ * ( da4_ + sa4_ );
      da4_    = sA3x4_ * ( da4_ + sa4_ );

      checkResult( da4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "SMatSVecMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ += sB3x3_ * sc3_;
      sc3_    += sB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ += ( sc3_ * trans( sd3_ ) ) * se3_;
      sc3_    += ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sc3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sd3_;
      result_ += ( sc3_ * trans( sd3_ ) ) * se3_;
      sd3_    += ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sd3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ += sB3x3_ * ( sc3_ * db3_ );
      sc3_    += sB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ += sB3x3_ * ( sc3_ * db3_ );
      db3_    += sB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "SMatSVecMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ -= sB3x3_ * sc3_;
      sc3_    -= sB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Subtraction assignment to first operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ -= ( sc3_ * trans( sd3_ ) ) * se3_;
      sc3_    -= ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sc3_, result_ );
   }

   // Subtraction assignment to second operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sd3_;
      result_ -= ( sc3_ * trans( sd3_ ) ) * se3_;
      sd3_    -= ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sd3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ -= sB3x3_ * ( sc3_ * db3_ );
      sc3_    -= sB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ -= sB3x3_ * ( sc3_ * db3_ );
      db3_    -= sB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "SMatSVecMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ *= sB3x3_ * sc3_;
      sc3_    *= sB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ *= ( sc3_ * trans( sd3_ ) ) * se3_;
      sc3_    *= ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sc3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "SMatSVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sd3_;
      result_ *= ( sc3_ * trans( sd3_ ) ) * se3_;
      sd3_    *= ( sc3_ * trans( sd3_ ) ) * se3_;

      checkResult( sd3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ *= sB3x3_ * ( sc3_ * db3_ );
      sc3_    *= sB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "SMatSVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ *= sB3x3_ * ( sc3_ * db3_ );
      db3_    *= sB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the transpose sparse matrix/sparse vector multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the transpose sparse matrix/sparse vector
// multiplication. In case an error is detected, a \a std::runtime_error exception is
// thrown.
*/
void AliasingTest::testTSMatSVecMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TSMatSVecMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tsA3x4_ * sa4_;
      sa4_    = tsA3x4_ * sa4_;

      checkResult( sa4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = tsA3x4_ * ( sa4_ * da4_ );
      sa4_    = tsA3x4_ * ( sa4_ * da4_ );

      checkResult( sa4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = tsA3x4_ * ( da4_ + sa4_ );
      da4_    = tsA3x4_ * ( da4_ + sa4_ );

      checkResult( da4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TSMatSVecMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ += tsB3x3_ * sc3_;
      sc3_    += tsB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ += tsB3x3_ * ( sc3_ * db3_ );
      sc3_    += tsB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ += tsB3x3_ * ( sc3_ * db3_ );
      db3_    += tsB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TSMatSVecMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ -= tsB3x3_ * sc3_;
      sc3_    -= tsB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ -= tsB3x3_ * ( sc3_ * db3_ );
      sc3_    -= tsB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ -= tsB3x3_ * ( sc3_ * db3_ );
      db3_    -= tsB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TSMatSVecMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  sc3_;
      result_ *= tsB3x3_ * sc3_;
      sc3_    *= tsB3x3_ * sc3_;

      checkResult( sc3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  sc3_;
      result_ *= tsB3x3_ * ( sc3_ * db3_ );
      sc3_    *= tsB3x3_ * ( sc3_ * db3_ );

      checkResult( sc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TSMatSVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  db3_;
      result_ *= tsB3x3_ * ( sc3_ * db3_ );
      db3_    *= tsB3x3_ * ( sc3_ * db3_ );

      checkResult( db3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the sparse matrices
   //=====================================================================================

   // Initializing the first row-major sparse matrix
   sA3x4_(0,0) = -1;
   sA3x4_(0,2) = -2;
   sA3x4_(1,1) =  2;
   sA3x4_(1,2) = -3;
   sA3x4_(1,3) =  1;
   sA3x4_(2,1) =  1;
   sA3x4_(2,2) =  2;
   sA3x4_(2,3) =  2;

   // Initializing the second row-major sparse matrix
   sB3x3_(0,0) = -1;
   sB3x3_(1,0) =  1;
   sB3x3_(1,1) = -2;
   sB3x3_(1,2) =  2;
   sB3x3_(2,2) = -3;

   // Initializing the first column-major sparse matrix
   tsA3x4_(0,0) = -1;
   tsA3x4_(0,2) = -2;
   tsA3x4_(1,1) =  2;
   tsA3x4_(1,2) = -3;
   tsA3x4_(1,3) =  1;
   tsA3x4_(2,1) =  1;
   tsA3x4_(2,2) =  2;
   tsA3x4_(2,3) =  2;

   // Initializing the second column-major sparse matrix
   tsB3x3_(0,0) = -1;
   tsB3x3_(1,0) =  1;
   tsB3x3_(1,1) = -2;
   tsB3x3_(1,2) =  2;
   tsB3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse column vector
   sa4_.resize( 4UL, false );
   sa4_.reset();
   sa4_[0] = -1;
   sa4_[2] = -3;
   sa4_[3] =  2;

   // Initializing the second sparse column vector
   sb4_.resize( 4UL, false );
   sb4_.reset();
   sb4_[1] =  1;
   sb4_[2] =  2;
   sb4_[3] = -1;

   // Initializing the third sparse column vector
   sc3_.resize( 3UL, false );
   sc3_.reset();
   sc3_[0] = 1;
   sc3_[1] = 2;
   sc3_[2] = 3;

   // Initializing the fourth sparse column vector
   sd3_.resize( 3UL, false );
   sd3_.reset();
   sd3_[1] = 2;
   sd3_[2] = 1;

   // Initializing the fifth sparse column vector
   se3_.resize( 3UL, false );
   se3_.reset();
   se3_[1] = 1;
   se3_[2] = 3;


   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense column vector
   da4_.resize( 4UL, false );
   da4_[0] = -1;
   da4_[1] =  0;
   da4_[2] = -3;
   da4_[3] =  2;

   // Initializing the second dense column vector
   db3_.resize( 3UL, false );
   db3_[0] = 1;
   db3_[1] = 2;
   db3_[2] = 3;
}
//*************************************************************************************************

} // namespace smatsvecmult

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_SMATSVECMULT_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
