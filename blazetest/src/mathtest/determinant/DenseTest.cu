//=================================================================================================
/*!
//  \file src/mathtest/determinant/DenseTest.cpp
//  \brief Source file for the dense matrix determinant test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/DiagonalMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/HermitianMatrix.h>
#include <blaze/math/LowerMatrix.h>
#include <blaze/math/SymmetricMatrix.h>
#include <blaze/math/UniLowerMatrix.h>
#include <blaze/math/UniUpperMatrix.h>
#include <blaze/math/UpperMatrix.h>
#include <blaze/util/Complex.h>
#include <blazetest/mathtest/determinant/DenseTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace determinant {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the DenseTest determinant test.
//
// \exception std::runtime_error Error during determinant computation error detected.
*/
DenseTest::DenseTest()
{
   using blaze::DynamicMatrix;
   using blaze::SymmetricMatrix;
   using blaze::HermitianMatrix;
   using blaze::LowerMatrix;
   using blaze::UniLowerMatrix;
   using blaze::UpperMatrix;
   using blaze::UniUpperMatrix;
   using blaze::DiagonalMatrix;
   using blaze::rowMajor;
   using blaze::columnMajor;

   using cfloat  = blaze::complex<float>;
   using cdouble = blaze::complex<double>;


   //=====================================================================================
   // Specific matrix tests
   //=====================================================================================

   testSpecific();


   //=====================================================================================
   // Random 2x2 matrix tests
   //=====================================================================================

   testRandom2x2< DynamicMatrix<float  ,rowMajor> >();
   testRandom2x2< DynamicMatrix<double ,rowMajor> >();
   testRandom2x2< DynamicMatrix<cfloat ,rowMajor> >();
   testRandom2x2< DynamicMatrix<cdouble,rowMajor> >();

   testRandom2x2< DynamicMatrix<float  ,columnMajor> >();
   testRandom2x2< DynamicMatrix<double ,columnMajor> >();
   testRandom2x2< DynamicMatrix<cfloat ,columnMajor> >();
   testRandom2x2< DynamicMatrix<cdouble,columnMajor> >();

   testRandom2x2< SymmetricMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom2x2< HermitianMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom2x2< LowerMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom2x2< UniLowerMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom2x2< UpperMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom2x2< UniUpperMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom2x2< DiagonalMatrix< DynamicMatrix<double,rowMajor>  > >();

   testRandom2x2< SymmetricMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom2x2< HermitianMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom2x2< LowerMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom2x2< UniLowerMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom2x2< UpperMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom2x2< UniUpperMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom2x2< DiagonalMatrix< DynamicMatrix<double,columnMajor>  > >();


   //=====================================================================================
   // Random 3x3 matrix tests
   //=====================================================================================

   testRandom3x3< DynamicMatrix<float  ,rowMajor> >();
   testRandom3x3< DynamicMatrix<double ,rowMajor> >();
   testRandom3x3< DynamicMatrix<cfloat ,rowMajor> >();
   testRandom3x3< DynamicMatrix<cdouble,rowMajor> >();

   testRandom3x3< DynamicMatrix<float  ,columnMajor> >();
   testRandom3x3< DynamicMatrix<double ,columnMajor> >();
   testRandom3x3< DynamicMatrix<cfloat ,columnMajor> >();
   testRandom3x3< DynamicMatrix<cdouble,columnMajor> >();

   testRandom3x3< SymmetricMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom3x3< HermitianMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom3x3< LowerMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom3x3< UniLowerMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom3x3< UpperMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom3x3< UniUpperMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom3x3< DiagonalMatrix< DynamicMatrix<double,rowMajor>  > >();

   testRandom3x3< SymmetricMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom3x3< HermitianMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom3x3< LowerMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom3x3< UniLowerMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom3x3< UpperMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom3x3< UniUpperMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom3x3< DiagonalMatrix< DynamicMatrix<double,columnMajor>  > >();


   //=====================================================================================
   // Random 4x4 matrix tests
   //=====================================================================================

   testRandom4x4< DynamicMatrix<float  ,rowMajor> >();
   testRandom4x4< DynamicMatrix<double ,rowMajor> >();
   testRandom4x4< DynamicMatrix<cfloat ,rowMajor> >();
   testRandom4x4< DynamicMatrix<cdouble,rowMajor> >();

   testRandom4x4< DynamicMatrix<float  ,columnMajor> >();
   testRandom4x4< DynamicMatrix<double ,columnMajor> >();
   testRandom4x4< DynamicMatrix<cfloat ,columnMajor> >();
   testRandom4x4< DynamicMatrix<cdouble,columnMajor> >();

   testRandom4x4< SymmetricMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom4x4< HermitianMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom4x4< LowerMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom4x4< UniLowerMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom4x4< UpperMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom4x4< UniUpperMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom4x4< DiagonalMatrix< DynamicMatrix<double,rowMajor>  > >();

   testRandom4x4< SymmetricMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom4x4< HermitianMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom4x4< LowerMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom4x4< UniLowerMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom4x4< UpperMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom4x4< UniUpperMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom4x4< DiagonalMatrix< DynamicMatrix<double,columnMajor>  > >();


   //=====================================================================================
   // Random 5x5 matrix tests
   //=====================================================================================

   testRandom5x5< DynamicMatrix<float  ,rowMajor> >();
   testRandom5x5< DynamicMatrix<double ,rowMajor> >();
   testRandom5x5< DynamicMatrix<cfloat ,rowMajor> >();
   testRandom5x5< DynamicMatrix<cdouble,rowMajor> >();

   testRandom5x5< DynamicMatrix<float  ,columnMajor> >();
   testRandom5x5< DynamicMatrix<double ,columnMajor> >();
   testRandom5x5< DynamicMatrix<cfloat ,columnMajor> >();
   testRandom5x5< DynamicMatrix<cdouble,columnMajor> >();

   testRandom5x5< SymmetricMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom5x5< HermitianMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom5x5< LowerMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom5x5< UniLowerMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom5x5< UpperMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom5x5< UniUpperMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom5x5< DiagonalMatrix< DynamicMatrix<double,rowMajor>  > >();

   testRandom5x5< SymmetricMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom5x5< HermitianMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom5x5< LowerMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom5x5< UniLowerMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom5x5< UpperMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom5x5< UniUpperMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom5x5< DiagonalMatrix< DynamicMatrix<double,columnMajor>  > >();


   //=====================================================================================
   // Random 6x6 matrix tests
   //=====================================================================================

   testRandom6x6< DynamicMatrix<float  ,rowMajor> >();
   testRandom6x6< DynamicMatrix<double ,rowMajor> >();
   testRandom6x6< DynamicMatrix<cfloat ,rowMajor> >();
   testRandom6x6< DynamicMatrix<cdouble,rowMajor> >();

   testRandom6x6< DynamicMatrix<float  ,columnMajor> >();
   testRandom6x6< DynamicMatrix<double ,columnMajor> >();
   testRandom6x6< DynamicMatrix<cfloat ,columnMajor> >();
   testRandom6x6< DynamicMatrix<cdouble,columnMajor> >();

   testRandom6x6< SymmetricMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom6x6< HermitianMatrix< DynamicMatrix<double,rowMajor> > >();
   testRandom6x6< LowerMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom6x6< UniLowerMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom6x6< UpperMatrix< DynamicMatrix<double,rowMajor>     > >();
   testRandom6x6< UniUpperMatrix< DynamicMatrix<double,rowMajor>  > >();
   testRandom6x6< DiagonalMatrix< DynamicMatrix<double,rowMajor>  > >();

   testRandom6x6< SymmetricMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom6x6< HermitianMatrix< DynamicMatrix<double,columnMajor> > >();
   testRandom6x6< LowerMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom6x6< UniLowerMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom6x6< UpperMatrix< DynamicMatrix<double,columnMajor>     > >();
   testRandom6x6< UniUpperMatrix< DynamicMatrix<double,columnMajor>  > >();
   testRandom6x6< DiagonalMatrix< DynamicMatrix<double,columnMajor>  > >();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the determinant functionality with specific, predetermined matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function computes determinants for specific, predetermined matrices. In case an error is
// detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSpecific()
{
#if BLAZETEST_MATHTEST_LAPACK_MODE

   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      {
         test_ = "Row-major det() function (0x0)";

         blaze::DynamicMatrix<double,blaze::rowMajor> A;

         const double determinant( det( A ) );

         if( determinant != 0.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Row-major det() function (1x1)";

         blaze::DynamicMatrix<double,blaze::rowMajor> A( 1UL, 1UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != A(0,0) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: " << A(0,0) << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Row-major det() function (unilower)";

         blaze::UniLowerMatrix< blaze::DynamicMatrix<double,blaze::rowMajor> > A( 9UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != 1.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Row-major det() function (uniupper)";

         blaze::UniUpperMatrix< blaze::DynamicMatrix<double,blaze::rowMajor> > A( 9UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != 1.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Row-major det() function (non-square)";

         blaze::DynamicMatrix<double,blaze::rowMajor> A( 2UL, 3UL );

         try {
            const double determinant( det( A ) );

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Computing the determinant for a non-square matrix succeeded\n"
                << " Details:\n"
                << "   Result:\n" << determinant << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      {
         test_ = "Column-major det() function (0x0)";

         blaze::DynamicMatrix<double,blaze::columnMajor> A;

         const double determinant( det( A ) );

         if( determinant != 0.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Column-major det() function (1x1)";

         blaze::DynamicMatrix<double,blaze::columnMajor> A( 1UL, 1UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != A(0,0) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: " << A(0,0) << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Column-major det() function (unilower)";

         blaze::UniLowerMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > A( 9UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != 1.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Column-major det() function (uniupper)";

         blaze::UniUpperMatrix< blaze::DynamicMatrix<double,blaze::columnMajor> > A( 9UL );
         randomize( A );

         const double determinant( det( A ) );

         if( determinant != 1.0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid determinant evaluation\n"
                << " Details:\n"
                << "   Result: " << determinant << "\n"
                << "   Expected result: 1\n";
            throw std::runtime_error( oss.str() );
         }
      }

      {
         test_ = "Column-major det() function (non-square)";

         blaze::DynamicMatrix<double,blaze::columnMajor> A( 2UL, 3UL );

         try {
            const double determinant( det( A ) );

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Computing the determinant for a non-square matrix succeeded\n"
                << " Details:\n"
                << "   Result:\n" << determinant << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }

#endif
}
//*************************************************************************************************

} // namespace determinant

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running dense matrix determinant test..." << std::endl;

   try
   {
      RUN_DETERMINANT_DENSE_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during dense matrix determinant test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
