//=================================================================================================
/*!
//  \file src/mathtest/tdvecsmatmult/AliasingTest.cpp
//  \brief Source file for the dense vector/sparse matrix multiplication aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/tdvecsmatmult/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace tdvecsmatmult {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : sA4x3_ ( 4UL, 3UL )
   , sB3x3_ ( 3UL, 3UL )
   , tsA4x3_( 4UL, 3UL )
   , tsB3x3_( 3UL, 3UL )
   , tda4_  ( 4UL )
   , tdb4_  ( 4UL )
   , tdc3_  ( 3UL )
   , tdd3_  ( 3UL )
   , tsa4_  ( 4UL )
   , tsb3_  ( 3UL )
   , result_()
   , test_  ()
{
   testTDVecSMatMult ();
   testTDVecTSMatMult();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the dense vector/sparse matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense vector/sparse matrix multiplication.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTDVecSMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TDVecSMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tda4_ * sA4x3_;
      tda4_   = tda4_ * sA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = tdb4_ * ( trans( tda4_ ) * tsb3_ );
      tda4_   = tdb4_ * ( trans( tda4_ ) * tsb3_ );

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Assignment to second operand of left-hand side compound";

      initialize();

      result_ = tdb4_ * ( trans( tda4_ ) * tsb3_ );
      tsb3_   = tdb4_ * ( trans( tda4_ ) * tsb3_ );

      checkResult( tsb3_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * sA4x3_;
      tda4_   = ( tda4_ + tsa4_ ) * sA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * sA4x3_;
      tsa4_   = ( tda4_ + tsa4_ ) * sA4x3_;

      checkResult( tsa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TDVecSMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ += tdc3_ * sB3x3_;
      tdc3_   += tdc3_ * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tdc3_   += tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ += tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tsb3_   += tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tsb3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += ( tdc3_ + tsb3_ ) * sB3x3_;
      tdc3_   += ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ += ( tdc3_ + tsb3_ ) * sB3x3_;
      tsb3_   += ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TDVecSMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ -= tdc3_ * sB3x3_;
      tdc3_   -= tdc3_ * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to first operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tdc3_   -= tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ -= tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tsb3_   -= tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tsb3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= ( tdc3_ + tsb3_ ) * sB3x3_;
      tdc3_   -= ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ -= ( tdc3_ + tsb3_ ) * sB3x3_;
      tsb3_   -= ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TDVecSMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ *= tdc3_ * sB3x3_;
      tdc3_   *= tdc3_ * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tdc3_   *= tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "TDVecSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ *= tdd3_ * ( trans( tdc3_ ) * tsb3_ );
      tsb3_   *= tdd3_ * ( trans( tdc3_ ) * tsb3_ );

      checkResult( tsb3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= ( tdc3_ + tsb3_ ) * sB3x3_;
      tdc3_   *= ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TDVecSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ *= ( tdc3_ + tsb3_ ) * sB3x3_;
      tsb3_   *= ( tdc3_ + tsb3_ ) * sB3x3_;

      checkResult( tsb3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the transpose dense vector/transpose dense matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the transpose dense vector/transpose dense matrix
// multiplication. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTDVecTSMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TDVecTSMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tda4_ * tsA4x3_;
      tda4_   = tda4_ * tsA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * tsA4x3_;
      tda4_   = ( tda4_ + tsa4_ ) * tsA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * tsA4x3_;
      tsa4_   = ( tda4_ + tsa4_ ) * tsA4x3_;

      checkResult( tsa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TDVecTSMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ += tdc3_ * tsB3x3_;
      tdc3_   += tdc3_ * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += ( tdc3_ + tsb3_ ) * tsB3x3_;
      tdc3_   += ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ += ( tdc3_ + tsb3_ ) * tsB3x3_;
      tsb3_   += ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TDVecTSMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ -= tdc3_ * tsB3x3_;
      tdc3_   -= tdc3_ * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= ( tdc3_ + tsb3_ ) * tsB3x3_;
      tdc3_   -= ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ -= ( tdc3_ + tsb3_ ) * tsB3x3_;
      tsb3_   -= ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TDVecTSMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ *= tdc3_ * tsB3x3_;
      tdc3_   *= tdc3_ * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= ( tdc3_ + tsb3_ ) * tsB3x3_;
      tdc3_   *= ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ *= ( tdc3_ + tsb3_ ) * tsB3x3_;
      tsb3_   *= ( tdc3_ + tsb3_ ) * tsB3x3_;

      checkResult( tsb3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the sparse matrices
   //=====================================================================================

   // Initializing the first row-major sparse matrix
   sA4x3_(0,0) = -1;
   sA4x3_(0,2) = -2;
   sA4x3_(1,1) =  2;
   sA4x3_(1,2) = -3;
   sA4x3_(2,1) =  1;
   sA4x3_(2,2) =  2;
   sA4x3_(3,0) =  1;
   sA4x3_(3,2) = -2;

   // Initializing the second row-major sparse matrix
   sB3x3_(0,1) = -1;
   sB3x3_(1,0) =  1;
   sB3x3_(1,1) = -2;
   sB3x3_(1,2) =  2;
   sB3x3_(2,2) = -3;

   // Initializing the first column-major sparse matrix
   tsA4x3_(0,0) = -1;
   tsA4x3_(0,2) = -2;
   tsA4x3_(1,1) =  2;
   tsA4x3_(1,2) = -3;
   tsA4x3_(2,1) =  1;
   tsA4x3_(2,2) =  2;
   tsA4x3_(3,0) =  1;
   tsA4x3_(3,2) = -2;

   // Initializing the second column-major sparse matrix
   tsB3x3_(0,1) = -1;
   tsB3x3_(1,0) =  1;
   tsB3x3_(1,1) = -2;
   tsB3x3_(1,2) =  2;
   tsB3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense row vector
   tda4_.resize( 4UL, false );
   tda4_[0] = -1;
   tda4_[1] =  0;
   tda4_[2] = -3;
   tda4_[3] =  2;

   // Initializing the second dense row vector
   tdb4_.resize( 4UL, false );
   tdb4_[0] =  0;
   tdb4_[1] =  1;
   tdb4_[2] =  2;
   tdb4_[3] = -1;

   // Initializing the third dense row vector
   tdc3_.resize( 3UL, false );
   tdc3_[0] = 1;
   tdc3_[1] = 2;
   tdc3_[2] = 3;

   // Initializing the fourth dense row vector
   tdd3_.resize( 3UL, false );
   tdd3_[0] = 0;
   tdd3_[1] = 2;
   tdd3_[2] = 1;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse row vector
   tsa4_.resize( 4UL, false );
   tsa4_.reset();
   tsa4_[0] = -1;
   tsa4_[2] = -3;
   tsa4_[3] =  2;

   // Initializing the second sparse row vector
   tsb3_.resize( 3UL, false );
   tsb3_.reset();
   tsb3_[1] = 2;
   tsb3_[2] = 1;
}
//*************************************************************************************************

} // namespace tdvecsmatmult

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_TDVECSMATMULT_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
