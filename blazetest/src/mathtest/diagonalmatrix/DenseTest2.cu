//=================================================================================================
/*!
//  \file src/mathtest/diagonalmatrix/DenseTest2.cpp
//  \brief Source file for the DiagonalMatrix dense test (part 2)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/Column.h>
#include <blaze/math/Row.h>
#include <blaze/math/Submatrix.h>
#include <blaze/util/Complex.h>
#include <blazetest/mathtest/diagonalmatrix/DenseTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace diagonalmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the DiagonalMatrix dense test.
//
// \exception std::runtime_error Operation error detected.
*/
DenseTest::DenseTest()
{
   testScaling();
   testFunctionCall();
   testIterator();
   testNonZeros();
   testReset();
   testClear();
   testResize();
   testExtend();
   testReserve();
   testShrinkToFit();
   testSwap();
   testIsDefault();
   testSubmatrix();
   testRow();
   testColumn();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of all DiagonalMatrix (self-)scaling operations.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all available ways to scale an instance of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testScaling()
{
   //=====================================================================================
   // Row-major self-scaling (M*=s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M*=s)";

      DT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag *= 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=M*s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=M*s)";

      DT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag = diag * 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=s*M)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=s*M)";

      DT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag = 2 * diag;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M/=s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M/=s)";

      DT diag( 3UL );
      diag(1,1) =  4;
      diag(2,2) = -6;

      diag /= 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=M/s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=M/s)";

      DT diag( 3UL );
      diag(1,1) =  4;
      diag(2,2) = -6;

      diag = diag / 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major DiagonalMatrix::scale()
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::scale()";

      // Initialization check
      DT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Integral scaling of the matrix
      diag.scale( 2 );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Floating point scaling of the matrix
      diag.scale( 0.5 );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major DiagonalMatrix::scale() (complex)";

      using blaze::complex;

      blaze::DiagonalMatrix< blaze::DynamicMatrix<complex<float>,blaze::rowMajor> > diag( 2UL );
      diag(0,0) = complex<float>( 1.0F, 0.0F );
      diag(1,1) = complex<float>( 2.0F, 0.0F );

      diag.scale( complex<float>( 3.0F, 0.0F ) );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );

      if( diag(0,0) != complex<float>( 3.0F, 0.0F ) || diag(0,1) != complex<float>( 0.0F, 0.0F ) ||
          diag(1,0) != complex<float>( 0.0F, 0.0F ) || diag(1,1) != complex<float>( 6.0F, 0.0F ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( (3,0) (0,0)\n(0,0) (6,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M*=s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M*=s)";

      ODT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag *= 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=M*s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=M*s)";

      ODT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag = diag * 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=s*M)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=s*M)";

      ODT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      diag = 2 * diag;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M/=s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M/=s)";

      ODT diag( 3UL );
      diag(1,1) =  4;
      diag(2,2) = -6;

      diag /= 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=M/s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=M/s)";

      ODT diag( 3UL );
      diag(1,1) =  4;
      diag(2,2) = -6;

      diag = diag / 2;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major DiagonalMatrix::scale()
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::scale()";

      // Initialization check
      ODT diag( 3UL );
      diag(1,1) =  2;
      diag(2,2) = -3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Integral scaling of the matrix
      diag.scale( 2 );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 4 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  4  0 )\n( 0  0 -6 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Floating point scaling of the matrix
      diag.scale( 0.5 );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) !=  0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) !=  0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0  0  0 )\n( 0  2  0 )\n( 0  0 -3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major DiagonalMatrix::scale() (complex)";

      using blaze::complex;

      blaze::DiagonalMatrix< blaze::DynamicMatrix<complex<float>,blaze::columnMajor> > diag( 2UL );
      diag(0,0) = complex<float>( 1.0F, 0.0F );
      diag(1,1) = complex<float>( 2.0F, 0.0F );

      diag.scale( complex<float>( 3.0F, 0.0F ) );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );

      if( diag(0,0) != complex<float>( 3.0F, 0.0F ) || diag(0,1) != complex<float>( 0.0F, 0.0F ) ||
          diag(1,0) != complex<float>( 0.0F, 0.0F ) || diag(1,1) != complex<float>( 6.0F, 0.0F ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( (3,0) (0,0)\n(0,0) (6,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the DiagonalMatrix function call operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the function call operator
// of the DiagonalMatrix specialization. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void DenseTest::testFunctionCall()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::operator()";

      // Good cases
      {
         DT diag( 3UL );

         // Writing the diagonal element (1,1)
         diag(1,1) = 1;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 1UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 1 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Writing the diagonal element (2,2)
         diag(2,2) = 2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 2UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 1 0 )\n( 0 0 2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Adding to the diagonal element (0,0)
         diag(0,0) += 3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3 0 0 )\n( 0 1 0 )\n( 0 0 2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Subtracting from the diagonal element (1,1)
         diag(1,1) -= 4;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0  2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Multiplying the diagonal element (2,2)
         diag(2,2) *= -3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != -6 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0 -6 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Dividing the diagonal element (2,2)
         diag(2,2) /= 2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != -3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0 -3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Failure cases
      {
         DT diag( 3UL );

         // Trying to write the lower element (2,1)
         try {
            diag(2,1) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the lower element (2,1)
         try {
            diag(2,1) += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the lower element (2,1)
         try {
            diag(2,1) -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the lower element (2,1)
         try {
            diag(2,1) *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the lower element (2,1)
         try {
            diag(2,1) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (1,2)
         try {
            diag(1,2) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the upper element (1,2)
         try {
            diag(1,2) += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the upper element (1,2)
         try {
            diag(1,2) -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the upper element (1,2)
         try {
            diag(1,2) *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the upper element (1,2)
         try {
            diag(1,2) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::operator()";

      // Good cases
      {
         ODT diag( 3UL );

         // Writing the diagonal element (1,1)
         diag(1,1) = 1;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 1UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 1 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Writing the diagonal element (2,2)
         diag(2,2) = 2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 2UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 1 0 )\n( 0 0 2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Adding to the diagonal element (0,0)
         diag(0,0) += 3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3 0 0 )\n( 0 1 0 )\n( 0 0 2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Subtracting from the diagonal element (1,1)
         diag(1,1) -= 4;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0  2 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Multiplying the diagonal element (2,2)
         diag(2,2) *= -3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != -6 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0 -6 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Dividing the diagonal element (2,2)
         diag(2,2) /= 2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != 3 || diag(0,1) !=  0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != -3 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != -3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 3  0  0 )\n( 0 -3  0 )\n( 0  0 -3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Failure cases
      {
         ODT diag( 3UL );

         // Trying to write the lower element (2,1)
         try {
            diag(2,1) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the lower element (2,1)
         try {
            diag(2,1) += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the lower element (2,1)
         try {
            diag(2,1) -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the lower element (2,1)
         try {
            diag(2,1) *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the lower element (2,1)
         try {
            diag(2,1) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (1,2)
         try {
            diag(1,2) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the upper element (1,2)
         try {
            diag(1,2) += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the upper element (1,2)
         try {
            diag(1,2) -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the upper element (1,2)
         try {
            diag(1,2) *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the upper element (1,2)
         try {
            diag(1,2) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the DiagonalMatrix iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testIterator()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      using Iterator      = DT::Iterator;
      using ConstIterator = DT::ConstIterator;

      DT diag( 3UL );
      diag(0,0) =  1;
      diag(1,1) = -2;
      diag(2,2) =  3;

      // Testing the Iterator default constructor
      {
         test_ = "Row-major Iterator default constructor";

         Iterator it{};

         if( it != Iterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing the ConstIterator default constructor
      {
         test_ = "Row-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing conversion from Iterator to ConstIterator
      {
         test_ = "Row-major Iterator/ConstIterator conversion";

         ConstIterator it( begin( diag, 1UL ) );

         if( it == end( diag, 1UL ) || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator conversion detected\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (end-begin)
      {
         test_ = "Row-major Iterator subtraction (end-begin)";

         const ptrdiff_t number( end( diag, 0UL ) - begin( diag, 0UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (begin-end)
      {
         test_ = "Row-major Iterator subtraction (begin-end)";

         const ptrdiff_t number( begin( diag, 0UL ) - end( diag, 0UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (end-begin)
      {
         test_ = "Row-major ConstIterator subtraction (end-begin)";

         const ptrdiff_t number( cend( diag, 1UL ) - cbegin( diag, 1UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (begin-end)
      {
         test_ = "Row-major ConstIterator subtraction (begin-end)";

         const ptrdiff_t number( cbegin( diag, 1UL ) - cend( diag, 1UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing read-only access via ConstIterator
      {
         test_ = "Row-major read-only access via ConstIterator";

         ConstIterator it ( cbegin( diag, 2UL ) );
         ConstIterator end( cend( diag, 2UL ) );

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid initial iterator detected\n";
            throw std::runtime_error( oss.str() );
         }

         ++it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         --it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it++;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it--;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it += 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator addition assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it -= 2UL;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator subtraction assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it + 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar addition failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it - 2UL;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar subtraction failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = 3UL + it;

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Scalar/iterator addition failed\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to diagonal elements via Iterator
      {
         test_ = "Row-major assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it = 4;

         if( diag(0,0) != 4 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4  0  0 )\n( 0 -2  0 )\n( 0  0  3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to lower elements via Iterator
      {
         test_ = "Row-major assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL );
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing assignment to upper elements via Iterator
      {
         test_ = "Row-major assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 1UL;
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to diagonal elements via Iterator
      {
         test_ = "Row-major addition assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 1UL ) + 1UL;
         *it += 3;

         if( diag(0,0) != 4 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4 0 0 )\n( 0 1 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing addition assignment to lower elements via Iterator
      {
         test_ = "Row-major addition assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL );
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to upper elements via Iterator
      {
         test_ = "Row-major addition assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 2UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to diagonal elements via Iterator
      {
         test_ = "Row-major subtraction assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 2UL ) + 2UL;
         *it -= 4;

         if( diag(0,0) != 4 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing subtraction assignment to lower elements via Iterator
      {
         test_ = "Row-major subtraction assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL ) + 1UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to upper elements via Iterator
      {
         test_ = "Row-major subtraction assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL ) + 2UL;
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to diagonal elements via Iterator
      {
         test_ = "Row-major multiplication assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it *= 2;

         if( diag(0,0) != 8 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 8  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing multiplication assignment to lower elements via Iterator
      {
         test_ = "Row-major multiplication assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL );
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to upper elements via Iterator
      {
         test_ = "Row-major multiplication assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 1UL;
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to diagonal elements via Iterator
      {
         test_ = "Row-major division assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it /= 4;

         if( diag(0,0) != 2 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 2  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing division assignment to lower elements via Iterator
      {
         test_ = "Row-major division assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL );
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to upper elements via Iterator
      {
         test_ = "Row-major division assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 2UL;
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      using Iterator      = ODT::Iterator;
      using ConstIterator = ODT::ConstIterator;

      ODT diag( 3UL );
      diag(0,0) =  1;
      diag(1,1) = -2;
      diag(2,2) =  3;

      // Testing the Iterator default constructor
      {
         test_ = "Column-major Iterator default constructor";

         Iterator it{};

         if( it != Iterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing the ConstIterator default constructor
      {
         test_ = "Column-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing conversion from Iterator to ConstIterator
      {
         test_ = "Column-major Iterator/ConstIterator conversion";

         ConstIterator it( begin( diag, 1UL ) );

         if( it == end( diag, 1UL ) || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator conversion detected\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (end-begin)
      {
         test_ = "Column-major Iterator subtraction (end-begin)";

         const ptrdiff_t number( end( diag, 0UL ) - begin( diag, 0UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (begin-end)
      {
         test_ = "Column-major Iterator subtraction (begin-end)";

         const ptrdiff_t number( begin( diag, 0UL ) - end( diag, 0UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (end-begin)
      {
         test_ = "Column-major ConstIterator subtraction (end-begin)";

         const ptrdiff_t number( cend( diag, 1UL ) - cbegin( diag, 1UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (begin-end)
      {
         test_ = "Column-major ConstIterator subtraction (begin-end)";

         const ptrdiff_t number( cbegin( diag, 1UL ) - cend( diag, 1UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing read-only access via ConstIterator
      {
         test_ = "Column-major read-only access via ConstIterator";

         ConstIterator it ( cbegin( diag, 2UL ) );
         ConstIterator end( cend( diag, 2UL ) );

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid initial iterator detected\n";
            throw std::runtime_error( oss.str() );
         }

         ++it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         --it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it++;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it--;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it += 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator addition assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it -= 2UL;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator subtraction assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it + 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar addition failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it - 2UL;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar subtraction failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = 3UL + it;

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Scalar/iterator addition failed\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to diagonal elements via Iterator
      {
         test_ = "Column-major assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it = 4;

         if( diag(0,0) != 4 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4  0  0 )\n( 0 -2  0 )\n( 0  0  3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to lower elements via Iterator
      {
         test_ = "Column-major assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL );
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing assignment to upper elements via Iterator
      {
         test_ = "Column-major assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 1UL;
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to diagonal elements via Iterator
      {
         test_ = "Column-major addition assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 1UL ) + 1UL;
         *it += 3;

         if( diag(0,0) != 4 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4 0 0 )\n( 0 1 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing addition assignment to lower elements via Iterator
      {
         test_ = "Column-major addition assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL );
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to upper elements via Iterator
      {
         test_ = "Column-major addition assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 2UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to diagonal elements via Iterator
      {
         test_ = "Column-major subtraction assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 2UL ) + 2UL;
         *it -= 4;

         if( diag(0,0) != 4 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 4  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing subtraction assignment to lower elements via Iterator
      {
         test_ = "Column-major subtraction assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL ) + 1UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to upper elements via Iterator
      {
         test_ = "Column-major subtraction assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL ) + 2UL;
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to diagonal elements via Iterator
      {
         test_ = "Column-major multiplication assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it *= 2;

         if( diag(0,0) != 8 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 8  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing multiplication assignment to lower elements via Iterator
      {
         test_ = "Column-major multiplication assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 1UL );
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to upper elements via Iterator
      {
         test_ = "Column-major multiplication assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 1UL;
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to diagonal elements via Iterator
      {
         test_ = "Column-major division assignment to diagonal elements via Iterator";

         const Iterator it = begin( diag, 0UL );
         *it /= 4;

         if( diag(0,0) != 2 || diag(0,1) != 0 || diag(0,2) !=  0 ||
             diag(1,0) != 0 || diag(1,1) != 1 || diag(1,2) !=  0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != -1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 2  0  0 )\n( 0  1  0 )\n( 0  0 -1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing division assignment to lower elements via Iterator
      {
         test_ = "Column-major division assignment to lower elements via Iterator";

         try {
            const Iterator it = begin( diag, 2UL );
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to lower matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to upper elements via Iterator
      {
         test_ = "Column-major division assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( diag, 0UL ) + 2UL;
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testNonZeros()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::nonZeros()";

      // Empty matrix
      {
         DT diag( 3UL );

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 0UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 0UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Partially filled matrix
      {
         DT diag( 3UL );
         diag(0,0) =  1;
         diag(1,1) = -2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 2UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0 -2  0 )\n( 0  0  0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Fully filled matrix
      {
         DT diag( 3UL );
         diag(0,0) = -1;
         diag(1,1) =  2;
         diag(2,2) =  3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != -1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) !=  0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) !=  0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( -1  0  0 )\n(  0  2  0 )\n(  0  0  3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::nonZeros()";

      // Empty matrix
      {
         ODT diag( 3UL );

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 0UL );
         checkNonZeros( diag, 0UL, 0UL );
         checkNonZeros( diag, 1UL, 0UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Partially filled matrix
      {
         ODT diag( 3UL );
         diag(0,0) =  1;
         diag(1,1) = -2;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 2UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 0UL );

         if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != -2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0 -2  0 )\n( 0  0  0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Fully filled matrix
      {
         ODT diag( 3UL );
         diag(0,0) = -1;
         diag(1,1) =  2;
         diag(2,2) =  3;

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag(0,0) != -1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) !=  0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) !=  0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( -1  0  0 )\n(  0  2  0 )\n(  0  0  3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reset() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reset() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testReset()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::reset()";

      // Initialization check
      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a diagonal element
      reset( diag(1,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a lower element
      reset( diag(1,0) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting an upper element
      reset( diag(0,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting row 2
      reset( diag, 2UL );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 1UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 0UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting the entire matrix
      reset( diag );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 0UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 0UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::reset()";

      // Initialization check
      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a diagonal element
      reset( diag(1,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a lower element
      reset( diag(1,0) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting an upper element
      reset( diag(0,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting row 2
      reset( diag, 2UL );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 1UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 0UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting the entire matrix
      reset( diag );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 0UL );
      checkNonZeros( diag, 0UL, 0UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 0UL );

      if( diag(0,0) != 0 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c clear() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c clear() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testClear()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::clear()";

      // Initialization check
      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( diag(1,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( diag(1,0) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( diag(0,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( diag );

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::clear()";

      // Initialization check
      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( diag(1,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( diag(1,0) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( diag(0,1) );

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkCapacity( diag, 9UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 0UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( diag );

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c resize() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c resize() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testResize()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::resize()";

      // Initialization check
      DT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Resizing to 2x2
      diag.resize( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );

      if( diag(0,1) != 0 || diag(1,0) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( x 0 )\n( 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 4x4 and preserving the elements
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag.resize( 4UL, true );

      checkRows    ( diag,  4UL );
      checkColumns ( diag,  4UL );
      checkCapacity( diag, 16UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 0 2 0 0 )\n( 0 0 x 0 )\n( 0 0 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 2x2
      diag(2,2) = 3;
      diag.resize( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 2 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 0x0
      diag.resize( 0UL );

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::resize()";

      // Initialization check
      ODT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Resizing to 2x2
      diag.resize( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );

      if( diag(0,1) != 0 || diag(1,0) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( x 0 )\n( 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 4x4 and preserving the elements
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag.resize( 4UL, true );

      checkRows    ( diag,  4UL );
      checkColumns ( diag,  4UL );
      checkCapacity( diag, 16UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 0 2 0 0 )\n( 0 0 x 0 )\n( 0 0 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 2x2
      diag(2,2) = 3;
      diag.resize( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );
      checkNonZeros( diag, 2UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 2 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 0x0
      diag.resize( 0UL );

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c extend() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c extend() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testExtend()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::extend()";

      // Initialization check
      DT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Extending the size of the matrix to 2x2
      diag.extend( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );

      if( diag(0,1) != 0 || diag(1,0) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( x 0 )\n( 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Extending to 4x4 and preserving the elements
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag.extend( 2UL, true );

      checkRows    ( diag,  4UL );
      checkColumns ( diag,  4UL );
      checkCapacity( diag, 16UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 0 2 0 0 )\n( 0 0 x 0 )\n( 0 0 0 x )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::extend()";

      // Initialization check
      ODT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Extending the size of the matrix to 2x2
      diag.extend( 2UL );

      checkRows    ( diag, 2UL );
      checkColumns ( diag, 2UL );
      checkCapacity( diag, 4UL );

      if( diag(0,1) != 0 || diag(1,0) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( x 0 )\n( 0 x )\n";
         throw std::runtime_error( oss.str() );
      }

      // Extending to 4x4 and preserving the elements
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag.extend( 2UL, true );

      checkRows    ( diag,  4UL );
      checkColumns ( diag,  4UL );
      checkCapacity( diag, 16UL );

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 0 2 0 0 )\n( 0 0 x 0 )\n( 0 0 0 x )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reserve() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reserve() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testReserve()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::reserve()";

      // Initialization check
      DT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Increasing the capacity of the matrix
      diag.reserve( 10UL );

      checkRows    ( diag,  0UL );
      checkColumns ( diag,  0UL );
      checkCapacity( diag, 10UL );
      checkNonZeros( diag,  0UL );

      // Further increasing the capacity of the matrix
      diag.reserve( 20UL );

      checkRows    ( diag,  0UL );
      checkColumns ( diag,  0UL );
      checkCapacity( diag, 20UL );
      checkNonZeros( diag,  0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::reserve()";

      // Initialization check
      ODT diag;

      checkRows    ( diag, 0UL );
      checkColumns ( diag, 0UL );
      checkNonZeros( diag, 0UL );

      // Increasing the capacity of the matrix
      diag.reserve( 10UL );

      checkRows    ( diag,  0UL );
      checkColumns ( diag,  0UL );
      checkCapacity( diag, 10UL );
      checkNonZeros( diag,  0UL );

      // Further increasing the capacity of the matrix
      diag.reserve( 20UL );

      checkRows    ( diag,  0UL );
      checkColumns ( diag,  0UL );
      checkCapacity( diag, 20UL );
      checkNonZeros( diag,  0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c shrinkToFit() member function of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c shrinkToFit() member function of the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testShrinkToFit()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix::shrinkToFit()";

      // Shrinking a matrix without excessive capacity
      {
         DT diag{ { 1, 0, 0 }, { 0, 2, 0 }, { 0, 0, 3 } };

         diag.shrinkToFit();

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag.capacity() != diag.rows() * diag.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << diag.capacity() << "\n"
                << "   Expected capacity: " << ( diag.rows() * diag.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Shrinking a matrix with excessive capacity
      {
         DT diag{ { 1, 0, 0 }, { 0, 2, 0 }, { 0, 0, 3 } };
         diag.reserve( 100UL );

         diag.shrinkToFit();

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag.capacity() != diag.rows() * diag.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << diag.capacity() << "\n"
                << "   Expected capacity: " << ( diag.rows() * diag.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix::shrinkToFit()";

      // Shrinking a matrix without excessive capacity
      {
         ODT diag{ { 1, 0, 0 }, { 0, 2, 0 }, { 0, 0, 3 } };

         diag.shrinkToFit();

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag.capacity() != diag.rows() * diag.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << diag.capacity() << "\n"
                << "   Expected capacity: " << ( diag.rows() * diag.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Shrinking a matrix with excessive capacity
      {
         ODT diag{ { 1, 0, 0 }, { 0, 2, 0 }, { 0, 0, 3 } };
         diag.reserve( 100UL );

         diag.shrinkToFit();

         checkRows    ( diag, 3UL );
         checkColumns ( diag, 3UL );
         checkCapacity( diag, 9UL );
         checkNonZeros( diag, 3UL );
         checkNonZeros( diag, 0UL, 1UL );
         checkNonZeros( diag, 1UL, 1UL );
         checkNonZeros( diag, 2UL, 1UL );

         if( diag.capacity() != diag.rows() * diag.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << diag.capacity() << "\n"
                << "   Expected capacity: " << ( diag.rows() * diag.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
             diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 ||
             diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << diag << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 2 0 )\n( 0 0 3 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the DiagonalMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSwap()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major DiagonalMatrix swap";

      DT diag1( 2UL );
      diag1(0,0) = 1;
      diag1(1,1) = 2;

      DT diag2( 3UL );
      diag2(0,0) = 3;
      diag2(1,1) = 4;
      diag2(2,2) = 5;

      swap( diag1, diag2 );

      checkRows    ( diag1, 3UL );
      checkColumns ( diag1, 3UL );
      checkCapacity( diag1, 9UL );
      checkNonZeros( diag1, 3UL );
      checkNonZeros( diag1, 0UL, 1UL );
      checkNonZeros( diag1, 1UL, 1UL );
      checkNonZeros( diag1, 2UL, 1UL );

      if( diag1(0,0) != 3 || diag1(0,1) != 0 || diag1(0,2) != 0 ||
          diag1(1,0) != 0 || diag1(1,1) != 4 || diag1(1,2) != 0 ||
          diag1(2,0) != 0 || diag1(2,1) != 0 || diag1(2,2) != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag1 << "\n"
             << "   Expected result:\n( 3 0 0 )\n( 0 4 0 )\n( 0 0 5 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( diag2, 2UL );
      checkColumns ( diag2, 2UL );
      checkCapacity( diag2, 4UL );
      checkNonZeros( diag2, 2UL );
      checkNonZeros( diag2, 0UL, 1UL );
      checkNonZeros( diag2, 1UL, 1UL );

      if( diag2(0,0) != 1 || diag2(0,1) != 0 || diag2(1,0) != 0 || diag2(1,1) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag2 << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major DiagonalMatrix swap";

      ODT diag1( 2UL );
      diag1(0,0) = 1;
      diag1(1,1) = 2;

      ODT diag2( 3UL );
      diag2(0,0) = 3;
      diag2(1,1) = 4;
      diag2(2,2) = 5;

      swap( diag1, diag2 );

      checkRows    ( diag1, 3UL );
      checkColumns ( diag1, 3UL );
      checkCapacity( diag1, 9UL );
      checkNonZeros( diag1, 3UL );
      checkNonZeros( diag1, 0UL, 1UL );
      checkNonZeros( diag1, 1UL, 1UL );
      checkNonZeros( diag1, 2UL, 1UL );

      if( diag1(0,0) != 3 || diag1(0,1) != 0 || diag1(0,2) != 0 ||
          diag1(1,0) != 0 || diag1(1,1) != 4 || diag1(1,2) != 0 ||
          diag1(2,0) != 0 || diag1(2,1) != 0 || diag1(2,2) != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag1 << "\n"
             << "   Expected result:\n( 3 0 0 )\n( 0 4 0 )\n( 0 0 5 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( diag2, 2UL );
      checkColumns ( diag2, 2UL );
      checkCapacity( diag2, 4UL );
      checkNonZeros( diag2, 2UL );
      checkNonZeros( diag2, 0UL, 1UL );
      checkNonZeros( diag2, 1UL, 1UL );

      if( diag2(0,0) != 1 || diag2(0,1) != 0 || diag2(1,0) != 0 || diag2(1,1) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag2 << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isDefault() function with the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isDefault() function with the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testIsDefault()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isDefault() function";

      // isDefault with 0x0 matrix
      {
         DT diag;

         if( isDefault( diag ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with default matrix
      {
         DT diag( 3UL );

         if( isDefault( diag(1,1) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << diag(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( diag ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         DT diag( 3UL );
         diag(1,1) = 1;

         if( isDefault( diag(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << diag(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( diag ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isDefault() function";

      // isDefault with 0x0 matrix
      {
         ODT diag;

         if( isDefault( diag ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with default matrix
      {
         ODT diag( 3UL );

         if( isDefault( diag(1,1) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << diag(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( diag ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         ODT diag( 3UL );
         diag(1,1) = 1;

         if( isDefault( diag(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << diag(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( diag ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << diag << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c submatrix() function with the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c submatrix() function with the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSubmatrix()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major submatrix() function";

      using SMT = blaze::Submatrix<DT>;

      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      SMT sm = submatrix( diag, 1UL, 1UL, 2UL, 2UL );

      if( sm(1,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << sm(1,1) << "\n"
             << "   Expected result: 3\n";
         throw std::runtime_error( oss.str() );
      }

      SMT::Iterator it = sm.begin(0UL);

      if( it == sm.end(0UL) || *it != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      sm(0,0) = -5;

      if( sm(0,0) != -5 || sm(0,1) != 0 ||
          sm(1,0) !=  0 || sm(1,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( -5  0 )\n(  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0 )\n( 0 -5  0 )\n( 0  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( sm );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0 12  0  0 )
   // ( 0  0  3  0 )      ( 0  0 12  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 1)";

      using SMT = blaze::Submatrix<DT>;

      DT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 0UL, 1UL, 4UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) !=  0 || sm(0,1) !=  0 ||
          sm(1,0) != 12 || sm(1,1) !=  0 ||
          sm(2,0) !=  0 || sm(2,1) != 12 ||
          sm(3,0) !=  0 || sm(3,1) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  0  0 )\n( 12  0 )\n(  0 12 )\n(  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) !=  0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 12 || diag(1,2) !=  0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 12 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) !=  0 || diag(3,2) !=  0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0 12  0  0 )\n"
                                     "( 0  0 12  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0 12  0  0 )
   // ( 0  0  3  0 )      ( 0  0 12  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 2)";

      using SMT = blaze::Submatrix<DT>;

      DT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 1UL, 0UL, 2UL, 4UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 12 || sm(0,2) !=  0 || sm(0,3) != 0 ||
          sm(1,0) != 0 || sm(1,1) !=  0 || sm(1,2) != 12 || sm(1,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 12  0  0 )\n( 0  0 12  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) !=  0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 12 || diag(1,2) !=  0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 12 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) !=  0 || diag(3,2) !=  0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0 12  0  0 )\n"
                                     "( 0  0 12  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0  2  0  0 )
   // ( 0  0  3  0 )      ( 0  0  3  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 3)";

      using SMT = blaze::Submatrix<DT>;

      DT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 0UL, 2UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0  2  0  0 )\n"
                                     "( 0  0  3  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0  2  0  0 )
   // ( 0  0  3  0 )      ( 0  0  3  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 4)";

      using SMT = blaze::Submatrix<DT>;

      DT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 2UL, 0UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0  2  0  0 )\n"
                                     "( 0  0  3  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major submatrix() function";

      using SMT = blaze::Submatrix<ODT>;

      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      SMT sm = submatrix( diag, 1UL, 1UL, 2UL, 2UL );

      if( sm(1,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << sm(1,1) << "\n"
             << "   Expected result: 3\n";
         throw std::runtime_error( oss.str() );
      }

      SMT::Iterator it = sm.begin(0UL);

      if( it == sm.end(0UL) || *it != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      sm(0,0) = -5;

      if( sm(0,0) != -5 || sm(0,1) != 0 ||
          sm(1,0) !=  0 || sm(1,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( -5  0 )\n(  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0 )\n( 0 -5  0 )\n( 0  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( sm );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0 12  0  0 )
   // ( 0  0  3  0 )      ( 0  0 12  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 1)";

      using SMT = blaze::Submatrix<ODT>;

      ODT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 0UL, 1UL, 4UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) !=  0 || sm(0,1) !=  0 ||
          sm(1,0) != 12 || sm(1,1) !=  0 ||
          sm(2,0) !=  0 || sm(2,1) != 12 ||
          sm(3,0) !=  0 || sm(3,1) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  0  0 )\n( 12  0 )\n(  0 12 )\n(  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) !=  0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 12 || diag(1,2) !=  0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 12 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) !=  0 || diag(3,2) !=  0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0 12  0  0 )\n"
                                     "( 0  0 12  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0 12  0  0 )
   // ( 0  0  3  0 )      ( 0  0 12  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 2)";

      using SMT = blaze::Submatrix<ODT>;

      ODT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 1UL, 0UL, 2UL, 4UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 12 || sm(0,2) !=  0 || sm(0,3) != 0 ||
          sm(1,0) != 0 || sm(1,1) !=  0 || sm(1,2) != 12 || sm(1,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 12  0  0 )\n( 0  0 12  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) !=  0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 12 || diag(1,2) !=  0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 12 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) !=  0 || diag(3,2) !=  0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0 12  0  0 )\n"
                                     "( 0  0 12  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0  2  0  0 )
   // ( 0  0  3  0 )      ( 0  0  3  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 3)";

      using SMT = blaze::Submatrix<ODT>;

      ODT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 0UL, 2UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0  2  0  0 )\n"
                                     "( 0  0  3  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // ( 1  0  0  0 )      ( 1  0  0  0 )
   // ( 0  2  0  0 )  =>  ( 0  2  0  0 )
   // ( 0  0  3  0 )      ( 0  0  3  0 )
   // ( 0  0  0  4 )      ( 0  0  0  4 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 4)";

      using SMT = blaze::Submatrix<ODT>;

      ODT diag( 4UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;
      diag(3,3) = 4;

      SMT sm = submatrix( diag, 2UL, 0UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( diag, 4UL );
      checkColumns ( diag, 4UL );
      checkNonZeros( diag, 4UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );
      checkNonZeros( diag, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 || diag(0,3) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 2 || diag(1,2) != 0 || diag(1,3) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 || diag(2,3) != 0 ||
          diag(3,0) != 0 || diag(3,1) != 0 || diag(3,2) != 0 || diag(3,3) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0  0 )\n"
                                     "( 0  2  0  0 )\n"
                                     "( 0  0  3  0 )\n"
                                     "( 0  0  0  4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c row() function with the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c row() function with the DiagonalMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testRow()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major row() function";

      using RT = blaze::Row<DT>;

      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      RT row1 = row( diag, 1UL );

      if( row1[1] != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << row1[1] << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      RT::Iterator it( row1.begin() );

      if( it == row1.end() || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 0\n";
         throw std::runtime_error( oss.str() );
      }

      row1[1] = -5;

      if( row1[0] != 0 || row1[1] != -5 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 -5  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n(  1  0  0 )\n( -4 -5  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( row1 );

      if( row1[0] != 0 || row1[1] != 0 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   {
      test_ = "Row-major row() function (scalar assignment test)";

      using RT = blaze::Row<DT>;

      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      RT row1 = row( diag, 1UL );
      row1 = 8;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( row1[0] != 0 || row1[1] != 8 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 8 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 8 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 8 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major row() function";

      using RT = blaze::Row<ODT>;

      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      RT row1 = row( diag, 1UL );

      if( row1[1] != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << row1[1] << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      RT::Iterator it( row1.begin() );

      if( it == row1.end() || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 0\n";
         throw std::runtime_error( oss.str() );
      }

      row1[1] = -5;

      if( row1[0] != 0 || row1[1] != -5 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( -4 -5  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0 )\n( 0 -5  0 )\n( 0  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( row1 );

      if( row1[0] != 0 || row1[1] != 0 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   {
      test_ = "Column-major row() function (scalar assignment test)";

      using RT = blaze::Row<ODT>;

      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      RT row1 = row( diag, 1UL );
      row1 = 8;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( row1[0] != 0 || row1[1] != 8 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 8 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 8 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 8 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c column() function with the DiagonalMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c column() function with the DiagonalMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testColumn()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major column() function";

      using CT = blaze::Column<DT>;

      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      CT col1 = column( diag, 1UL );

      if( col1[1] != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << col1[1] << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      CT::Iterator it( col1.begin() );

      if( it == col1.end() || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 0\n";
         throw std::runtime_error( oss.str() );
      }

      col1[1] = -5;

      if( col1[0] != 0 || col1[1] != -5 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 -5  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0 )\n( 0 -5  0 )\n( 0  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( col1 );

      if( col1[0] != 0 || col1[1] != 0 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   {
      test_ = "Row-major column() function (scalar assignment test)";

      using CT = blaze::Column<DT>;

      DT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      CT col1 = column( diag, 1UL );
      col1 = 8;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( col1[0] != 0 || col1[1] != 8 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 8 8 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 8 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 8 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major column() function";

      using CT = blaze::Column<ODT>;

      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      CT col1 = column( diag, 1UL );

      if( col1[1] != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << col1[1] << "\n"
             << "   Expected result: 2\n";
         throw std::runtime_error( oss.str() );
      }

      CT::Iterator it( col1.begin() );

      if( it == col1.end() || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 0\n";
         throw std::runtime_error( oss.str() );
      }

      col1[1] = -5;

      if( col1[0] != 0 || col1[1] != -5 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 -5  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) !=  0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != -5 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) !=  0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1  0  0 )\n( 0 -5  0 )\n( 0  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( col1 );

      if( col1[0] != 0 || col1[1] != 0 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 0 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 0 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   {
      test_ = "Column-major column() function (scalar assignment test)";

      using CT = blaze::Column<ODT>;

      ODT diag( 3UL );
      diag(0,0) = 1;
      diag(1,1) = 2;
      diag(2,2) = 3;

      CT col1 = column( diag, 1UL );
      col1 = 8;

      checkRows    ( diag, 3UL );
      checkColumns ( diag, 3UL );
      checkNonZeros( diag, 3UL );
      checkNonZeros( diag, 0UL, 1UL );
      checkNonZeros( diag, 1UL, 1UL );
      checkNonZeros( diag, 2UL, 1UL );

      if( col1[0] != 0 || col1[1] != 8 || col1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 8 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( diag(0,0) != 1 || diag(0,1) != 0 || diag(0,2) != 0 ||
          diag(1,0) != 0 || diag(1,1) != 8 || diag(1,2) != 0 ||
          diag(2,0) != 0 || diag(2,1) != 0 || diag(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << diag << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 8 0 )\n( 0 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace diagonalmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running DiagonalMatrix dense test (part 2)..." << std::endl;

   try
   {
      RUN_DIAGONALMATRIX_DENSE_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during DiagonalMatrix dense test (part 2):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
