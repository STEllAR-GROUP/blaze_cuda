//=================================================================================================
/*!
//  \file src/mathtest/tsvecsmatmult/AliasingTest.cpp
//  \brief Source file for the sparse vector/sparse matrix multiplication aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/tsvecsmatmult/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace tsvecsmatmult {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : sA4x3_ ( 4UL, 3UL )
   , sB3x3_ ( 3UL, 3UL )
   , tsA4x3_( 4UL, 3UL )
   , tsB3x3_( 3UL, 3UL )
   , tsa4_  ( 4UL )
   , tsb4_  ( 4UL )
   , tsc3_  ( 3UL )
   , tsd3_  ( 3UL )
   , tda4_  ( 4UL )
   , tdb3_  ( 3UL )
   , result_()
   , test_  ()
{
   testTSVecSMatMult ();
   testTSVecTSMatMult();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the sparse vector/sparse matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the sparse vector/sparse matrix multiplication.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTSVecSMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TSVecSMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tsa4_ * sA4x3_;
      tsa4_   = tsa4_ * sA4x3_;

      checkResult( tsa4_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = tsb4_ * ( trans( tsa4_ ) * tdb3_ );
      tsa4_   = tsb4_ * ( trans( tsa4_ ) * tdb3_ );

      checkResult( tsa4_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Assignment to second operand of left-hand side compound";

      initialize();

      result_ = tsb4_ * ( trans( tsa4_ ) * tdb3_ );
      tdb3_   = tsb4_ * ( trans( tsa4_ ) * tdb3_ );

      checkResult( tdb3_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tsa4_ * tda4_ ) * sA4x3_;
      tsa4_   = ( tsa4_ * tda4_ ) * sA4x3_;

      checkResult( tsa4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tsa4_ * tda4_ ) * sA4x3_;
      tda4_   = ( tsa4_ * tda4_ ) * sA4x3_;

      checkResult( tda4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TSVecSMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ += tsc3_ * sB3x3_;
      tsc3_   += tsc3_ * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ += tsd3_ * ( trans( tsc3_ ) * tdb3_ );
      tsc3_   += tsd3_ * ( trans( tsc3_ ) * tdb3_ );

      checkResult( tsc3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ += tsd3_ * ( trans( tdb3_ ) * tsc3_ );
      tdb3_   += tsd3_ * ( trans( tdb3_ ) * tsc3_ );

      checkResult( tdb3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ += ( tsc3_ * tdb3_ ) * sB3x3_;
      tsc3_   += ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ += ( tsc3_ * tdb3_ ) * sB3x3_;
      tdb3_   += ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tdb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TSVecSMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ -= tsc3_ * sB3x3_;
      tsc3_   -= tsc3_ * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Subtraction assignment to first operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ -= tsd3_ * ( trans( tsc3_ ) * tdb3_ );
      tsc3_   -= tsd3_ * ( trans( tsc3_ ) * tdb3_ );

      checkResult( tsc3_, result_ );
   }

   // Subtraction assignment to second operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ -= tsd3_ * ( trans( tdb3_ ) * tsc3_ );
      tdb3_   -= tsd3_ * ( trans( tdb3_ ) * tsc3_ );

      checkResult( tdb3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ -= ( tsc3_ * tdb3_ ) * sB3x3_;
      tsc3_   -= ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ -= ( tsc3_ * tdb3_ ) * sB3x3_;
      tdb3_   -= ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tdb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TSVecSMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ *= tsc3_ * sB3x3_;
      tsc3_   *= tsc3_ * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ *= tsd3_ * ( trans( tsc3_ ) * tdb3_ );
      tsc3_   *= tsd3_ * ( trans( tsc3_ ) * tdb3_ );

      checkResult( tsc3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "TSVecSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ *= tsd3_ * ( trans( tdb3_ ) * tsc3_ );
      tdb3_   *= tsd3_ * ( trans( tdb3_ ) * tsc3_ );

      checkResult( tdb3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ *= ( tsc3_ * tdb3_ ) * sB3x3_;
      tsc3_   *= ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TSVecSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ *= ( tsc3_ * tdb3_ ) * sB3x3_;
      tdb3_   *= ( tsc3_ * tdb3_ ) * sB3x3_;

      checkResult( tdb3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the transpose sparse vector/transpose sparse matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the transpose sparse vector/transpose sparse matrix
// multiplication. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTSVecTSMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TSVecTSMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tsa4_ * tsA4x3_;
      tsa4_   = tsa4_ * tsA4x3_;

      checkResult( tsa4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tsa4_ * tda4_ ) * tsA4x3_;
      tsa4_   = ( tsa4_ * tda4_ ) * tsA4x3_;

      checkResult( tsa4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tsa4_ * tda4_ ) * tsA4x3_;
      tda4_   = ( tsa4_ * tda4_ ) * tsA4x3_;

      checkResult( tda4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TSVecTSMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ += tsc3_ * tsB3x3_;
      tsc3_   += tsc3_ * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ += ( tsc3_ * tdb3_ ) * tsB3x3_;
      tsc3_   += ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ += ( tsc3_ * tdb3_ ) * tsB3x3_;
      tdb3_   += ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tdb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TSVecTSMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ -= tsc3_ * tsB3x3_;
      tsc3_   -= tsc3_ * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ -= ( tsc3_ * tdb3_ ) * tsB3x3_;
      tsc3_   -= ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ -= ( tsc3_ * tdb3_ ) * tsB3x3_;
      tdb3_   -= ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tdb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TSVecTSMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tsc3_;
      result_ *= tsc3_ * tsB3x3_;
      tsc3_   *= tsc3_ * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tsc3_;
      result_ *= ( tsc3_ * tdb3_ ) * tsB3x3_;
      tsc3_   *= ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tsc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TSVecTSMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdb3_;
      result_ *= ( tsc3_ * tdb3_ ) * tsB3x3_;
      tdb3_   *= ( tsc3_ * tdb3_ ) * tsB3x3_;

      checkResult( tdb3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the sparse matrices
   //=====================================================================================

   // Initializing the first row-major sparse matrix
   sA4x3_(0,0) = -1;
   sA4x3_(0,2) = -2;
   sA4x3_(1,1) =  2;
   sA4x3_(1,2) = -3;
   sA4x3_(2,1) =  1;
   sA4x3_(2,2) =  2;
   sA4x3_(3,0) =  1;
   sA4x3_(3,2) = -2;

   // Initializing the second row-major sparse matrix
   sB3x3_(0,0) = -1;
   sB3x3_(1,0) =  1;
   sB3x3_(1,1) = -2;
   sB3x3_(1,2) =  2;
   sB3x3_(2,2) = -3;

   // Initializing the first column-major sparse matrix
   tsA4x3_(0,0) = -1;
   tsA4x3_(0,2) = -2;
   tsA4x3_(1,1) =  2;
   tsA4x3_(1,2) = -3;
   tsA4x3_(2,1) =  1;
   tsA4x3_(2,2) =  2;
   tsA4x3_(3,0) =  1;
   tsA4x3_(3,2) = -2;

   // Initializing the second column-major sparse matrix
   tsB3x3_(0,0) = -1;
   tsB3x3_(1,0) =  1;
   tsB3x3_(1,1) = -2;
   tsB3x3_(1,2) =  2;
   tsB3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse row vector
   tsa4_.resize( 4UL, false );
   tsa4_.reset();
   tsa4_[0] = -1;
   tsa4_[2] = -3;
   tsa4_[3] =  2;

   // Initializing the second sparse row vector
   tsb4_.resize( 4UL, false );
   tsb4_.reset();
   tsb4_[1] =  1;
   tsb4_[2] =  2;
   tsb4_[3] = -1;

   // Initializing the third sparse row vector
   tsc3_.resize( 3UL, false );
   tsc3_.reset();
   tsc3_[0] = 1;
   tsc3_[1] = 2;
   tsc3_[2] = 3;

   // Initializing the fourth sparse row vector
   tsd3_.resize( 3UL, false );
   tsd3_.reset();
   tsd3_[1] = 2;
   tsd3_[2] = 1;


   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense row vector
   tda4_.resize( 4UL, false );
   tda4_[0] = -1;
   tda4_[1] =  0;
   tda4_[2] = -3;
   tda4_[3] =  2;

   // Initializing the second dense row vector
   tdb3_.resize( 3UL, false );
   tdb3_[0] = 0;
   tdb3_[1] = 2;
   tdb3_[2] = 1;
}
//*************************************************************************************************

} // namespace tsvecsmatmult

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_TSVECSMATMULT_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
