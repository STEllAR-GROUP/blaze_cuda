//=================================================================================================
/*!
//  \file src/mathtest/unilowermatrix/DenseTest2.cpp
//  \brief Source file for the UniLowerMatrix dense test (part 2)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/Column.h>
#include <blaze/math/Row.h>
#include <blaze/math/StaticMatrix.h>
#include <blaze/math/Submatrix.h>
#include <blazetest/mathtest/unilowermatrix/DenseTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace unilowermatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the UniLowerMatrix dense test.
//
// \exception std::runtime_error Operation error detected.
*/
DenseTest::DenseTest()
{
   testFunctionCall();
   testIterator();
   testNonZeros();
   testReset();
   testClear();
   testResize();
   testExtend();
   testReserve();
   testShrinkToFit();
   testSwap();
   testIsDefault();
   testSubmatrix();
   testRow();
   testColumn();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the UniLowerMatrix function call operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the function call operator
// of the UniLowerMatrix specialization. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void DenseTest::testFunctionCall()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::operator()";

      // Good cases
      {
         LT lower( 3UL );

         // Writing the lower element (2,1)
         lower(2,1) = 2;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 4UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 1UL );
         checkNonZeros( lower, 2UL, 2UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Writing the lower element (1,0)
         lower(1,0) = lower(2,1);

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 5UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 2UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Adding to the lower element (2,0)
         lower(2,0) += 3;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Subtracting from the lower element (1,0)
         lower(1,0) -= 4;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -2 1 0 )\n(  3 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Multiplying the lower element (2,1)
         lower(2,1) *= -3;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != -6 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n( -2  1  0 )\n(  3 -6  1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Dividing the lower element (2,1)
         lower(2,1) /= 2;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != -3 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n( -2  1  0 )\n(  3 -3  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Failure cases
      {
         LT lower( 3UL );

         // Trying to write the diagonal element (1,1)
         try {
            lower(1,1) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (1,2)
         try {
            lower(1,2) = 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the diagonal element (2,2)
         try {
            lower(2,2) = lower(1,1);

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (0,1)
         try {
            lower(0,1) = lower(2,1);

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the diagonal element (1,1)
         try {
            lower(1,1) += 6;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the upper element (0,2)
         try {
            lower(0,2) += 3;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the diagonal element (1,1)
         try {
            lower(1,1) -= 8;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the upper element (0,1)
         try {
            lower(0,1) -= 4;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the diagonal element (1,1)
         try {
            lower(1,1) *= -6;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the upper element (1,2)
         try {
            lower(1,2) *= -3;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the diagonal element (1,1)
         try {
            lower(1,1) /= 4;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the upper element (1,2)
         try {
            lower(1,2) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::operator()";

      // Good cases
      {
         OLT lower( 3UL );

         // Writing the lower element (2,1)
         lower(2,1) = 2;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 4UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Writing the lower element (1,0)
         lower(1,0) = lower(2,1);

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 5UL );
         checkNonZeros( lower, 0UL, 2UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Adding to the lower element (2,0)
         lower(2,0) += 3;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Subtracting from the lower element (1,0)
         lower(1,0) -= 4;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != 2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -2 1 0 )\n(  3 2 1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Multiplying the lower element (2,1)
         lower(2,1) *= -3;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != -6 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n( -2  1  0 )\n(  3 -6  1 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Dividing the lower element (2,1)
         lower(2,1) /= 2;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) !=  3 || lower(2,1) != -3 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Function call operator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n( -2  1  0 )\n(  3 -3  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Failure cases
      {
         OLT lower( 3UL );

         // Trying to write the diagonal element (1,1)
         try {
            lower(1,1) = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (1,2)
         try {
            lower(1,2) = 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the diagonal element (2,2)
         try {
            lower(2,2) = lower(1,1);

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to write the upper element (0,1)
         try {
            lower(0,1) = lower(2,1);

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the diagonal element (1,1)
         try {
            lower(1,1) += 6;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to add to the upper element (0,2)
         try {
            lower(0,2) += 3;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Addition assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the diagonal element (1,1)
         try {
            lower(1,1) -= 8;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to subtract from the upper element (0,1)
         try {
            lower(0,1) -= 4;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Subtraction assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the diagonal element (1,1)
         try {
            lower(1,1) *= -6;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to multiply the upper element (1,2)
         try {
            lower(1,2) *= -3;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Multiplication assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the diagonal element (1,1)
         try {
            lower(1,1) /= 4;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}

         // Trying to divide the upper element (1,2)
         try {
            lower(1,2) /= 2;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Division assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the UniLowerMatrix iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testIterator()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      using Iterator      = LT::Iterator;
      using ConstIterator = LT::ConstIterator;

      LT lower( 3UL );
      lower(2,0) = 3;

      // Testing the Iterator default constructor
      {
         test_ = "Row-major Iterator default constructor";

         Iterator it{};

         if( it != Iterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing the ConstIterator default constructor
      {
         test_ = "Row-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing conversion from Iterator to ConstIterator
      {
         test_ = "Row-major Iterator/ConstIterator conversion";

         ConstIterator it( begin( lower, 1UL ) );

         if( it == end( lower, 1UL ) || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator conversion detected\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (end-begin)
      {
         test_ = "Row-major Iterator subtraction (end-begin)";

         const ptrdiff_t number( end( lower, 0UL ) - begin( lower, 0UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th row via Iterator (begin-end)
      {
         test_ = "Row-major Iterator subtraction (begin-end)";

         const ptrdiff_t number( begin( lower, 0UL ) - end( lower, 0UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (end-begin)
      {
         test_ = "Row-major ConstIterator subtraction (end-begin)";

         const ptrdiff_t number( cend( lower, 1UL ) - cbegin( lower, 1UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (begin-end)
      {
         test_ = "Row-major ConstIterator subtraction (begin-end)";

         const ptrdiff_t number( cbegin( lower, 1UL ) - cend( lower, 1UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing read-only access via ConstIterator
      {
         test_ = "Row-major read-only access via ConstIterator";

         ConstIterator it ( cbegin( lower, 2UL ) );
         ConstIterator end( cend( lower, 2UL ) );

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid initial iterator detected\n";
            throw std::runtime_error( oss.str() );
         }

         ++it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         --it;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it++;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it--;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it += 2UL;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator addition assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it -= 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator subtraction assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it + 2UL;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar addition failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it - 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar subtraction failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = 3UL + it;

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Scalar/iterator addition failed\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to lower elements via Iterator
      {
         test_ = "Row-major assignment to lower elements via Iterator";

         Iterator it = begin( lower, 2UL );
         *it = 2;
         ++it;
         *it = -2;

         if( lower(0,0) != 1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != 2 || lower(2,1) != -2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0  1  0 )\n( 2 -2  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to diagonal elements via Iterator
      {
         test_ = "Row-major assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL );
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing assignment to upper elements via Iterator
      {
         test_ = "Row-major assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL ) + 1UL;
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to lower elements via Iterator
      {
         test_ = "Row-major addition assignment to lower elements via Iterator";

         Iterator it = begin( lower, 2UL );
         *it += 2;
         ++it;
         *it += -2;

         if( lower(0,0) != 1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != 4 || lower(2,1) != -4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0  1  0 )\n( 4 -4  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing addition assignment to diagonal elements via Iterator
      {
         test_ = "Row-major addition assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to upper elements via Iterator
      {
         test_ = "Row-major addition assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 2UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to lower elements via Iterator
      {
         test_ = "Row-major subtraction assignment to lower elements via Iterator";

         Iterator it = begin( lower, 2UL );
         *it -= 2;
         ++it;
         *it -= -2;

         if( lower(0,0) != 1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != 2 || lower(2,1) != -2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0  1  0 )\n( 2 -2  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing subtraction assignment to diagonal elements via Iterator
      {
         test_ = "Row-major subtraction assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL );
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to upper elements via Iterator
      {
         test_ = "Row-major subtraction assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL ) + 1UL;
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to lower elements via Iterator
      {
         test_ = "Row-major multiplication assignment to lower elements via Iterator";

         Iterator it = begin( lower, 2UL );
         *it *= 2;
         ++it;
         *it *= 2;

         if( lower(0,0) != 1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != 4 || lower(2,1) != -4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0  1  0 )\n( 4 -4  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing multiplication assignment to diagonal elements via Iterator
      {
         test_ = "Row-major multiplication assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to upper elements via Iterator
      {
         test_ = "Row-major multiplication assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 2UL;
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to lower elements via Iterator
      {
         test_ = "Row-major division assignment to lower elements via Iterator";

         Iterator it = begin( lower, 2UL );
         *it /= 2;
         ++it;
         *it /= 2;

         if( lower(0,0) != 1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != 2 || lower(2,1) != -2 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1  0  0 )\n( 0  1  0 )\n( 2 -2  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing division assignment to diagonal elements via Iterator
      {
         test_ = "Row-major division assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL );
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to upper elements via Iterator
      {
         test_ = "Row-major division assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 0UL ) + 1UL;
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      using Iterator      = OLT::Iterator;
      using ConstIterator = OLT::ConstIterator;

      OLT lower( 3UL );
      lower(2,0) = 3;

      // Testing the Iterator default constructor
      {
         test_ = "Column-major Iterator default constructor";

         Iterator it{};

         if( it != Iterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing the ConstIterator default constructor
      {
         test_ = "Column-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing conversion from Iterator to ConstIterator
      {
         test_ = "Column-major Iterator/ConstIterator conversion";

         ConstIterator it( begin( lower, 1UL ) );

         if( it == end( lower, 1UL ) || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator conversion detected\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th column via Iterator (end-begin)
      {
         test_ = "Column-major Iterator subtraction (end-begin)";

         const ptrdiff_t number( end( lower, 0UL ) - begin( lower, 0UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 0th column via Iterator (begin-end)
      {
         test_ = "Column-major Iterator subtraction (begin-end)";

         const ptrdiff_t number( begin( lower, 0UL ) - end( lower, 0UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st column via ConstIterator (end-begin)
      {
         test_ = "Column-major ConstIterator subtraction (end-begin)";

         const ptrdiff_t number( cend( lower, 1UL ) - cbegin( lower, 1UL ) );

         if( number != 3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st column via ConstIterator (begin-end)
      {
         test_ = "Column-major ConstIterator subtraction (begin-end)";

         const ptrdiff_t number( cbegin( lower, 1UL ) - cend( lower, 1UL ) );

         if( number != -3L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: -3\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing read-only access via ConstIterator
      {
         test_ = "Column-major read-only access via ConstIterator";

         ConstIterator it ( cbegin( lower, 0UL ) );
         ConstIterator end( cend( lower, 0UL ) );

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid initial iterator detected\n";
            throw std::runtime_error( oss.str() );
         }

         ++it;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         --it;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator pre-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it++;

         if( it == end || *it != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-increment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it--;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator post-decrement failed\n";
            throw std::runtime_error( oss.str() );
         }

         it += 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator addition assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it -= 2UL;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator subtraction assignment failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it + 2UL;

         if( it == end || *it != 3 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar addition failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = it - 2UL;

         if( it == end || *it != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator/scalar subtraction failed\n";
            throw std::runtime_error( oss.str() );
         }

         it = 3UL + it;

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Scalar/iterator addition failed\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to lower elements via Iterator
      {
         test_ = "Column-major assignment to lower elements via Iterator";

         Iterator it = begin( lower, 0UL );
         ++it;
         *it = -2;
         ++it;
         *it = 2;

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  2 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -2 1 0 )\n(  2 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing assignment to diagonal elements via Iterator
      {
         test_ = "Column-major assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing assignment to upper elements via Iterator
      {
         test_ = "Column-major assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL );
            *it = 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to lower elements via Iterator
      {
         test_ = "Column-major addition assignment to lower elements via Iterator";

         Iterator it = begin( lower, 0UL );
         ++it;
         *it += -2;
         ++it;
         *it += 2;

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  4 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  4 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing addition assignment to diagonal elements via Iterator
      {
         test_ = "Column-major addition assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing addition assignment to upper elements via Iterator
      {
         test_ = "Column-major addition assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL );
            *it += 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to lower elements via Iterator
      {
         test_ = "Column-major subtraction assignment to lower elements via Iterator";

         Iterator it = begin( lower, 0UL );
         ++it;
         *it -= -2;
         ++it;
         *it -= 2;

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  2 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -2 1 0 )\n(  2 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing subtraction assignment to diagonal elements via Iterator
      {
         test_ = "Column-major subtraction assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing subtraction assignment to upper elements via Iterator
      {
         test_ = "Column-major subtraction assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL );
            *it -= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to lower elements via Iterator
      {
         test_ = "Column-major multiplication assignment to lower elements via Iterator";

         Iterator it = begin( lower, 0UL );
         ++it;
         *it *= 2;
         ++it;
         *it *= 2;

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  4 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  4 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing multiplication assignment to diagonal elements via Iterator
      {
         test_ = "Column-major multiplication assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing multiplication assignment to upper elements via Iterator
      {
         test_ = "Column-major multiplication assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL );
            *it *= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to lower elements via Iterator
      {
         test_ = "Column-major division assignment to lower elements via Iterator";

         Iterator it = begin( lower, 0UL );
         ++it;
         *it /= 2;
         ++it;
         *it /= 2;

         if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != -2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) !=  2 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment via iterator failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1 0 0 )\n( -2 1 0 )\n(  2 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing division assignment to diagonal elements via Iterator
      {
         test_ = "Column-major division assignment to diagonal elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL ) + 1UL;
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to diagonal matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }

      // Testing division assignment to upper elements via Iterator
      {
         test_ = "Column-major division assignment to upper elements via Iterator";

         try {
            const Iterator it = begin( lower, 1UL );
            *it /= 5;

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment to upper matrix element succeeded\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testNonZeros()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::nonZeros()";

      // Default matrix
      {
         LT lower( 3UL );

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 3UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 1UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Fully filled matrix
      {
         LT lower( 3UL );
         lower(1,0) =  2;
         lower(2,0) = -4;
         lower(2,1) = -5;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) !=  2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != -4 || lower(2,1) != -5 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n(  2  1  0 )\n( -4 -5  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::nonZeros()";

      // Default matrix
      {
         OLT lower( 3UL );

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 3UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 1UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Fully filled matrix
      {
         OLT lower( 3UL );
         lower(1,0) =  2;
         lower(2,0) = -4;
         lower(2,1) = -5;

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
             lower(1,0) !=  2 || lower(1,1) !=  1 || lower(1,2) != 0 ||
             lower(2,0) != -4 || lower(2,1) != -5 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n(  1  0  0 )\n(  2  1  0 )\n( -4 -5  1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reset() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reset() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testReset()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::reset()";

      // Initialization check
      LT lower( 3UL );
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 3UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a lower element
      reset( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a diagonal element
      reset( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting an upper element
      reset( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting row 1
      reset( lower, 1UL );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 4UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting the entire matrix
      reset( lower );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::reset()";

      // Initialization check
      OLT lower( 3UL );
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 3UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a lower element
      reset( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a diagonal element
      reset( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting an upper element
      reset( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting column 1
      reset( lower, 1UL );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 4UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting the entire matrix
      reset( lower );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c clear() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c clear() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testClear()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::clear() (StaticMatrix)";

      // Initialization check
      blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > lower;
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 3UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( lower );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major UniLowerMatrix::clear() (DynamicMatrix)";

      // Initialization check
      LT lower( 3UL );
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 3UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( lower );

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::clear() (StaticMatrix)";

      // Initialization check
      blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > lower;
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 3UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( lower );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major UniLowerMatrix::clear() (DynamicMatrix)";

      // Initialization check
      OLT lower( 3UL );
      lower(1,0) = 2;
      lower(2,0) = 4;
      lower(2,1) = 5;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 3UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 4 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a lower element
      clear( lower(2,0) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing a diagonal element
      clear( lower(1,1) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing an upper element
      clear( lower(0,2) );

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkCapacity( lower, 9UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Clear operation failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 0 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Clearing the matrix
      clear( lower );

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c resize() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c resize() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testResize()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::resize()";

      // Initialization check
      LT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Resizing to 2x2
      lower.resize( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 4x4 and preserving the elements
      lower(1,0) = 2;
      lower.resize( 4UL, true );

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkCapacity( lower, 16UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
                                                lower(2,2) != 1 || lower(2,3) != 0 ||
                                                                   lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 2 1 0 0 )\n( x x 1 0 )\n( x x x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 2x2
      lower(2,1) = 4;
      lower.resize( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 0x0
      lower.resize( 0UL );

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::resize()";

      // Initialization check
      OLT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Resizing to 2x2
      lower.resize( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 4x4 and preserving the elements
      lower(1,0) = 2;
      lower.resize( 4UL, true );

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkCapacity( lower, 16UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
                                                lower(2,2) != 1 || lower(2,3) != 0 ||
                                                                   lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 2 1 0 0 )\n( x x 1 0 )\n( x x x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 2x2
      lower(2,1) = 4;
      lower.resize( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );
      checkNonZeros( lower, 3UL );
      checkNonZeros( lower, 0UL, 2UL );
      checkNonZeros( lower, 1UL, 1UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Resizing the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resizing to 0x0
      lower.resize( 0UL );

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c extend() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c extend() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testExtend()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::extend()";

      // Initialization check
      LT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Extending the size of the matrix to 2x2
      lower.extend( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Extending to 4x4 and preserving the elements
      lower(1,0) = 2;
      lower.extend( 2UL, true );

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkCapacity( lower, 16UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
                                                lower(2,2) != 1 || lower(2,3) != 0 ||
                                                                   lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 2 1 0 0 )\n( x x 1 0 )\n( x x x 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::extend()";

      // Initialization check
      OLT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Extending the size of the matrix to 2x2
      lower.extend( 2UL );

      checkRows    ( lower, 2UL );
      checkColumns ( lower, 2UL );
      checkCapacity( lower, 4UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 )\n( x 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Extending to 4x4 and preserving the elements
      lower(1,0) = 2;
      lower.extend( 2UL, true );

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkCapacity( lower, 16UL );

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
                                                lower(2,2) != 1 || lower(2,3) != 0 ||
                                                                   lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Extending the matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 0 )\n( 2 1 0 0 )\n( x x 1 0 )\n( x x x 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reserve() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reserve() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testReserve()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::reserve()";

      // Initialization check
      LT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Increasing the capacity of the matrix
      lower.reserve( 10UL );

      checkRows    ( lower,  0UL );
      checkColumns ( lower,  0UL );
      checkCapacity( lower, 10UL );
      checkNonZeros( lower,  0UL );

      // Further increasing the capacity of the matrix
      lower.reserve( 20UL );

      checkRows    ( lower,  0UL );
      checkColumns ( lower,  0UL );
      checkCapacity( lower, 20UL );
      checkNonZeros( lower,  0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::reserve()";

      // Initialization check
      OLT lower;

      checkRows    ( lower, 0UL );
      checkColumns ( lower, 0UL );
      checkNonZeros( lower, 0UL );

      // Increasing the capacity of the matrix
      lower.reserve( 10UL );

      checkRows    ( lower,  0UL );
      checkColumns ( lower,  0UL );
      checkCapacity( lower, 10UL );
      checkNonZeros( lower,  0UL );

      // Further increasing the capacity of the matrix
      lower.reserve( 20UL );

      checkRows    ( lower,  0UL );
      checkColumns ( lower,  0UL );
      checkCapacity( lower, 20UL );
      checkNonZeros( lower,  0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c shrinkToFit() member function of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c shrinkToFit() member function of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testShrinkToFit()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix::shrinkToFit()";

      // Shrinking a matrix without excessive capacity
      {
         LT lower{ { 1, 0, 0 }, { 2, 1, 0 }, { 3, 4, 1 } };

         lower.shrinkToFit();

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower.capacity() != lower.rows() * lower.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << lower.capacity() << "\n"
                << "   Expected capacity: " << ( lower.rows() * lower.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 4 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Shrinking a matrix with excessive capacity
      {
         LT lower{ { 1, 0, 0 }, { 2, 1, 0 }, { 3, 4, 1 } };
         lower.reserve( 100UL );

         lower.shrinkToFit();

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 1UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 3UL );

         if( lower.capacity() != lower.rows() * lower.spacing() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << lower.capacity() << "\n"
                << "   Expected capacity: " << ( lower.rows() * lower.spacing() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 4 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix::shrinkToFit()";

      // Shrinking a matrix without excessive capacity
      {
         OLT lower{ { 1, 0, 0 }, { 2, 1, 0 }, { 3, 4, 1 } };

         lower.shrinkToFit();

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower.capacity() != lower.spacing() * lower.columns() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << lower.capacity() << "\n"
                << "   Expected capacity: " << ( lower.spacing() * lower.columns() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 4 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Shrinking a matrix with excessive capacity
      {
         OLT lower{ { 1, 0, 0 }, { 2, 1, 0 }, { 3, 4, 1 } };
         lower.reserve( 100UL );

         lower.shrinkToFit();

         checkRows    ( lower, 3UL );
         checkColumns ( lower, 3UL );
         checkCapacity( lower, 9UL );
         checkNonZeros( lower, 6UL );
         checkNonZeros( lower, 0UL, 3UL );
         checkNonZeros( lower, 1UL, 2UL );
         checkNonZeros( lower, 2UL, 1UL );

         if( lower.capacity() != lower.spacing() * lower.columns() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Capacity         : " << lower.capacity() << "\n"
                << "   Expected capacity: " << ( lower.spacing() * lower.columns() ) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
             lower(1,0) != 2 || lower(1,1) != 1 || lower(1,2) != 0 ||
             lower(2,0) != 3 || lower(2,1) != 4 || lower(2,2) != 1 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Shrinking the matrix failed\n"
                << " Details:\n"
                << "   Result:\n" << lower << "\n"
                << "   Expected result:\n( 1 0 0 )\n( 2 1 0 )\n( 3 4 1 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the UniLowerMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSwap()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major UniLowerMatrix swap";

      LT lower1( 2UL );
      lower1(1,0) = 2;

      LT lower2( 3UL );
      lower2(1,0) = 3;
      lower2(2,0) = 4;
      lower2(2,1) = 5;

      swap( lower1, lower2 );

      checkRows    ( lower1, 3UL );
      checkColumns ( lower1, 3UL );
      checkCapacity( lower1, 9UL );
      checkNonZeros( lower1, 6UL );
      checkNonZeros( lower1, 0UL, 1UL );
      checkNonZeros( lower1, 1UL, 2UL );
      checkNonZeros( lower1, 2UL, 3UL );

      if( lower1(0,0) != 1 || lower1(0,1) != 0 || lower1(0,2) != 0 ||
          lower1(1,0) != 3 || lower1(1,1) != 1 || lower1(1,2) != 0 ||
          lower1(2,0) != 4 || lower1(2,1) != 5 || lower1(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower1 << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 3 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( lower2, 2UL );
      checkColumns ( lower2, 2UL );
      checkCapacity( lower2, 4UL );
      checkNonZeros( lower2, 3UL );
      checkNonZeros( lower2, 0UL, 1UL );
      checkNonZeros( lower2, 1UL, 2UL );

      if( lower2(0,0) != 1 || lower2(0,1) != 0 || lower2(1,0) != 2 || lower2(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower2 << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major UniLowerMatrix swap";

      OLT lower1( 2UL );
      lower1(1,0) = 2;

      OLT lower2( 3UL );
      lower2(1,0) = 3;
      lower2(2,0) = 4;
      lower2(2,1) = 5;

      swap( lower1, lower2 );

      checkRows    ( lower1, 3UL );
      checkColumns ( lower1, 3UL );
      checkCapacity( lower1, 9UL );
      checkNonZeros( lower1, 6UL );
      checkNonZeros( lower1, 0UL, 3UL );
      checkNonZeros( lower1, 1UL, 2UL );
      checkNonZeros( lower1, 2UL, 1UL );

      if( lower1(0,0) != 1 || lower1(0,1) != 0 || lower1(0,2) != 0 ||
          lower1(1,0) != 3 || lower1(1,1) != 1 || lower1(1,2) != 0 ||
          lower1(2,0) != 4 || lower1(2,1) != 5 || lower1(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower1 << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 3 1 0 )\n( 4 5 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( lower2, 2UL );
      checkColumns ( lower2, 2UL );
      checkCapacity( lower2, 4UL );
      checkNonZeros( lower2, 3UL );
      checkNonZeros( lower2, 0UL, 2UL );
      checkNonZeros( lower2, 1UL, 1UL );

      if( lower2(0,0) != 1 || lower2(0,1) != 0 || lower2(1,0) != 2 || lower2(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower2 << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isDefault() function with the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isDefault() function with the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testIsDefault()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isDefault() function (StaticMatrix)";

      // isDefault with default matrix
      {
         blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > lower;

         if( isDefault( lower(1,0) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > lower;
         lower(1,0) = 1;

         if( isDefault( lower(1,0) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major isDefault() function (DynamicMatrix)";

      // isDefault with 0x0 matrix
      {
         LT lower;

         if( isDefault( lower ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with default matrix
      {
         LT lower( 3UL );

         if( isDefault( lower(1,0) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         LT lower( 3UL );
         lower(1,0) = 1;

         if( isDefault( lower(1,0) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isDefault() function (StaticMatrix)";

      // isDefault with default matrix
      {
         blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > lower;

         if( isDefault( lower(1,0) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         blaze::UniLowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > lower;
         lower(1,0) = 1;

         if( isDefault( lower(1,0) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major isDefault() function (DynamicMatrix)";

      // isDefault with 0x0 matrix
      {
         OLT lower;

         if( isDefault( lower ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with default matrix
      {
         OLT lower( 3UL );

         if( isDefault( lower(1,0) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with non-default matrix
      {
         OLT lower( 3UL );
         lower(1,0) = 1;

         if( isDefault( lower(1,0) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower(1,1) ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element:\n" << lower(1,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( lower ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << lower << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the assignment to submatrices of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the assignment to submatrices of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testSubmatrix()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major submatrix() function";

      using SMT = blaze::Submatrix<LT>;

      LT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      SMT sm = submatrix( lower, 1UL, 1UL, 2UL, 2UL );

      if( sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << sm(1,1) << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      SMT::Iterator it = sm.begin(0UL);

      if( it == sm.end(0UL) || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      sm(1,0) = -5;

      if( sm(0,0) !=  1 || sm(0,1) != 0 ||
          sm(1,0) != -5 || sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  1  0 )\n( -5  1 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) !=  1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != -5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0 )\n( -4  1  0 )\n(  7 -5  1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( sm );

      if( sm(0,0) != 1 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  1  0  0 )  =>  ( -4  1  0  0 )
   // (  7  0  1  0 )      (  7 12  1  0 )
   // ( -2  0  1  1 )      ( -2 12 12  1 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 1)";

      using SMT = blaze::Submatrix<LT>;

      LT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 0UL, 1UL, 4UL, 2UL );
      sm = 12;

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkNonZeros( lower, 10UL );
      checkNonZeros( lower,  0UL, 1UL );
      checkNonZeros( lower,  1UL, 2UL );
      checkNonZeros( lower,  2UL, 3UL );
      checkNonZeros( lower,  3UL, 4UL );

      if( sm(0,0) !=  0 || sm(0,1) !=  0 ||
          sm(1,0) !=  1 || sm(1,1) !=  0 ||
          sm(2,0) != 12 || sm(2,1) !=  1 ||
          sm(3,0) != 12 || sm(3,1) != 12 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  0  0 )\n(  1  0 )\n( 12  1 )\n( 12 12 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) !=  0 || lower(0,3) != 0 ||
          lower(1,0) != -4 || lower(1,1) !=  1 || lower(1,2) !=  0 || lower(1,3) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 12 || lower(2,2) !=  1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) != 12 || lower(3,2) != 12 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( -4  1  0  0 )\n"
                                     "(  7 12  1  0 )\n"
                                     "( -2 12 12  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  1  0  0 )  =>  ( 12  1  0  0 )
   // (  7  0  1  0 )      ( 12 12  1  0 )
   // ( -2  0  1  1 )      ( -2  0  1  1 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 2)";

      using SMT = blaze::Submatrix<LT>;

      LT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 1UL, 0UL, 2UL, 4UL );
      sm = 12;

      checkRows    ( lower, 4UL );
      checkColumns ( lower, 4UL );
      checkNonZeros( lower, 9UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 3UL );
      checkNonZeros( lower, 3UL, 3UL );

      if( sm(0,0) != 12 || sm(0,1) !=  1 || sm(0,2) != 0 || sm(0,3) != 0 ||
          sm(1,0) != 12 || sm(1,1) != 12 || sm(1,2) != 1 || sm(1,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 12  1  0  0 )\n( 12 12  1  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) !=  0 || lower(0,3) != 0 ||
          lower(1,0) != 12 || lower(1,1) !=  1 || lower(1,2) !=  0 || lower(1,3) != 0 ||
          lower(2,0) != 12 || lower(2,1) != 12 || lower(2,2) !=  1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) !=  0 || lower(3,2) !=  1 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( 12  1  0  0 )\n"
                                     "( 12 12  1  0 )\n"
                                     "( -2  0  1  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  1  0  0 )  =>  ( -4  1  0  0 )
   // (  7  0  1  0 )      (  7  0  1  0 )
   // ( -2  0  1  1 )      ( -2  0  1  1 )
   {
      test_ = "Row-major submatrix() function (scalar assignment test 3)";

      using SMT = blaze::Submatrix<LT>;

      LT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 0UL, 2UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( lower, 4UL );
      checkColumns ( lower, 4UL );
      checkNonZeros( lower, 8UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );
      checkNonZeros( lower, 3UL, 3UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) != 0 || lower(3,2) != 1 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( -4  1  0  0 )\n"
                                     "(  7  0  1  0 )\n"
                                     "( -2  0  1  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major submatrix() function";

      using SMT = blaze::Submatrix<OLT>;

      OLT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      SMT sm = submatrix( lower, 1UL, 1UL, 2UL, 2UL );

      if( sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << sm(1,1) << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      SMT::Iterator it = sm.begin(0UL);

      if( it == sm.end(0UL) || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      sm(1,0) = -5;

      if( sm(0,0) !=  1 || sm(0,1) != 0 ||
          sm(1,0) != -5 || sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  1  0 )\n( -5  1 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) !=  1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != -5 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0 )\n( -4  1  0 )\n(  7 -5  1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( sm );

      if( sm(0,0) != 1 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 1 0 )\n( 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Submatrix reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  2  0  0 )  =>  ( -4  1  0  0 )
   // (  7  0  3  0 )      (  7 12  1  0 )
   // ( -2  0  1  5 )      ( -2 12 12  1 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 1)";

      using SMT = blaze::Submatrix<OLT>;

      OLT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 0UL, 1UL, 4UL, 2UL );
      sm = 12;

      checkRows    ( lower,  4UL );
      checkColumns ( lower,  4UL );
      checkNonZeros( lower, 10UL );
      checkNonZeros( lower,  0UL, 4UL );
      checkNonZeros( lower,  1UL, 3UL );
      checkNonZeros( lower,  2UL, 2UL );
      checkNonZeros( lower,  3UL, 1UL );

      if( sm(0,0) !=  0 || sm(0,1) !=  0 ||
          sm(1,0) !=  1 || sm(1,1) !=  0 ||
          sm(2,0) != 12 || sm(2,1) !=  1 ||
          sm(3,0) != 12 || sm(3,1) != 12 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n(  0  0 )\n(  1  0 )\n( 12  1 )\n( 12 12 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) !=  0 || lower(0,3) != 0 ||
          lower(1,0) != -4 || lower(1,1) !=  1 || lower(1,2) !=  0 || lower(1,3) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 12 || lower(2,2) !=  1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) != 12 || lower(3,2) != 12 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( -4  1  0  0 )\n"
                                     "(  7 12  1  0 )\n"
                                     "( -2 12 12  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  1  0  0 )  =>  ( 12  1  0  0 )
   // (  7  0  1  0 )      ( 12 12  1  0 )
   // ( -2  0  1  1 )      ( -2  0  1  1 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 2)";

      using SMT = blaze::Submatrix<OLT>;

      OLT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 1UL, 0UL, 2UL, 4UL );
      sm = 12;

      checkRows    ( lower, 4UL );
      checkColumns ( lower, 4UL );
      checkNonZeros( lower, 9UL );
      checkNonZeros( lower, 0UL, 4UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );
      checkNonZeros( lower, 3UL, 1UL );

      if( sm(0,0) != 12 || sm(0,1) !=  1 || sm(0,2) != 0 || sm(0,3) != 0 ||
          sm(1,0) != 12 || sm(1,1) != 12 || sm(1,2) != 1 || sm(1,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 12  1  0  0 )\n( 12 12  1  0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) !=  0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != 12 || lower(1,1) !=  1 || lower(1,2) != 0 || lower(1,3) != 0 ||
          lower(2,0) != 12 || lower(2,1) != 12 || lower(2,2) != 1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) !=  0 || lower(3,2) != 1 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( 12  1  0  0 )\n"
                                     "( 12 12  1  0 )\n"
                                     "( -2  0  1  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // (  1  0  0  0 )      (  1  0  0  0 )
   // ( -4  1  0  0 )  =>  ( -4  1  0  0 )
   // (  7  0  1  0 )      (  7  0  1  0 )
   // ( -2  0  1  1 )      ( -2  0  1  1 )
   {
      test_ = "Column-major submatrix() function (scalar assignment test 3)";

      using SMT = blaze::Submatrix<OLT>;

      OLT lower( 4UL );
      lower(1,0) = -4;
      lower(2,0) =  7;
      lower(3,0) = -2;
      lower(3,2) =  1;

      SMT sm = submatrix( lower, 0UL, 2UL, 2UL, 2UL );
      sm = 12;

      checkRows    ( lower, 4UL );
      checkColumns ( lower, 4UL );
      checkNonZeros( lower, 8UL );
      checkNonZeros( lower, 0UL, 4UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 2UL );
      checkNonZeros( lower, 3UL, 1UL );

      if( sm(0,0) != 0 || sm(0,1) != 0 ||
          sm(1,0) != 0 || sm(1,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << sm << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 || lower(0,3) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 || lower(1,3) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 || lower(2,3) != 0 ||
          lower(3,0) != -2 || lower(3,1) != 0 || lower(3,2) != 1 || lower(3,3) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment to submatrix failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1  0  0  0 )\n"
                                     "( -4  1  0  0 )\n"
                                     "(  7  0  1  0 )\n"
                                     "( -2  0  1  1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the assignment to rows of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the assignment to rows of the UniLowerMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testRow()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major row() function";

      using RT = blaze::Row<LT>;

      LT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      RT row1 = row( lower, 1UL );

      if( row1[0] != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << row1[0] << "\n"
             << "   Expected result: -4\n";
         throw std::runtime_error( oss.str() );
      }

      RT::Iterator it( row1.begin() );

      if( it == row1.end() || *it != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: -4\n";
         throw std::runtime_error( oss.str() );
      }

      row1[0] = -5;

      if( row1[0] != -5 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( -5 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -5 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -5 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( row1 );

      if( row1[0] != 0 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   {
      test_ = "Row-major row() function (scalar assignment test)";

      using RT = blaze::Row<LT>;

      LT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      RT row1 = row( lower, 1UL );
      row1 = 8;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 2UL );

      if( row1[0] != 8 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 8 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 8 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 8 1 0 )\n( 7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major row() function";

      using RT = blaze::Row<OLT>;

      OLT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      RT row1 = row( lower, 1UL );

      if( row1[0] != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << row1[0] << "\n"
             << "   Expected result: -4\n";
         throw std::runtime_error( oss.str() );
      }

      RT::Iterator it( row1.begin() );

      if( it == row1.end() || *it != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: -4\n";
         throw std::runtime_error( oss.str() );
      }

      row1[0] = -5;

      if( row1[0] != -5 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( -5 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -5 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -5 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( row1 );

      if( row1[0] != 0 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 0 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   {
      test_ = "Column-major row() function (scalar assignment test)";

      using RT = blaze::Row<OLT>;

      OLT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      RT row1 = row( lower, 1UL );
      row1 = 8;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkNonZeros( lower, 5UL );
      checkNonZeros( lower, 0UL, 3UL );
      checkNonZeros( lower, 1UL, 1UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( row1[0] != 8 || row1[1] != 1 || row1[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << row1 << "\n"
             << "   Expected result:\n( 8 1 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 8 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Row access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 8 1 0 )\n( 7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the assignment to columns of the UniLowerMatrix specialization.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the assignment to columns of the UniLowerMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void DenseTest::testColumn()
{
   //=====================================================================================
   // Row-major general tests
   //=====================================================================================

   {
      test_ = "Row-major column() function";

      using CT = blaze::Column<LT>;

      LT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      CT col0 = column( lower, 0UL );

      if( col0[0] != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << col0[0] << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      CT::Iterator it( col0.begin() );

      if( it == col0.end() || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      col0[1] = -5;

      if( col0[0] != 1 || col0[1] != -5 || col0[2] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col0 << "\n"
             << "   Expected result:\n( 1 -5  7 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -5 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -5 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( col0 );

      if( col0[0] != 1 || col0[1] != 0 || col0[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << col0 << "\n"
             << "   Expected result:\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major scalar assignment
   //=====================================================================================

   {
      test_ = "Row-major column() function (scalar assignment test)";

      using CT = blaze::Column<LT>;

      LT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      CT col1 = column( lower, 1UL );
      col1 = 8;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 1UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 3UL );

      if( col1[0] != 0 || col1[1] != 1 || col1[2] != 8 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 1 8 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 8 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  7 8 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major general tests
   //=====================================================================================

   {
      test_ = "Column-major column() function";

      using CT = blaze::Column<OLT>;

      OLT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      CT col0 = column( lower, 0UL );

      if( col0[0] != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator access failed\n"
             << " Details:\n"
             << "   Result: " << col0[0] << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      CT::Iterator it( col0.begin() );

      if( it == col0.end() || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator access failed\n"
             << " Details:\n"
             << "   Result: " << *it << "\n"
             << "   Expected result: 1\n";
         throw std::runtime_error( oss.str() );
      }

      col0[1] = -5;

      if( col0[0] != 1 || col0[1] != -5 || col0[2] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col0 << "\n"
             << "   Expected result:\n( 1 -5  7 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -5 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -5 1 0 )\n(  7 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }

      reset( col0 );

      if( col0[0] != 1 || col0[1] != 0 || col0[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << col0 << "\n"
             << "   Expected result:\n( 1 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) != 1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != 0 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) != 0 || lower(2,1) != 0 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column reset failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 0 1 0 )\n( 0 0 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major scalar assignment
   //=====================================================================================

   {
      test_ = "Column-major column() function (scalar assignment test)";

      using CT = blaze::Column<OLT>;

      OLT lower( 3UL );
      lower(1,0) = -4;
      lower(2,0) =  7;

      CT col1 = column( lower, 1UL );
      col1 = 8;

      checkRows    ( lower, 3UL );
      checkColumns ( lower, 3UL );
      checkNonZeros( lower, 6UL );
      checkNonZeros( lower, 0UL, 3UL );
      checkNonZeros( lower, 1UL, 2UL );
      checkNonZeros( lower, 2UL, 1UL );

      if( col1[0] != 0 || col1[1] != 1 || col1[2] != 8 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << col1 << "\n"
             << "   Expected result:\n( 0 1 8 )\n";
         throw std::runtime_error( oss.str() );
      }

      if( lower(0,0) !=  1 || lower(0,1) != 0 || lower(0,2) != 0 ||
          lower(1,0) != -4 || lower(1,1) != 1 || lower(1,2) != 0 ||
          lower(2,0) !=  7 || lower(2,1) != 8 || lower(2,2) != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Column access failed\n"
             << " Details:\n"
             << "   Result:\n" << lower << "\n"
             << "   Expected result:\n(  1 0 0 )\n( -4 1 0 )\n(  7 8 1 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace unilowermatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running UniLowerMatrix dense test (part 2)..." << std::endl;

   try
   {
      RUN_UNILOWERMATRIX_DENSE_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during UniLowerMatrix dense test (part 2):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
