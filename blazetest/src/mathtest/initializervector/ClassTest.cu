//=================================================================================================
/*!
//  \file src/mathtest/initializervector/ClassTest.cpp
//  \brief Source file for the InitializerVector class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/initializervector/ClassTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace initializervector {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the InitializerVector class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testConstructors();
   testSubscript();
   testAt();
   testIterator();
   testNonZeros();
   testSwap();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the InitializerVector constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the InitializerVector class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   using blaze::initializer_list;


   //=====================================================================================
   // Single argument constructor
   //=====================================================================================

   {
      test_ = "InitializerVector single argument constructor (size 0)";

      blaze::initializer_list<int> list = {};

      blaze::InitializerVector<int,blaze::rowVector> vec( list );

      checkSize    ( vec, 0UL );
      checkNonZeros( vec, 0UL );
   }

   {
      test_ = "InitializerVector single argument constructor (size 5)";

      auto list = { 1, 0, 3, 4, 0 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list );

      checkSize    ( vec, 5UL );
      checkNonZeros( vec, 3UL );
   }


   //=====================================================================================
   // Two argument constructor
   //=====================================================================================

   {
      test_ = "InitializerVector two argument constructor (size 0)";

      blaze::initializer_list<int> list = {};

      blaze::InitializerVector<int,blaze::rowVector> vec( list, 0UL );

      checkSize    ( vec, 0UL );
      checkNonZeros( vec, 0UL );
   }

   {
      test_ = "InitializerVector two argument constructor (size 5)";

      auto list = { 1, 0, 3, 4, 0 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list, 5UL );

      checkSize    ( vec, 5UL );
      checkNonZeros( vec, 3UL );
   }

   {
      test_ = "InitializerVector size constructor (size 7)";

      auto list = { 1, 0, 3, 4, 0 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list, 7UL );

      checkSize    ( vec, 7UL );
      checkCapacity( vec, 3UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the InitializerVector subscript operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the subscript operator of
// the InitializerVector class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testSubscript()
{
   test_ = "InitializerVector::operator[]";

   auto list = { 1, 0, 3, 4, 0 };

   blaze::InitializerVector<int,blaze::rowVector> vec( list, 7UL );

   // Access to the element at index 2
   if( vec[2] != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Access to the element at index 5
   if( vec[5] != 0 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c at() member function of the InitializerVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the \c at() member function
// of the InitializerVector class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testAt()
{
   test_ = "InitializerVector::at()";

   auto list = { 1, 0, 3, 4, 0 };

   blaze::InitializerVector<int,blaze::rowVector> vec( list, 7UL );

   // Access to the element at index 2
   if( vec.at(2) != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Access to the element at index 5
   if( vec.at(5) != 0 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Attempt to access the element at index 7
   try {
      vec.at(7);

      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Out-of-bound access succeeded\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 0 3 4 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }
   catch( std::out_of_range& ) {}
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the InitializerVector iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the InitializerVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testIterator()
{
   using VectorType    = blaze::InitializerVector<int>;
   using Iterator      = VectorType::Iterator;
   using ConstIterator = VectorType::ConstIterator;

   auto list = { 1, 0, -2, -3 };

   VectorType vec( list, 6UL );

   // Testing the Iterator default constructor
   {
      test_ = "Iterator default constructor";

      Iterator it{};

      if( it != Iterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing the ConstIterator default constructor
   {
      test_ = "ConstIterator default constructor";

      ConstIterator it{};

      if( it != ConstIterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing conversion from Iterator to ConstIterator
   {
      test_ = "Iterator/ConstIterator conversion";

      ConstIterator it( begin( vec ) );

      if( it == end( vec ) || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator conversion detected\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via Iterator (end-begin)
   {
      test_ = "Iterator subtraction (end-begin)";

      const ptrdiff_t number( end( vec ) - begin( vec ) );

      if( number != 6L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 6\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via Iterator (begin-end)
   {
      test_ = "Iterator subtraction (begin-end)";

      const ptrdiff_t number( begin( vec ) - end( vec ) );

      if( number != -6L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -6\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via ConstIterator (end-begin)
   {
      test_ = "ConstIterator subtraction (end-begin)";

      const ptrdiff_t number( cend( vec ) - cbegin( vec ) );

      if( number != 6L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 6\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via ConstIterator (begin-end)
   {
      test_ = "ConstIterator subtraction (begin-end)";

      const ptrdiff_t number( cbegin( vec ) - cend( vec ) );

      if( number != -6L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -6\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing read-only access via ConstIterator
   {
      test_ = "Read-only access via ConstIterator";

      ConstIterator it ( cbegin( vec ) );
      ConstIterator end( cend( vec ) );

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid initial iterator detected\n";
         throw std::runtime_error( oss.str() );
      }

      ++it;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      --it;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it++;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it--;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it += 2UL;

      if( it == end || *it != -2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator addition assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it -= 2UL;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator subtraction assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it + 3UL;

      if( it == end || *it != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar addition failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it - 3UL;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar subtraction failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = 6UL + it;

      if( it != end ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scalar/iterator addition failed\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the InitializerVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the InitializerVector
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testNonZeros()
{
   test_ = "InitializerVector::nonZeros()";

   {
      auto list = { 0, 0, 0, 0 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 0UL );

      if( vec[0] != 0 || vec[1] != 0 || vec[2] != 0 || vec[3] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      auto list = { 1, 2, 0, 3 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      auto list = { 1, 2, 0, 3 };

      blaze::InitializerVector<int,blaze::rowVector> vec( list, 6UL );

      checkSize    ( vec, 6UL );
      checkCapacity( vec, 6UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 3 || vec[4] != 0 || vec[5] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 0 3 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the InitializerVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the InitializerVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSwap()
{
   test_ = "InitializerVector swap";

   auto list1 = { 1, 2, 3 };
   auto list2 = { 4, 3, 2, 1 };

   blaze::InitializerVector<int,blaze::rowVector> vec1( list1 );
   blaze::InitializerVector<int,blaze::rowVector> vec2( list2, 5UL );

   swap( vec1, vec2 );

   checkSize    ( vec1, 5UL );
   checkCapacity( vec1, 5UL );
   checkNonZeros( vec1, 4UL );

   if( vec1[0] != 4 || vec1[1] != 3 || vec1[2] != 2 || vec1[3] != 1 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the first vector failed\n"
          << " Details:\n"
          << "   Result:\n" << vec1 << "\n"
          << "   Expected result:\n( 4 3 2 1 )\n";
      throw std::runtime_error( oss.str() );
   }

   checkSize    ( vec2, 3UL );
   checkCapacity( vec2, 3UL );
   checkNonZeros( vec2, 3UL );

   if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the second vector failed\n"
          << " Details:\n"
          << "   Result:\n" << vec1 << "\n"
          << "   Expected result:\n( 1 2 3 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************

} // namespace initializervector

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running InitializerVector class test..." << std::endl;

   try
   {
      RUN_INITIALIZERVECTOR_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during InitializerVector class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
