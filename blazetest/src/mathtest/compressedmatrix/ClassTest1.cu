#include "hip/hip_runtime.h"
//=================================================================================================
/*!
//  \file src/mathtest/compressedmatrix/ClassTest1.cpp
//  \brief Source file for the CompressedMatrix class test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/DiagonalMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/LowerMatrix.h>
#include <blaze/math/UpperMatrix.h>
#include <blaze/util/Complex.h>
#include <blaze/util/Random.h>
#include <blazetest/mathtest/compressedmatrix/ClassTest.h>
#include <blazetest/mathtest/RandomMaximum.h>
#include <blazetest/mathtest/RandomMinimum.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace compressedmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the CompressedMatrix class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
   testSchurAssign();
   testMultAssign();
   testScaling();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the CompressedMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the CompressedMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix default constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix size constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 0UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size constructor (0x4)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 4UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size constructor (3x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 0UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size constructor (3x4)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 4UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );
   }


   //=====================================================================================
   // Row-major size/non-zeros constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix size/non-zeros constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 0UL, 5UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size/non-zeros constructor (0x4)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 4UL, 5UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size/non-zeros constructor (3x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 0UL, 5UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size/non-zeros constructor (3x4)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 4UL, 5UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );
   }


   //=====================================================================================
   // Row-major size/vector constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix size/vector constructor (0x0)";

      std::vector<size_t> nonzeros;
      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 0UL, nonzeros );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size/vector constructor (0x5)";

      std::vector<size_t> nonzeros;
      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 0UL, 5UL, nonzeros );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 5UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix size/vector constructor (3x4)";

      std::vector<size_t> nonzeros( 3UL );
      nonzeros[0] = 2UL;
      nonzeros[1] = 1UL;
      nonzeros[2] = 3UL;
      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 4UL, nonzeros );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );

      mat.append( 0UL, 0UL, 1 );
      mat.append( 0UL, 1UL, 2 );
      mat.append( 1UL, 0UL, 3 );
      mat.append( 2UL, 0UL, 4 );
      mat.append( 2UL, 1UL, 5 );
      mat.append( 2UL, 2UL, 6 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 3UL );
   }


   //=====================================================================================
   // Row-major list initialization
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix initializer list constructor (complete list)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 0, 2, 0 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 2UL );

      if( mat(0,0) != 0 || mat(0,1) != 2 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 2 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major CompressedMatrix initializer list constructor (incomplete list)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 1 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix copy constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 0UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix copy constructor (0x3)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 0UL, 3UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix copy constructor (2x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix copy constructor (2x3)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix move constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 0UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix move constructor (0x3)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 0UL, 3UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix move constructor (2x0)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Row-major CompressedMatrix move constructor (2x3)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 1 },
                                                      { 0, 2 },
                                                      { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 1 },
                                                         { 0, 2 },
                                                         { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( trans( mat1 ) );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 ||
          mat2(2,0) != 1 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 )\n( 0 3 )\n( 1 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix default constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix size constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 0UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size constructor (0x4)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 4UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size constructor (3x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 0UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size constructor (3x4)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 4UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );
      checkNonZeros( mat, 3UL, 0UL );
   }


   //=====================================================================================
   // Column-major size/non-zeros constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix size/non-zeros constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 0UL, 5UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size/non-zeros constructor (0x4)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 4UL, 5UL );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size/non-zeros constructor (3x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 0UL, 5UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size/non-zeros constructor (3x4)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 4UL, 5UL );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 5UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );
      checkNonZeros( mat, 3UL, 0UL );
   }


   //=====================================================================================
   // Column-major size/vector constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix size/vector constructor (0x0)";

      std::vector<size_t> nonzeros;
      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 0UL, 0UL, nonzeros );

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size/vector constructor (5x0)";

      std::vector<size_t> nonzeros;
      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 5UL, 0UL, nonzeros );

      checkRows    ( mat, 5UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix size/vector constructor (4x3)";

      std::vector<size_t> nonzeros( 3UL );
      nonzeros[0] = 2UL;
      nonzeros[1] = 1UL;
      nonzeros[2] = 3UL;
      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 4UL, 3UL, nonzeros );

      checkRows    ( mat, 4UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 0UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 0UL );

      mat.append( 0UL, 0UL, 1 );
      mat.append( 1UL, 0UL, 2 );
      mat.append( 0UL, 1UL, 3 );
      mat.append( 0UL, 2UL, 4 );
      mat.append( 1UL, 2UL, 5 );
      mat.append( 2UL, 2UL, 6 );

      checkRows    ( mat, 4UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 3UL );
   }


   //=====================================================================================
   // Column-major list initialization
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix initializer list constructor (complete list)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ { 0, 2, 0 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 0 || mat(0,1) != 2 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 2 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major CompressedMatrix initializer list constructor (incomplete list)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ { 1 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix copy constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 0UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix copy constructor (0x3)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 0UL, 3UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix copy constructor (2x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix copy constructor (2x3)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix move constructor (0x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 0UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix move constructor (0x3)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 0UL, 3UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 0UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix move constructor (2x0)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 0UL, 3UL );
      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 0UL );
      checkNonZeros( mat2, 0UL );
   }

   {
      test_ = "Column-major CompressedMatrix move constructor (2x3)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( std::move( mat1 ) );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 1 },
                                                      { 0, 2 },
                                                      { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 1 },
                                                         { 0, 2 },
                                                         { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/Column-major CompressedMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( trans( mat1 ) );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 ||
          mat2(2,0) != 1 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 )\n( 0 3 )\n( 1 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CompressedMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the CompressedMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAssignment()
{
   //=====================================================================================
   // Row-major list assignment
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix initializer list assignment (complete list)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat;
      mat = { { 0, 2, 0 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 2UL );

      if( mat(0,0) != 0 || mat(0,1) != 2 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 2 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major CompressedMatrix initializer list assignment (incomplete list)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat;
      mat = { { 1 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix copy assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major CompressedMatrix copy assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::rowMajor>;

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix move assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ {}, { 11 }, {}, { 12 } };

      mat2 = std::move( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 1 },
                                                      { 0, 2 },
                                                      { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL );
      mat2 = mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::rowMajor>;

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 1 },
                                                         { 0, 2 },
                                                         { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL );
      mat2 = mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::columnMajor>;

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 2UL, 3UL );
      mat2 = trans( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 ||
          mat2(2,0) != 1 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 )\n( 0 3 )\n( 1 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<short,blaze::rowMajor>;

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1;
      const short min( randmin );
      const short max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 2UL );
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<short,blaze::columnMajor>;

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1;
      const short min( randmin );
      const short max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major list assignment
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix initializer list assignment (complete list)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat;
      mat = { { 0, 2, 0 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 0 || mat(0,1) != 2 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 2 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major CompressedMatrix initializer list assignment (incomplete list)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat;
      mat = { { 1 }, { 4, 0, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 0 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix copy assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major CompressedMatrix copy assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::columnMajor>;

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix move assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 1, 0, 2 },
                                                            { 0, 3, 0 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ {}, { 11 }, {}, { 12 } };

      mat2 = std::move( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 1 },
                                                      { 0, 2 },
                                                      { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL );
      mat2 = mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::rowMajor>;

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 1 },
                                                         { 0, 2 },
                                                         { 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL );
      mat2 = mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 1 ||
          mat2(1,0) != 0 || mat2(1,1) != 2 ||
          mat2(2,0) != 3 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 1 )\n( 0 2 )\n( 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::columnMajor>;

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 1, 0, 2 },
                                                         { 0, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 2UL );
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 2 )\n( 0 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<short,blaze::rowMajor>;

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1;
      const short min( randmin );
      const short max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/Column-major CompressedMatrix sparse matrix assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 2UL, 3UL );
      mat2 = trans( mat1 );

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 2UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 ||
          mat2(1,0) != 0 || mat2(1,1) != 3 ||
          mat2(2,0) != 1 || mat2(2,1) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 )\n( 0 3 )\n( 1 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<short,blaze::columnMajor>;

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1;
      const short min( randmin );
      const short max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t rows   ( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t columns( blaze::rand<size_t>( 0UL, 10UL ) );
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( rows, columns, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CompressedMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the CompressedMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAddAssign()
{
   //=====================================================================================
   // Row-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix addition assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix addition assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix addition assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix addition assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 5 || mat2(0,1) !=  0 || mat2(0,2) != 7 ||
          mat2(1,0) != 2 || mat2(1,1) != 10 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 5  0 7 )\n( 2 10 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CompressedMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the CompressedMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSubAssign()
{
   //=====================================================================================
   // Row-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix subtraction assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix subtraction assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix subtraction assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix subtraction assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 5UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) !=  5 || mat2(0,1) != 0 || mat2(0,2) !=  5 ||
          mat2(1,0) != -2 || mat2(1,1) != 4 || mat2(1,2) != -4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n(  5 0  5 )\n( -2 4 -4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( 3UL, 3UL );

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CompressedMatrix Schur product assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the Schur product assignment operators of the CompressedMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSchurAssign()
{
   //=====================================================================================
   // Row-major dense matrix Schur product assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix Schur product assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix Schur product assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix Schur product assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 2UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 5, 0, 6 },
                                                         { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 2UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix Schur product assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix Schur product assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                      { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix Schur product assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 3UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix dense matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::DynamicMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix Schur product assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix Schur product assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 0, 1 },
                                                         { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 2UL );
      checkNonZeros( mat2, 0UL, 0UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major CompressedMatrix sparse matrix Schur product assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 0, 1 },
                                                            { 2, 3, 4 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 5, 0, 6 },
                                                            { 0, 7, 0 } };

      mat2 %= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkNonZeros( mat2, 2UL );
      checkNonZeros( mat2, 0UL, 0UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 0 || mat2(0,1) !=  0 || mat2(0,2) != 6 ||
          mat2(1,0) != 0 || mat2(1,1) != 21 || mat2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0  0 6 )\n( 0 21 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix Schur product assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2( blaze::DynamicMatrix<int>( 3UL, 3UL, 1 ) );

      mat2 %= mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CompressedMatrix multiplication assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the multiplication assignment operators of the CompressedMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testMultAssign()
{
   //=====================================================================================
   // Row-major dense matrix multiplication assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix dense matrix multiplication assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 2, 0, 0 },
                                                      { 1, 3, 0, 4 },
                                                      { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 1, 0, 2 },
                                                         { 0, 3, 0 },
                                                         { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix multiplication assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 2, 0, 0 },
                                                         { 1, 3, 0, 4 },
                                                         { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 1, 0, 2 },
                                                         { 0, 3, 0 },
                                                         { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix multiplication assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major CompressedMatrix sparse matrix multiplication assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 2, 0, 0 },
                                                         { 1, 3, 0, 4 },
                                                         { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 1, 0, 2 },
                                                         { 0, 3, 0 },
                                                         { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix multiplication assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 2, 0, 0 },
                                                            { 1, 3, 0, 4 },
                                                            { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::rowMajor> mat2{ { 1, 0, 2 },
                                                         { 0, 3, 0 },
                                                         { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix multiplication assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix dense matrix multiplication assignment";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat1{ { 0, 2, 0, 0 },
                                                      { 1, 3, 0, 4 },
                                                      { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 1, 0, 2 },
                                                            { 0, 3, 0 },
                                                            { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 0UL );
      checkNonZeros( mat2, 3UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix dense matrix multiplication assignment";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat1{ { 0, 2, 0, 0 },
                                                         { 1, 3, 0, 4 },
                                                         { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 1, 0, 2 },
                                                            { 0, 3, 0 },
                                                            { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 0UL );
      checkNonZeros( mat2, 3UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix multiplication assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major CompressedMatrix sparse matrix multiplication assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1{ { 0, 2, 0, 0 },
                                                         { 1, 3, 0, 4 },
                                                         { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 1, 0, 2 },
                                                            { 0, 3, 0 },
                                                            { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 0UL );
      checkNonZeros( mat2, 3UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major CompressedMatrix sparse matrix multiplication assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1{ { 0, 2, 0, 0 },
                                                            { 1, 3, 0, 4 },
                                                            { 0, 0, 0, 5 } };

      blaze::CompressedMatrix<int,blaze::columnMajor> mat2{ { 1, 0, 2 },
                                                            { 0, 3, 0 },
                                                            { 4, 0, 5 } };

      mat2 *= mat1;

      checkRows    ( mat2, 3UL );
      checkColumns ( mat2, 4UL );
      checkNonZeros( mat2, 7UL );
      checkNonZeros( mat2, 0UL, 1UL );
      checkNonZeros( mat2, 1UL, 3UL );
      checkNonZeros( mat2, 2UL, 0UL );
      checkNonZeros( mat2, 3UL, 3UL );

      if( mat2(0,0) != 0 || mat2(0,1) != 2 || mat2(0,2) != 0 || mat2(0,3) != 10 ||
          mat2(1,0) != 3 || mat2(1,1) != 9 || mat2(1,2) != 0 || mat2(1,3) != 12 ||
          mat2(2,0) != 0 || mat2(2,1) != 8 || mat2(2,2) != 0 || mat2(2,3) != 25 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 0 2 0 10 )\n( 3 9 0 12 )\n( 0 8 0 25 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of all CompressedMatrix (self-)scaling operations.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all available ways to scale an instance of the CompressedMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testScaling()
{
   //=====================================================================================
   // Row-major self-scaling (M*=s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M*=s)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ {  0, 0, 0 },
                                                        {  0, 0, 1 },
                                                        { -2, 0, 3 } };

      mat *= 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=M*s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=M*s)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ {  0, 0, 0 },
                                                        {  0, 0, 1 },
                                                        { -2, 0, 3 } };

      mat = mat * 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=s*M)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=s*M)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ {  0, 0, 0 },
                                                        {  0, 0, 1 },
                                                        { -2, 0, 3 } };

      mat = 2 * mat;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M/=s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M/=s)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ {  0, 0, 0 },
                                                        {  0, 0, 2 },
                                                        { -4, 0, 6 } };

      mat /= 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 1 ||
          mat(2,0) != -2 || mat(2,1) != 0 || mat(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 1 )\n( -2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major self-scaling (M=M/s)
   //=====================================================================================

   {
      test_ = "Row-major self-scaling (M=M/s)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ {  0, 0, 0 },
                                                        {  0, 0, 2 },
                                                        { -4, 0, 6 } };

      mat = mat / 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 0UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 1 ||
          mat(2,0) != -2 || mat(2,1) != 0 || mat(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 1 )\n( -2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major CompressedMatrix::scale()
   //=====================================================================================

   {
      test_ = "Row-major CompressedMatrix::scale() (int)";

      // Initialization check
      blaze::CompressedMatrix<int,blaze::rowMajor> mat{ { 1, 0 },
                                                        { 2, 0 },
                                                        { 0, 3 } };

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(1,0) != 2 || mat(2,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 0 )\n( 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Integral scaling of the matrix
      mat.scale( 2 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 2 || mat(1,0) != 4 || mat(2,1) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 0 )\n( 4 0 )\n( 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Floating point scaling of the matrix
      mat.scale( 0.5 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(1,0) != 2 || mat(2,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 0 )\n( 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major CompressedMatrix::scale() (complex)";

      using blaze::complex;

      blaze::CompressedMatrix<complex<float>,blaze::rowMajor> mat( 2UL, 2UL );
      mat(0,0) = complex<float>( 1.0F, 0.0F );
      mat(1,1) = complex<float>( 2.0F, 0.0F );
      mat.scale( complex<float>( 3.0F, 0.0F ) );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 2UL );
      checkNonZeros( mat, 2UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != complex<float>( 3.0F, 0.0F ) || mat(1,1) != complex<float>( 6.0F, 0.0F ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( (3,0) (0,0)\n(0,0) (6,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M*=s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M*=s)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0, 0, 0 },
                                                           {  0, 0, 1 },
                                                           { -2, 0, 3 } };

      mat *= 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=M*s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=M*s)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0, 0, 0 },
                                                           {  0, 0, 1 },
                                                           { -2, 0, 3 } };

      mat = mat * 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=s*M)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=s*M)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0, 0, 0 },
                                                           {  0, 0, 1 },
                                                           { -2, 0, 3 } };

      mat = 2 * mat;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 2 ||
          mat(2,0) != -4 || mat(2,1) != 0 || mat(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 2 )\n( -4 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M/=s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M/=s)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0, 0, 0 },
                                                           {  0, 0, 2 },
                                                           { -4, 0, 6 } };

      mat /= 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 1 ||
          mat(2,0) != -2 || mat(2,1) != 0 || mat(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 1 )\n( -2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major self-scaling (M=M/s)
   //=====================================================================================

   {
      test_ = "Column-major self-scaling (M=M/s)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ {  0, 0, 0 },
                                                           {  0, 0, 2 },
                                                           { -4, 0, 6 } };

      mat = mat / 2;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 0UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) !=  0 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) !=  0 || mat(1,1) != 0 || mat(1,2) != 1 ||
          mat(2,0) != -2 || mat(2,1) != 0 || mat(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n(  0 0 0 )\n(  0 0 1 )\n( -2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major CompressedMatrix::scale()
   //=====================================================================================

   {
      test_ = "Column-major CompressedMatrix::scale() (int)";

      // Initialization check
      blaze::CompressedMatrix<int,blaze::columnMajor> mat{ { 1, 0 },
                                                           { 2, 0 },
                                                           { 0, 3 } };

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != 1 || mat(1,0) != 2 || mat(2,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 0 )\n( 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Integral scaling of the matrix
      mat.scale( 2 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != 2 || mat(1,0) != 4 || mat(2,1) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 0 )\n( 4 0 )\n( 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Floating point scaling of the matrix
      mat.scale( 0.5 );

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 3UL );
      checkNonZeros( mat, 3UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != 1 || mat(1,0) != 2 || mat(2,1) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 )\n( 2 0 )\n( 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major CompressedMatrix::scale() (complex)";

      using blaze::complex;

      blaze::CompressedMatrix<complex<float>,blaze::columnMajor> mat( 2UL, 2UL );
      mat(0,0) = complex<float>( 1.0F, 0.0F );
      mat(1,1) = complex<float>( 2.0F, 0.0F );
      mat.scale( complex<float>( 3.0F, 0.0F ) );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 2UL );
      checkCapacity( mat, 2UL );
      checkNonZeros( mat, 2UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 1UL );

      if( mat(0,0) != complex<float>( 3.0F, 0.0F ) || mat(1,1) != complex<float>( 6.0F, 0.0F ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( (3,0) (0,0)\n(0,0) (6,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace compressedmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running CompressedMatrix class test (part 1)..." << std::endl;

   try
   {
      RUN_COMPRESSEDMATRIX_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during CompressedMatrix class test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
