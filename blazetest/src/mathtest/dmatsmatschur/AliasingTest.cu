//=================================================================================================
/*!
//  \file src/mathtest/dmatsmatschur/AliasingTest.cpp
//  \brief Source file for the dense matrix/dense matrix Schur product aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/dmatsmatschur/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace dmatsmatschur {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : dA3x4_ ( 3UL, 4UL )
   , dB4x3_ ( 4UL, 3UL )
   , dC3x3_ ( 3UL, 3UL )
   , dD3x3_ ( 3UL, 3UL )
   , sA3x4_ ( 3UL, 4UL )
   , sB4x3_ ( 4UL, 3UL )
   , sC3x3_ ( 3UL, 3UL )
   , sD3x3_ ( 3UL, 3UL )
   , tsA3x4_( 3UL, 4UL )
   , tsB4x3_( 4UL, 3UL )
   , tsC3x3_( 3UL, 3UL )
   , tsD3x3_( 3UL, 3UL )
{
   testDMatSMatSchur ();
   testDMatTSMatSchur();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the dense matrix/dense matrix Schur product.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense matrix/dense matrix Schur product.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testDMatSMatSchur()
{
   //=====================================================================================
   // Schur product
   //=====================================================================================

   // Assignment to left-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Assignment to left-hand side operand (1)";

      initialize();

      result_ = dC3x3_ % sD3x3_;
      dC3x3_  = dC3x3_ % sD3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Assignment to left-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Assignment to left-hand side operand (2)";

      initialize();

      result_ = dC3x3_ % eval( sD3x3_ );
      dC3x3_  = dC3x3_ % eval( sD3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % sC3x3_;
      dA3x4_  = ( dA3x4_ * dB4x3_ ) % sC3x3_;

      checkResult( dA3x4_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % sC3x3_;
      dB4x3_  = ( dA3x4_ * dB4x3_ ) % sC3x3_;

      checkResult( dB4x3_, result_ );
   }

   // Assignment to right-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Assignment to right-hand side operand (1)";

      initialize();

      result_ = dC3x3_ % sD3x3_;
      sD3x3_  = dC3x3_ % sD3x3_;

      checkResult( sD3x3_, result_ );
   }

   // Assignment to right-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Assignment to right-hand side operand (2)";

      initialize();

      result_ = eval( dC3x3_ ) % sD3x3_;
      sD3x3_  = eval( dC3x3_ ) % sD3x3_;

      checkResult( sD3x3_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = dC3x3_ % ( sA3x4_ * sB4x3_ );
      sA3x4_  = dC3x3_ % ( sA3x4_ * sB4x3_ );

      checkResult( sA3x4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = dC3x3_ % ( sA3x4_ * sB4x3_ );
      sB4x3_  = dC3x3_ % ( sA3x4_ * sB4x3_ );

      checkResult( sB4x3_, result_ );
   }

   // Complex operation: A = ( 2*A ) % ( B * C )
   {
      test_ = "DMatSMatSchur - Complex operation: A = ( 2*A ) % ( B * C )";

      initialize();

      result_ = ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );
      dC3x3_  = ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A = ( B * C ) % ( 2*A )
   {
      test_ = "DMatSMatSchur - Complex operation: A = ( B * C ) % ( 2*A )";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );
      sD3x3_  = ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );

      checkResult( sD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Addition assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ += dC3x3_ % sC3x3_;
      dC3x3_  += dC3x3_ % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to left-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Addition assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ += dC3x3_ % eval( sC3x3_ );
      dC3x3_  += dC3x3_ % eval( sC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ += ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dC3x3_  += ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ += ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dD3x3_  += ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Addition assignment to right-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Addition assignment to right-hand side operand (1)";

      initialize();

      result_ =  sC3x3_;
      result_ += dC3x3_ % sC3x3_;
      sC3x3_  += dC3x3_ % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Addition assignment to right-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Addition assignment to right-hand side operand (2)";

      initialize();

      result_ =  sC3x3_;
      result_ += eval( dC3x3_ ) % sC3x3_;
      sC3x3_  += eval( dC3x3_ ) % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Addition assignment to first operand of right-hand side compound";

      initialize();

      result_ =  sC3x3_;
      result_ += dC3x3_ % ( sC3x3_ * sD3x3_ );
      sC3x3_  += dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sC3x3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Addition assignment to second operand of right-hand side compound";

      initialize();

      result_ =  sD3x3_;
      result_ += dC3x3_ % ( sC3x3_ * sD3x3_ );
      sD3x3_  += dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sD3x3_, result_ );
   }

   // Complex operation: A += ( 2*A ) % ( B * C )
   {
      test_ = "DMatSMatSchur - Complex operation: A += ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ += ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );
      dC3x3_  += ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A += ( B * C ) % ( 2*A )
   {
      test_ = "DMatSMatSchur - Complex operation: A += ( B * C ) % ( 2*A )";

      initialize();

      result_ =  sD3x3_;
      result_ += ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );
      sD3x3_  += ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );

      checkResult( sD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with subtraction assignment
   //=====================================================================================

   // Schur product assignment to left-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Subtraction assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ -= dC3x3_ % sC3x3_;
      dC3x3_  -= dC3x3_ % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to left-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Subtraction assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ -= dC3x3_ % eval( sC3x3_ );
      dC3x3_  -= dC3x3_ % eval( sC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to first operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ -= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dC3x3_  -= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to second operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ -= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dD3x3_  -= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Subtraction assignment to right-hand side operand (1)";

      initialize();

      result_ =  sC3x3_;
      result_ -= dC3x3_ % sC3x3_;
      sC3x3_  -= dC3x3_ % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Subtraction assignment to right-hand side operand (2)";

      initialize();

      result_ =  sC3x3_;
      result_ -= eval( dC3x3_ ) % sC3x3_;
      sC3x3_  -= eval( dC3x3_ ) % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to first operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Subtraction assignment to first operand of right-hand side compound";

      initialize();

      result_ =  sC3x3_;
      result_ -= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sC3x3_  -= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to second operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Subtraction assignment to second operand of right-hand side compound";

      initialize();

      result_ =  sD3x3_;
      result_ -= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sD3x3_  -= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sD3x3_, result_ );
   }

   // Complex operation: A -= ( 2*A ) % ( B * C )
   {
      test_ = "DMatSMatSchur - Complex operation: A -= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ -= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );
      dC3x3_  -= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A -= ( B * C ) % ( 2*A )
   {
      test_ = "DMatSMatSchur - Complex operation: A -= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  sD3x3_;
      result_ -= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );
      sD3x3_  -= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );

      checkResult( sD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with Schur product assignment
   //=====================================================================================

   // Schur product assignment to left-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Schur product assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ %= dC3x3_ % sC3x3_;
      dC3x3_  %= dC3x3_ % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to left-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Schur product assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ %= dC3x3_ % eval( sC3x3_ );
      dC3x3_  %= dC3x3_ % eval( sC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to first operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Schur product assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ %= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dC3x3_  %= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to second operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Schur product assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ %= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dD3x3_  %= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Schur product assignment to right-hand side operand (1)";

      initialize();

      result_ =  sC3x3_;
      result_ %= dC3x3_ % sC3x3_;
      sC3x3_  %= dC3x3_ % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Schur product assignment to right-hand side operand (2)";

      initialize();

      result_ =  sC3x3_;
      result_ %= eval( dC3x3_ ) % sC3x3_;
      sC3x3_  %= eval( dC3x3_ ) % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to first operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Schur product assignment to first operand of right-hand side compound";

      initialize();

      result_ =  sC3x3_;
      result_ %= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sC3x3_  %= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sC3x3_, result_ );
   }

   // Schur product assignment to second operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Schur product assignment to second operand of right-hand side compound";

      initialize();

      result_ =  sD3x3_;
      result_ %= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sD3x3_  %= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sD3x3_, result_ );
   }

   // Complex operation: A %= ( 2*A ) % ( B * C )
   {
      test_ = "DMatSMatSchur - Complex operation: A %= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ %= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );
      dC3x3_  %= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A %= ( B * C ) % ( 2*A )
   {
      test_ = "DMatSMatSchur - Complex operation: A %= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  sD3x3_;
      result_ %= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );
      sD3x3_  %= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );

      checkResult( sD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Multiplication assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ *= dC3x3_ % sC3x3_;
      dC3x3_  *= dC3x3_ % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to left-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Multiplication assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ *= dC3x3_ % eval( sC3x3_ );
      dC3x3_  *= dC3x3_ % eval( sC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ *= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dC3x3_  *= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "DMatSMatSchur - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ *= ( dC3x3_ * dD3x3_ ) % sC3x3_;
      dD3x3_  *= ( dC3x3_ * dD3x3_ ) % sC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (1)
   {
      test_ = "DMatSMatSchur - Multiplication assignment to right-hand side operand (1)";

      initialize();

      result_ =  sC3x3_;
      result_ *= dC3x3_ % sC3x3_;
      sC3x3_  *= dC3x3_ % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (2)
   {
      test_ = "DMatSMatSchur - Multiplication assignment to right-hand side operand (2)";

      initialize();

      result_ =  sC3x3_;
      result_ *= eval( dC3x3_ ) % sC3x3_;
      sC3x3_  *= eval( dC3x3_ ) % sC3x3_;

      checkResult( sC3x3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Multiplication assignment to first operand of right-hand side compound";

      initialize();

      result_ =  sC3x3_;
      result_ *= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sC3x3_  *= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sC3x3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "DMatSMatSchur - Multiplication assignment to second operand of right-hand side compound";

      initialize();

      result_ =  sD3x3_;
      result_ *= dC3x3_ % ( sC3x3_ * sD3x3_ );
      sD3x3_  *= dC3x3_ % ( sC3x3_ * sD3x3_ );

      checkResult( sD3x3_, result_ );
   }

   // Complex operation: A *= ( 2*A ) % ( B * C )
   {
      test_ = "DMatSMatSchur - Complex operation: A *= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ *= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );
      dC3x3_  *= ( 2*dC3x3_ ) % ( sA3x4_ * sB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A *= ( B * C ) % ( 2*A )
   {
      test_ = "DMatSMatSchur - Complex operation: A *= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  sD3x3_;
      result_ *= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );
      sD3x3_  *= ( dA3x4_ * dB4x3_ ) % ( 2*sD3x3_ );

      checkResult( sD3x3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the dense matrix/transpose dense matrix Schur product.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense matrix/transpose dense matrix Schur
// product. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testDMatTSMatSchur()
{
   //=====================================================================================
   // Schur product
   //=====================================================================================

   // Assignment to left-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Assignment to left-hand side operand (1)";

      initialize();

      result_ = dC3x3_ % tsD3x3_;
      dC3x3_  = dC3x3_ % tsD3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Assignment to left-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Assignment to left-hand side operand (2)";

      initialize();

      result_ = dC3x3_ % eval( tsD3x3_ );
      dC3x3_  = dC3x3_ % eval( tsD3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % tsC3x3_;
      dA3x4_  = ( dA3x4_ * dB4x3_ ) % tsC3x3_;

      checkResult( dA3x4_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % tsC3x3_;
      dB4x3_  = ( dA3x4_ * dB4x3_ ) % tsC3x3_;

      checkResult( dB4x3_, result_ );
   }

   // Assignment to right-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Assignment to right-hand side operand (1)";

      initialize();

      result_ = dC3x3_ % tsD3x3_;
      tsD3x3_ = dC3x3_ % tsD3x3_;

      checkResult( tsD3x3_, result_ );
   }

   // Assignment to right-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Assignment to right-hand side operand (2)";

      initialize();

      result_ = eval( dC3x3_ ) % tsD3x3_;
      tsD3x3_ = eval( dC3x3_ ) % tsD3x3_;

      checkResult( tsD3x3_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = dC3x3_ % ( tsA3x4_ * tsB4x3_ );
      tsA3x4_ = dC3x3_ % ( tsA3x4_ * tsB4x3_ );

      checkResult( tsA3x4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = dC3x3_ % ( tsA3x4_ * tsB4x3_ );
      tsB4x3_ = dC3x3_ % ( tsA3x4_ * tsB4x3_ );

      checkResult( tsB4x3_, result_ );
   }

   // Complex operation: A = ( 2*A ) % ( B * C )
   {
      test_ = "DMatTSMatSchur - Complex operation: A = ( 2*A ) % ( B * C )";

      initialize();

      result_ = ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );
      dC3x3_  = ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A = ( B * C ) % ( 2*A )
   {
      test_ = "DMatTSMatSchur - Complex operation: A = ( B * C ) % ( 2*A )";

      initialize();

      result_ = ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );
      tsD3x3_ = ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Addition assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ += dC3x3_ % tsC3x3_;
      dC3x3_  += dC3x3_ % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to left-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Addition assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ += dC3x3_ % eval( tsC3x3_ );
      dC3x3_  += dC3x3_ % eval( tsC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ += ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dC3x3_  += ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ += ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dD3x3_  += ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Addition assignment to right-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Addition assignment to right-hand side operand (1)";

      initialize();

      result_ =  tsC3x3_;
      result_ += dC3x3_ % tsC3x3_;
      tsC3x3_ += dC3x3_ % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Addition assignment to right-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Addition assignment to right-hand side operand (2)";

      initialize();

      result_ =  tsC3x3_;
      result_ += eval( dC3x3_ ) % tsC3x3_;
      tsC3x3_ += eval( dC3x3_ ) % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Addition assignment to first operand of right-hand side compound";

      initialize();

      result_ =  tsC3x3_;
      result_ += dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsC3x3_ += dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsC3x3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Addition assignment to second operand of right-hand side compound";

      initialize();

      result_ =  tsD3x3_;
      result_ += dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsD3x3_ += dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }

   // Complex operation: A += ( 2*A ) % ( B * C )
   {
      test_ = "DMatTSMatSchur - Complex operation: A += ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ += ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );
      dC3x3_  += ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A += ( B * C ) % ( 2*A )
   {
      test_ = "DMatTSMatSchur - Complex operation: A += ( B * C ) % ( 2*A )";

      initialize();

      result_ =  tsD3x3_;
      result_ += ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );
      tsD3x3_ += ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with subtraction assignment
   //=====================================================================================

   // Schur product assignment to left-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ -= dC3x3_ % tsC3x3_;
      dC3x3_  -= dC3x3_ % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to left-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ -= dC3x3_ % eval( tsC3x3_ );
      dC3x3_  -= dC3x3_ % eval( tsC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to first operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ -= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dC3x3_  -= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to second operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ -= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dD3x3_  -= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to right-hand side operand (1)";

      initialize();

      result_ =  tsC3x3_;
      result_ -= dC3x3_ % tsC3x3_;
      tsC3x3_ -= dC3x3_ % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to right-hand side operand (2)";

      initialize();

      result_ =  tsC3x3_;
      result_ -= eval( dC3x3_ ) % tsC3x3_;
      tsC3x3_ -= eval( dC3x3_ ) % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to first operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to first operand of right-hand side compound";

      initialize();

      result_ =  tsC3x3_;
      result_ -= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsC3x3_ -= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to second operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Subtraction assignment to second operand of right-hand side compound";

      initialize();

      result_ =  tsD3x3_;
      result_ -= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsD3x3_ -= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }

   // Complex operation: A -= ( 2*A ) % ( B * C )
   {
      test_ = "DMatTSMatSchur - Complex operation: A -= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ -= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );
      dC3x3_  -= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A -= ( B * C ) % ( 2*A )
   {
      test_ = "DMatTSMatSchur - Complex operation: A -= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  tsD3x3_;
      result_ -= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );
      tsD3x3_ -= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with Schur product assignment
   //=====================================================================================

   // Schur product assignment to left-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Schur product assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ %= dC3x3_ % tsC3x3_;
      dC3x3_  %= dC3x3_ % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to left-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Schur product assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ %= dC3x3_ % eval( tsC3x3_ );
      dC3x3_  %= dC3x3_ % eval( tsC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to first operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Schur product assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ %= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dC3x3_  %= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Schur product assignment to second operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Schur product assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ %= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dD3x3_  %= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Schur product assignment to right-hand side operand (1)";

      initialize();

      result_ =  tsC3x3_;
      result_ %= dC3x3_ % tsC3x3_;
      tsC3x3_ %= dC3x3_ % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to right-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Schur product assignment to right-hand side operand (2)";

      initialize();

      result_ =  tsC3x3_;
      result_ %= eval( dC3x3_ ) % tsC3x3_;
      tsC3x3_ %= eval( dC3x3_ ) % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to first operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Schur product assignment to first operand of right-hand side compound";

      initialize();

      result_ =  tsC3x3_;
      result_ %= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsC3x3_ %= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsC3x3_, result_ );
   }

   // Schur product assignment to second operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Schur product assignment to second operand of right-hand side compound";

      initialize();

      result_ =  tsD3x3_;
      result_ %= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsD3x3_ %= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }

   // Complex operation: A %= ( 2*A ) % ( B * C )
   {
      test_ = "DMatTSMatSchur - Complex operation: A %= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ %= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );
      dC3x3_  %= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A %= ( B * C ) % ( 2*A )
   {
      test_ = "DMatTSMatSchur - Complex operation: A %= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  tsD3x3_;
      result_ %= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );
      tsD3x3_ %= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }


   //=====================================================================================
   // Schur product with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to left-hand side operand (1)";

      initialize();

      result_ =  dC3x3_;
      result_ *= dC3x3_ % tsC3x3_;
      dC3x3_  *= dC3x3_ % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to left-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to left-hand side operand (2)";

      initialize();

      result_ =  dC3x3_;
      result_ *= dC3x3_ % eval( tsC3x3_ );
      dC3x3_  *= dC3x3_ % eval( tsC3x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dC3x3_;
      result_ *= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dC3x3_  *= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dC3x3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dD3x3_;
      result_ *= ( dC3x3_ * dD3x3_ ) % tsC3x3_;
      dD3x3_  *= ( dC3x3_ * dD3x3_ ) % tsC3x3_;

      checkResult( dD3x3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (1)
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to right-hand side operand (1)";

      initialize();

      result_ =  tsC3x3_;
      result_ *= dC3x3_ % tsC3x3_;
      tsC3x3_ *= dC3x3_ % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Multiplication assignment to right-hand side operand (2)
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to right-hand side operand (2)";

      initialize();

      result_ =  tsC3x3_;
      result_ *= eval( dC3x3_ ) % tsC3x3_;
      tsC3x3_ *= eval( dC3x3_ ) % tsC3x3_;

      checkResult( tsC3x3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to first operand of right-hand side compound";

      initialize();

      result_ =  tsC3x3_;
      result_ *= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsC3x3_ *= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsC3x3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "DMatTSMatSchur - Multiplication assignment to second operand of right-hand side compound";

      initialize();

      result_ =  tsD3x3_;
      result_ *= dC3x3_ % ( tsC3x3_ * tsD3x3_ );
      tsD3x3_ *= dC3x3_ % ( tsC3x3_ * tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }

   // Complex operation: A *= ( 2*A ) % ( B * C )
   {
      test_ = "DMatTSMatSchur - Complex operation: A *= ( 2*A ) % ( B * C )";

      initialize();

      result_ =  dC3x3_;
      result_ *= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );
      dC3x3_  *= ( 2*dC3x3_ ) % ( tsA3x4_ * tsB4x3_ );

      checkResult( dC3x3_, result_ );
   }

   // Complex operation: A *= ( B * C ) % ( 2*A )
   {
      test_ = "DMatTSMatSchur - Complex operation: A *= ( B * C ) % ( 2*A )";

      initialize();

      result_ =  tsD3x3_;
      result_ *= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );
      tsD3x3_ *= ( dA3x4_ * dB4x3_ ) % ( 2*tsD3x3_ );

      checkResult( tsD3x3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the dense matrices
   //=====================================================================================

   // Initializing the first row-major dense matrix
   dA3x4_.resize( 3UL, 4UL, false );
   dA3x4_(0,0) = -1;
   dA3x4_(0,1) =  0;
   dA3x4_(0,2) = -2;
   dA3x4_(0,3) =  0;
   dA3x4_(1,0) =  0;
   dA3x4_(1,1) =  2;
   dA3x4_(1,2) = -3;
   dA3x4_(1,3) =  1;
   dA3x4_(2,0) =  0;
   dA3x4_(2,1) =  1;
   dA3x4_(2,2) =  2;
   dA3x4_(2,3) =  2;

   // Initializing the second row-major dense matrix
   dB4x3_.resize( 4UL, 3UL, false );
   dB4x3_(0,0) =  1;
   dB4x3_(0,1) =  0;
   dB4x3_(0,2) = -3;
   dB4x3_(1,0) =  0;
   dB4x3_(1,1) = -1;
   dB4x3_(1,2) =  0;
   dB4x3_(2,0) =  0;
   dB4x3_(2,1) =  2;
   dB4x3_(2,2) =  1;
   dB4x3_(3,0) =  2;
   dB4x3_(3,1) =  1;
   dB4x3_(3,2) = -2;

   // Initializing the third row-major dense matrix
   dC3x3_.resize( 3UL, 3UL, false );
   dC3x3_(0,0) =  1;
   dC3x3_(0,1) =  0;
   dC3x3_(0,2) =  2;
   dC3x3_(1,0) =  0;
   dC3x3_(1,1) =  3;
   dC3x3_(1,2) = -1;
   dC3x3_(2,0) = -1;
   dC3x3_(2,1) =  0;
   dC3x3_(2,2) =  2;

   // Initializing the fourth row-major dense matrix
   dD3x3_.resize( 3UL, 3UL, false );
   dD3x3_(0,0) =  0;
   dD3x3_(0,1) = -1;
   dD3x3_(0,2) =  0;
   dD3x3_(1,0) =  1;
   dD3x3_(1,1) = -2;
   dD3x3_(1,2) =  2;
   dD3x3_(2,0) =  0;
   dD3x3_(2,1) =  0;
   dD3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the sparse matrices
   //=====================================================================================

   // Initializing the first row-major dense matrix
   sA3x4_.resize( 3UL, 4UL, false );
   sA3x4_.reset();
   sA3x4_(0,0) = -1;
   sA3x4_(0,2) = -2;
   sA3x4_(1,1) =  2;
   sA3x4_(1,2) = -3;
   sA3x4_(1,3) =  1;
   sA3x4_(2,1) =  1;
   sA3x4_(2,2) =  2;
   sA3x4_(2,3) =  2;

   // Initializing the second row-major dense matrix
   sB4x3_.resize( 4UL, 3UL, false );
   sB4x3_.reset();
   sB4x3_(0,0) =  1;
   sB4x3_(0,2) = -3;
   sB4x3_(1,1) = -1;
   sB4x3_(2,1) =  2;
   sB4x3_(2,2) =  1;
   sB4x3_(3,0) =  2;
   sB4x3_(3,1) =  1;
   sB4x3_(3,2) = -2;

   // Initializing the third row-major dense matrix
   sC3x3_.resize( 3UL, 3UL, false );
   sC3x3_.reset();
   sC3x3_(0,0) =  1;
   sC3x3_(0,2) =  2;
   sC3x3_(1,1) =  3;
   sC3x3_(1,2) = -1;
   sC3x3_(2,0) = -1;
   sC3x3_(2,2) =  2;

   // Initializing the fourth row-major dense matrix
   sD3x3_.resize( 3UL, 3UL, false );
   sD3x3_.reset();
   sD3x3_(0,1) = -1;
   sD3x3_(1,0) =  1;
   sD3x3_(1,1) = -2;
   sD3x3_(1,2) =  2;
   sD3x3_(2,2) = -3;

   // Initializing the first column-major dense matrix
   tsA3x4_.resize( 3UL, 4UL, false );
   tsA3x4_.reset();
   tsA3x4_(0,0) = -1;
   tsA3x4_(0,2) = -2;
   tsA3x4_(1,1) =  2;
   tsA3x4_(1,2) = -3;
   tsA3x4_(1,3) =  1;
   tsA3x4_(2,1) =  1;
   tsA3x4_(2,2) =  2;
   tsA3x4_(2,3) =  2;

   // Initializing the second column-major dense matrix
   tsB4x3_.resize( 4UL, 3UL, false );
   tsB4x3_.reset();
   tsB4x3_(0,0) =  1;
   tsB4x3_(0,2) = -3;
   tsB4x3_(1,1) = -1;
   tsB4x3_(2,1) =  2;
   tsB4x3_(2,2) =  1;
   tsB4x3_(3,0) =  2;
   tsB4x3_(3,1) =  1;
   tsB4x3_(3,2) = -2;

   // Initializing the third column-major dense matrix
   tsC3x3_.resize( 3UL, 3UL, false );
   tsC3x3_.reset();
   tsC3x3_(0,0) =  1;
   tsC3x3_(0,2) =  2;
   tsC3x3_(1,1) =  3;
   tsC3x3_(1,2) = -1;
   tsC3x3_(2,0) = -1;
   tsC3x3_(2,2) =  2;

   // Initializing the fourth column-major dense matrix
   tsD3x3_.resize( 3UL, 3UL, false );
   tsD3x3_.reset();
   tsD3x3_(0,1) = -1;
   tsD3x3_(1,0) =  1;
   tsD3x3_(1,1) = -2;
   tsD3x3_(1,2) =  2;
   tsD3x3_(2,2) = -3;
}
//*************************************************************************************************

} // namespace dmatsmatschur

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_DMATSMATSCHUR_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
