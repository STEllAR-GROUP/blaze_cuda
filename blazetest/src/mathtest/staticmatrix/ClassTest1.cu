#include "hip/hip_runtime.h"
//=================================================================================================
/*!
//  \file src/mathtest/staticmatrix/ClassTest1.cpp
//  \brief Source file for the StaticMatrix class test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/CompressedMatrix.h>
#include <blaze/math/CustomMatrix.h>
#include <blaze/math/DiagonalMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/LowerMatrix.h>
#include <blaze/math/UpperMatrix.h>
#include <blaze/util/Complex.h>
#include <blaze/util/Memory.h>
#include <blaze/util/policies/Deallocate.h>
#include <blaze/util/Random.h>
#include <blazetest/mathtest/staticmatrix/ClassTest.h>
#include <blazetest/mathtest/RandomMaximum.h>
#include <blazetest/mathtest/RandomMinimum.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace staticmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the StaticMatrix class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testAlignment< char           >( "char"           );
   testAlignment< signed char    >( "signed char"    );
   testAlignment< unsigned char  >( "unsigned char"  );
   testAlignment< wchar_t        >( "wchar_t"        );
   testAlignment< short          >( "short"          );
   testAlignment< unsigned short >( "unsigned short" );
   testAlignment< int            >( "int"            );
   testAlignment< unsigned int   >( "unsigned int"   );
   testAlignment< long           >( "long"           );
   testAlignment< unsigned long  >( "unsigned long"  );
   testAlignment< float          >( "float"          );
   testAlignment< double         >( "double"         );

   testAlignment< complex<char>           >( "complex<char>"           );
   testAlignment< complex<signed char>    >( "complex<signed char>"    );
   testAlignment< complex<unsigned char>  >( "complex<unsigned char>"  );
   testAlignment< complex<wchar_t>        >( "complex<wchar_t>"        );
   testAlignment< complex<short>          >( "complex<short>"          );
   testAlignment< complex<unsigned short> >( "complex<unsigned short>" );
   testAlignment< complex<int>            >( "complex<int>"            );
   testAlignment< complex<unsigned int>   >( "complex<unsigned int>"   );
   testAlignment< complex<float>          >( "complex<float>"          );
   testAlignment< complex<double>         >( "complex<double>"         );

   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the StaticMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the StaticMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix default constructor (0x0)";

      blaze::StaticMatrix<int,0UL,0UL,blaze::rowMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major StaticMatrix default constructor (0x4)";

      blaze::StaticMatrix<int,0UL,4UL,blaze::rowMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major StaticMatrix default constructor (3x0)";

      blaze::StaticMatrix<int,3UL,0UL,blaze::rowMajor> mat;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Row-major StaticMatrix default constructor (3x4)";

      blaze::StaticMatrix<int,3UL,4UL,blaze::rowMajor> mat;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat,  0UL );
      checkNonZeros( mat,  0UL, 0UL );
      checkNonZeros( mat,  1UL, 0UL );
      checkNonZeros( mat,  2UL, 0UL );

      if( mat(0,0) != 0 || mat(0,1) != 0 || mat(0,2) != 0 || mat(0,3) != 0 ||
          mat(1,0) != 0 || mat(1,1) != 0 || mat(1,2) != 0 || mat(1,3) != 0 ||
          mat(2,0) != 0 || mat(2,1) != 0 || mat(2,2) != 0 || mat(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major homogeneous initialization
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix homogeneous initialization constructor";

      blaze::StaticMatrix<int,3UL,4UL,blaze::rowMajor> mat( 2 );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 4UL );
      checkNonZeros( mat,  1UL, 4UL );
      checkNonZeros( mat,  2UL, 4UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major list initialization
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix initializer list constructor (incomplete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat{ { 1 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 4UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix initializer list constructor (complete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat{ { 1, 2, 3 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 3UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major array initialization
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix dynamic array initialization constructor";

      std::unique_ptr<int[]> array( new int[6] );
      array[0] = 1;
      array[1] = 2;
      array[2] = 3;
      array[3] = 4;
      array[4] = 5;
      array[5] = 6;
      blaze::StaticMatrix<int,3UL,4UL,blaze::rowMajor> mat( 2UL, 3UL, array.get() );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat,  6UL );
      checkNonZeros( mat,  0UL, 3UL );
      checkNonZeros( mat,  1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 || mat(0,3) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 || mat(1,3) != 0 ||
          mat(2,0) != 0 || mat(2,1) != 0 || mat(2,2) != 0 || mat(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 0 )\n( 4 5 6 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix dynamic array initialization constructor";

      std::unique_ptr<int[]> array( new int[6] );
      array[0] = 1;
      array[1] = 2;
      array[2] = 3;
      array[3] = 4;
      array[4] = 5;
      array[5] = 6;
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat( 2UL, 3UL, array.get() );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 3UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix static array initialization constructor";

      const int array[2][3] = { { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat( array );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 3UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix copy constructor";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat1{ { 1, 2, 3 },
                                                             { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix default constructor (0x0)";

      blaze::StaticMatrix<int,0UL,0UL,blaze::columnMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major StaticMatrix default constructor (0x4)";

      blaze::StaticMatrix<int,0UL,4UL,blaze::columnMajor> mat;

      checkRows    ( mat, 0UL );
      checkColumns ( mat, 4UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major StaticMatrix default constructor (3x0)";

      blaze::StaticMatrix<int,3UL,0UL,blaze::columnMajor> mat;

      checkRows    ( mat, 3UL );
      checkColumns ( mat, 0UL );
      checkCapacity( mat, 0UL );
      checkNonZeros( mat, 0UL );
   }

   {
      test_ = "Column-major StaticMatrix default constructor (3x4)";

      blaze::StaticMatrix<int,3UL,4UL,blaze::columnMajor> mat;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat,  0UL );
      checkNonZeros( mat,  0UL, 0UL );
      checkNonZeros( mat,  1UL, 0UL );
      checkNonZeros( mat,  2UL, 0UL );
      checkNonZeros( mat,  3UL, 0UL );

      if( mat(0,0) != 0 || mat(0,1) != 0 || mat(0,2) != 0 || mat(0,3) != 0 ||
          mat(1,0) != 0 || mat(1,1) != 0 || mat(1,2) != 0 || mat(1,3) != 0 ||
          mat(2,0) != 0 || mat(2,1) != 0 || mat(2,2) != 0 || mat(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major homogeneous initialization
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix homogeneous initialization constructor";

      blaze::StaticMatrix<int,3UL,4UL,blaze::columnMajor> mat( 2 );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 3UL );
      checkNonZeros( mat,  1UL, 3UL );
      checkNonZeros( mat,  2UL, 3UL );
      checkNonZeros( mat,  3UL, 3UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major list initialization
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix initializer list constructor (incomplete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat{ { 1 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 4UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major StaticMatrix initializer list constructor (complete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat{ { 1, 2, 3 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 2UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major array initialization
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix dynamic array initialization constructor";

      std::unique_ptr<int[]> array( new int[6] );
      array[0] = 1;
      array[1] = 2;
      array[2] = 3;
      array[3] = 4;
      array[4] = 5;
      array[5] = 6;
      blaze::StaticMatrix<int,3UL,4UL,blaze::columnMajor> mat( 2UL, 3UL, array.get() );

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat,  6UL );
      checkNonZeros( mat,  0UL, 2UL );
      checkNonZeros( mat,  1UL, 2UL );
      checkNonZeros( mat,  2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 3 || mat(0,2) != 5 || mat(0,3) != 0 ||
          mat(1,0) != 2 || mat(1,1) != 4 || mat(1,2) != 6 || mat(1,3) != 0 ||
          mat(2,0) != 0 || mat(2,1) != 0 || mat(2,2) != 0 || mat(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 3 5 0 )\n( 2 4 6 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major StaticMatrix dynamic array initialization constructor";

      std::unique_ptr<int[]> array( new int[6] );
      array[0] = 1;
      array[1] = 2;
      array[2] = 3;
      array[3] = 4;
      array[4] = 5;
      array[5] = 6;
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat( 2UL, 3UL, array.get() );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 2UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 3 || mat(0,2) != 5 ||
          mat(1,0) != 2 || mat(1,1) != 4 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 3 5 )\n( 2 4 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major StaticMatrix static array initialization constructor";

      const int array[2][3] = { { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat( array );

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 2UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix copy constructor";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat1{ { 1, 3, 5 },
                                                                { 2, 4, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 3 || mat2(0,2) != 5 ||
          mat2(1,0) != 2 || mat2(1,1) != 4 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 3 5 )\n( 2 4 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix constructor (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix constructor (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      const blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2( mat1 );

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the StaticMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the StaticMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAssignment()
{
   //=====================================================================================
   // Row-major homogeneous assignment
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix homogeneous assignment";

      blaze::StaticMatrix<int,3UL,4UL,blaze::rowMajor> mat;
      mat = 2;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 4UL );
      checkNonZeros( mat,  1UL, 4UL );
      checkNonZeros( mat,  2UL, 4UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major list assignment
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix initializer list assignment (complete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat;
      mat = { { 1, 2, 3 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 3UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix initializer list assignment (incomplete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat;
      mat = { { 1 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 4UL );
      checkNonZeros( mat, 0UL, 1UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major array assignment
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix array assignment";

      const int array[2][3] = { { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat;
      mat = array;

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 3UL );
      checkNonZeros( mat, 1UL, 3UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   {
      test_ = "Row-major StaticMatrix copy assignment";

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat1{ { 1, 2, 3 },
                                                             { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix copy assignment stress test";

      using RandomMatrixType = blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::rowMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 3UL );
      checkNonZeros( mat2, 1UL, 3UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::columnMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::rowMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::columnMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::rowMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major homogeneous assignment
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix homogeneous assignment";

      blaze::StaticMatrix<int,3UL,4UL,blaze::columnMajor> mat;
      mat = 2;

      checkRows    ( mat,  3UL );
      checkColumns ( mat,  4UL );
      checkCapacity( mat, 12UL );
      checkNonZeros( mat, 12UL );
      checkNonZeros( mat,  0UL, 3UL );
      checkNonZeros( mat,  1UL, 3UL );
      checkNonZeros( mat,  2UL, 3UL );
      checkNonZeros( mat,  3UL, 3UL );

      if( mat(0,0) != 2 || mat(0,1) != 2 || mat(0,2) != 2 || mat(0,3) != 2 ||
          mat(1,0) != 2 || mat(1,1) != 2 || mat(1,2) != 2 || mat(1,3) != 2 ||
          mat(2,0) != 2 || mat(2,1) != 2 || mat(2,2) != 2 || mat(2,3) != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 2 2 2 2 )\n( 2 2 2 2 )\n( 2 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major list assignment
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix initializer list assignment (complete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat;
      mat = { { 1, 2, 3 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 2UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major StaticMatrix initializer list assignment (incomplete list)";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat;
      mat = { { 1 }, { 4, 5, 6 } };

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 4UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 1UL );
      checkNonZeros( mat, 2UL, 1UL );

      if( mat(0,0) != 1 || mat(0,1) != 0 || mat(0,2) != 0 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 0 0 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major array assignment
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix array assignment";

      const int array[2][3] = { { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat;
      mat = array;

      checkRows    ( mat, 2UL );
      checkColumns ( mat, 3UL );
      checkCapacity( mat, 6UL );
      checkNonZeros( mat, 6UL );
      checkNonZeros( mat, 0UL, 2UL );
      checkNonZeros( mat, 1UL, 2UL );
      checkNonZeros( mat, 2UL, 2UL );

      if( mat(0,0) != 1 || mat(0,1) != 2 || mat(0,2) != 3 ||
          mat(1,0) != 4 || mat(1,1) != 5 || mat(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   {
      test_ = "Column-major StaticMatrix copy assignment";

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat1{ { 1, 3, 5 },
                                                                { 2, 4, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 3 || mat2(0,2) != 5 ||
          mat2(1,0) != 2 || mat2(1,1) != 4 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 3 5 )\n( 2 4 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major StaticMatrix copy assignment stress test";

      using RandomMatrixType = blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::rowMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ { 1, 2, 3 }, { 4, 5, 6 } };
      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(0,2) = 3;
      mat1(1,0) = 4;
      mat1(1,1) = 5;
      mat1(1,2) = 6;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 6UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 3 ||
          mat2(1,0) != 4 || mat2(1,1) != 5 || mat2(1,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 4 5 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment stress test";

      using RandomMatrixType = blaze::DynamicMatrix<int,blaze::columnMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::rowMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL );
      mat1(0,0) = 1;
      mat1(0,1) = 2;
      mat1(1,0) = 3;
      mat1(1,2) = 4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2;
      mat2 = mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 1UL );
      checkNonZeros( mat2, 2UL, 1UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 2 || mat2(0,2) != 0 ||
          mat2(1,0) != 3 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 2 0 )\n( 3 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix assignment stress test";

      using RandomMatrixType = blaze::CompressedMatrix<int,blaze::columnMajor>;

      blaze::StaticMatrix<int,4UL,3UL,blaze::columnMajor> mat1;
      const int min( randmin );
      const int max( randmax );

      for( size_t i=0UL; i<100UL; ++i )
      {
         const RandomMatrixType mat2( blaze::rand<RandomMatrixType>( 4UL, 3UL, min, max ) );

         mat1 = mat2;

         if( mat1 != mat2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << mat1 << "\n"
                << "   Expected result:\n" << mat2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;
      randomize( mat2 );

      mat2 = mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the StaticMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the StaticMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAddAssign()
{
   //=====================================================================================
   // Row-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ {  1, 2, 0 },
                                                               { -3, 0, 4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ {  1, 2, 0 },
                                                                  { -3, 0, 4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ {  1, 2, 0 },
                                                               { -3, 0, 4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ {  1, 2, 0 },
                                                                  { -3, 0, 4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix addition assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix addition assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) =  1;
      mat1(0,1) =  2;
      mat1(1,0) = -3;
      mat1(1,2) =  4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 += mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix addition assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix addition assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix addition assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 += mat1;

      if( mat1 != mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the StaticMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the StaticMatrix
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSubAssign()
{
   //=====================================================================================
   // Row-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ { -1, -2,  0 },
                                                               {  3,  0, -4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ { -1, -2,  0 },
                                                                  {  3,  0, -4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::rowMajor> mat2{ { 0, -2, 6 },
                                                             { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/row-major StaticMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major StaticMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::rowMajor> mat1{ { -1, -2,  0 },
                                                               {  3,  0, -4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,rowMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 32UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,rowMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (mixed type)";

      blaze::StaticMatrix<short,2UL,3UL,blaze::columnMajor> mat1{ { -1, -2,  0 },
                                                                  {  3,  0, -4 } };

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::columnMajor;

      using AlignedPadded = blaze::CustomMatrix<int,aligned,padded,columnMajor>;
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 48UL ) );
      AlignedPadded mat1( memory.get(), 2UL, 3UL, 16UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::columnMajor;

      using UnalignedUnpadded = blaze::CustomMatrix<int,unaligned,unpadded,columnMajor>;
      std::unique_ptr<int[]> memory( new int[7UL] );
      UnalignedUnpadded mat1( memory.get()+1UL, 2UL, 3UL );
      mat1 = 0;
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix dense matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > mat1;
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix subtraction assignment
   //=====================================================================================

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix subtraction assignment";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat1( 2UL, 3UL, 4UL );
      mat1(0,0) = -1;
      mat1(0,1) = -2;
      mat1(1,0) =  3;
      mat1(1,2) = -4;

      blaze::StaticMatrix<int,2UL,3UL,blaze::columnMajor> mat2{ { 0, -2, 6 },
                                                                { 5,  0, 0 } };

      mat2 -= mat1;

      checkRows    ( mat2, 2UL );
      checkColumns ( mat2, 3UL );
      checkCapacity( mat2, 6UL );
      checkNonZeros( mat2, 4UL );
      checkNonZeros( mat2, 0UL, 2UL );
      checkNonZeros( mat2, 1UL, 0UL );
      checkNonZeros( mat2, 2UL, 2UL );

      if( mat2(0,0) != 1 || mat2(0,1) != 0 || mat2(0,2) != 6 ||
          mat2(1,0) != 2 || mat2(1,1) != 0 || mat2(1,2) != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat2 << "\n"
             << "   Expected result:\n( 1 0 6 )\n( 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix subtraction assignment (lower)";

      blaze::LowerMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix subtraction assignment (upper)";

      blaze::UpperMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/row-major StaticMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major StaticMatrix sparse matrix subtraction assignment (diagonal)";

      blaze::DiagonalMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > mat1( 3UL );
      randomize( mat1 );

      blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat2;

      mat2 -= mat1;

      if( mat1 != -mat2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << mat1 << "\n"
             << "   Expected result:\n" << mat2 << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace staticmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running StaticMatrix class test (part 1)..." << std::endl;

   try
   {
      RUN_STATICMATRIX_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during StaticMatrix class test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
