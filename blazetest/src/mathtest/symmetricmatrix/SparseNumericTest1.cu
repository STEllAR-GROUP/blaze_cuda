//=================================================================================================
/*!
//  \file src/mathtest/symmetricmatrix/SparseNumericTest1.cpp
//  \brief Source file for the SymmetricMatrix sparse numeric test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/StaticMatrix.h>
#include <blazetest/mathtest/symmetricmatrix/SparseNumericTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace symmetricmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the SymmetricMatrix sparse numeric test.
//
// \exception std::runtime_error Operation error detected.
*/
SparseNumericTest::SparseNumericTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
   testSchurAssign();
   testMultAssign();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the SymmetricMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   // Default constructor (CompressedMatrix)
   {
      test_ = "Row-major SymmetricMatrix default constructor (CompressedMatrix)";

      const ST sym;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   // Size constructor (CompressedMatrix)
   {
      test_ = "Row-major SymmetricMatrix size constructor (CompressedMatrix)";

      const ST sym( 2UL );

      checkRows    ( sym, 2UL );
      checkColumns ( sym, 2UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Row-major list initialization
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Row-major SymmetricMatrix initializer list constructor (complete list)";

      const ST sym{ { 1, 2, 3 }, { 2, 4, 0 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Row-major SymmetricMatrix initializer list constructor (incomplete list)";

      const ST sym{ { 1, 2, 3 }, { 2, 4 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix copy constructor (0x0)";

      const ST sym1;
      const ST sym2( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Row-major SymmetricMatrix copy constructor (3x3)";

      ST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      const ST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix move constructor (0x0)";

      ST sym1;
      ST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Row-major SymmetricMatrix move constructor (3x3)";

      ST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<int,blaze::rowMajor> mat;
      const ST sym( mat );

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Conversion constructor (symmetric)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    {  7,  0, 3 } } );

      const ST sym( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (non-symmetric)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    { -5,  0, 3 } } );

      try {
         const ST sym( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-symmetric SymmetricMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (SymmetricMatrix)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      const ST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   // Default constructor (CompressedMatrix)
   {
      test_ = "Column-major SymmetricMatrix default constructor (CompressedMatrix)";

      const OST sym;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   // Size constructor (CompressedMatrix)
   {
      test_ = "Column-major SymmetricMatrix size constructor (CompressedMatrix)";

      const OST sym( 2UL );

      checkRows    ( sym, 2UL );
      checkColumns ( sym, 2UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Column-major list initialization
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Column-major SymmetricMatrix initializer list constructor (complete list)";

      const OST sym{ { 1, 2, 3 }, { 2, 4, 0 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Column-major SymmetricMatrix initializer list constructor (incomplete list)";

      const OST sym{ { 1, 2, 3 }, { 2, 4 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix copy constructor (0x0)";

      const OST sym1;
      const OST sym2( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Column-major SymmetricMatrix copy constructor (3x3)";

      OST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      const OST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix move constructor (0x0)";

      OST sym1;
      OST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Column-major SymmetricMatrix move constructor (3x3)";

      OST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<int,blaze::columnMajor> mat;
      const OST sym( mat );

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Conversion constructor (symmetric)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       {  7,  0, 3 } } );

      const OST sym( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (non-symmetric)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       { -5,  0, 3 } } );

      try {
         const OST sym( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-symmetric SymmetricMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (SymmetricMatrix)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      const OST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the SymmetricMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testAssignment()
{
   //=====================================================================================
   // Row-major list assignment
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Row-major SymmetricMatrix initializer list assignment";

      ST sym;
      sym = { { 1, 2, 3 }, { 2, 4, 0 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Row-major SymmetricMatrix initializer list assignment";

      ST sym;
      sym = { { 1, 2, 3 }, { 2, 4 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix copy assignment (0x0)";

      ST sym1, sym2;

      sym2 = sym1;

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Row-major SymmetricMatrix copy assignment (3x3)";

      ST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix move assignment (0x0)";

      ST sym1, sym2;

      sym2 = std::move( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Row-major SymmetricMatrix move assignment (3x3)";

      ST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = std::move( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<int,blaze::rowMajor> mat;

      ST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Row-major/row-major dense matrix assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    {  7,  0, 3 } } );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       {  7,  0, 3 } } );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    { -5,  0, 3 } } );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       { -5,  0, 3 } } );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<int,blaze::rowMajor> mat;

      ST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Row-major/row-major sparse matrix assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) =  7;
      mat(2,2) =  3;
      mat.insert( 1UL, 2UL, 0 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 8UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) =  7;
      mat(2,2) =  3;
      mat.insert( 1UL, 2UL, 0 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 8UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) = -5;
      mat(2,2) =  3;

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) = -5;
      mat(2,2) =  3;

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 7UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 7UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major list assignment
   //=====================================================================================

   // Complete initializer list
   {
      test_ = "Column-major SymmetricMatrix initializer list assignment";

      OST sym;
      sym = { { 1, 2, 3 }, { 2, 4, 0 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Incomplete initializer list
   {
      test_ = "Column-major SymmetricMatrix initializer list assignment";

      OST sym;
      sym = { { 1, 2, 3 }, { 2, 4 }, { 3, 0, 6 } };

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != 1 || sym(0,1) != 2 || sym(0,2) != 3 ||
          sym(1,0) != 2 || sym(1,1) != 4 || sym(1,2) != 0 ||
          sym(2,0) != 3 || sym(2,1) != 0 || sym(2,2) != 6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( 1 2 3 )\n( 2 4 0 )\n( 3 0 6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix copy assignment (0x0)";

      OST sym1, sym2;

      sym2 = sym1;

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Column-major SymmetricMatrix copy assignment (3x3)";

      OST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix move assignment (0x0)";

      OST sym1, sym2;

      sym2 = std::move( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Column-major SymmetricMatrix move assignment (3x3)";

      OST sym1( 3UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = std::move( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<int,blaze::columnMajor> mat;

      OST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Column-major/row-major dense matrix assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    {  7,  0, 3 } } );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       {  7,  0, 3 } } );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> mat( { {  1, -4, 7 },
                                                                    { -4,  2, 0 },
                                                                    { -5,  0, 3 } } );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      const blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> mat( { {  1, -4, 7 },
                                                                       { -4,  2, 0 },
                                                                       { -5,  0, 3 } } );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<int,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<int,blaze::columnMajor> mat;

      OST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Column-major/row-major sparse matrix assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) =  7;
      mat(2,2) =  3;
      mat.insert( 1UL, 2UL, 0 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 8UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 8UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) =  7;
      mat(2,2) =  3;
      mat.insert( 1UL, 2UL, 0 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 8UL );

      if( sym(0,0) !=  1 || sym(0,1) != -4 || sym(0,2) != 7 ||
          sym(1,0) != -4 || sym(1,1) !=  2 || sym(1,2) != 0 ||
          sym(2,0) !=  7 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) = -5;
      mat(2,2) =  3;

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) =  1;
      mat(0,1) = -4;
      mat(0,2) =  7;
      mat(1,0) = -4;
      mat(1,1) =  2;
      mat(2,0) = -5;
      mat(2,2) =  3;

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 7UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 7UL );
      sym1(0,0) =  1;
      sym1(0,1) = -4;
      sym1(0,2) =  7;
      sym1(1,1) =  2;
      sym1(2,2) =  3;

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -4 || sym2(0,2) != 7 ||
          sym2(1,0) != -4 || sym2(1,1) !=  2 || sym2(1,2) != 0 ||
          sym2(2,0) !=  7 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -4  7 )\n( -4  2  0 )\n(  7  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testAddAssign()
{
   //=====================================================================================
   // Row-major dense matrix addition assignment
   //=====================================================================================

   // Row-major/row-major dense matrix addition assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix addition assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix addition assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix addition assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix addition assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix addition assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix addition assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix addition assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix addition assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix addition assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix addition assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix addition assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix addition assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix addition assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix addition assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix addition assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix addition assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix addition assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix addition assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix addition assignment
   //=====================================================================================

   // Column-major/row-major dense matrix addition assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix addition assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix addition assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix addition assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix addition assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix addition assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix addition assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix addition assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix addition assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix addition assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix addition assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix addition assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix addition assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) !=  1 || sym(0,1) != -6 || sym(0,2) != 13 ||
          sym(1,0) != -6 || sym(1,1) !=  5 || sym(1,2) !=  0 ||
          sym(2,0) != 13 || sym(2,1) !=  0 || sym(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix addition assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix addition assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix addition assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix addition assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix addition assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix addition assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -6 || sym2(0,2) != 13 ||
          sym2(1,0) != -6 || sym2(1,1) !=  5 || sym2(1,2) !=  0 ||
          sym2(2,0) != 13 || sym2(2,1) !=  0 || sym2(2,2) !=  3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -6 13 )\n( -6  5  0 )\n( 13  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testSubAssign()
{
   //=====================================================================================
   // Row-major dense matrix subtraction assignment
   //=====================================================================================

   // Row-major/row-major dense matrix subtraction assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix subtraction assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix subtraction assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix subtraction assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix subtraction assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix subtraction assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix subtraction assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix subtraction assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix subtraction assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix subtraction assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix subtraction assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix subtraction assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix subtraction assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix subtraction assignment
   //=====================================================================================

   // Column-major/row-major dense matrix subtraction assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix subtraction assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix subtraction assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix subtraction assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix subtraction assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix subtraction assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix subtraction assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix subtraction assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix subtraction assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 8UL );
      checkNonZeros( sym, 8UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) !=  1 || sym(0,1) != -2 || sym(0,2) != 1 ||
          sym(1,0) != -2 || sym(1,1) != -1 || sym(1,2) != 0 ||
          sym(2,0) !=  1 || sym(2,1) !=  0 || sym(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix subtraction assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix subtraction assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix subtraction assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix subtraction assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  1 || sym2(0,1) != -2 || sym2(0,2) != 1 ||
          sym2(1,0) != -2 || sym2(1,1) != -1 || sym2(1,2) != 0 ||
          sym2(2,0) !=  1 || sym2(2,1) !=  0 || sym2(2,2) != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1 -2  1 )\n( -2 -1  0 )\n(  1  0  3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix Schur product assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the Schur product assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testSchurAssign()
{
   //=====================================================================================
   // Row-major dense matrix Schur product assignment
   //=====================================================================================

   // Row-major/row-major dense matrix Schur product assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix Schur product assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix Schur product assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix Schur product assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix Schur product assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix Schur product assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix Schur product assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 5UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix Schur product assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix Schur product assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 5UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix Schur product assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix Schur product assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix Schur product assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix Schur product assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix Schur product assignment
   //=====================================================================================

   // Column-major/row-major dense matrix Schur product assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix Schur product assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix Schur product assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix Schur product assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix Schur product assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix Schur product assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix Schur product assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 5UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix Schur product assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix Schur product assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 6UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,0) = -2;
      mat(1,1) =  3;
      mat(2,0) =  6;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 5UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( sym(0,0) !=  0 || sym(0,1) != 8 || sym(0,2) != 42 ||
          sym(1,0) !=  8 || sym(1,1) != 6 || sym(1,2) !=  0 ||
          sym(2,0) != 42 || sym(2,1) != 0 || sym(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix Schur product assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix Schur product assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix Schur product assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix Schur product assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,1) = -2;
      sym1(0,2) =  6;
      sym1(1,1) =  3;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 5UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( sym2(0,0) !=  0 || sym2(0,1) != 8 || sym2(0,2) != 42 ||
          sym2(1,0) !=  8 || sym2(1,1) != 6 || sym2(1,2) !=  0 ||
          sym2(2,0) != 42 || sym2(2,1) != 0 || sym2(2,2) !=  0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  1  8 42 )\n(  8  6  0 )\n( 42  0  0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix multiplication assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the multiplication assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNumericTest::testMultAssign()
{
   //=====================================================================================
   // Row-major dense matrix multiplication assignment
   //=====================================================================================

   // Row-major/row-major dense matrix multiplication assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix multiplication assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix multiplication assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix multiplication assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix multiplication assignment
   //=====================================================================================

   // Row-major/row-major sparse matrix multiplication assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix multiplication assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix multiplication assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix multiplication assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;
      mat.insert( 1UL, 2UL, 0 );

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix multiplication assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix multiplication assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      ST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix multiplication assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix multiplication assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      ST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix multiplication assignment
   //=====================================================================================

   // Column-major/row-major dense matrix multiplication assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix multiplication assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 0 );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix multiplication assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix multiplication assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix multiplication assignment
   //=====================================================================================

   // Column-major/row-major sparse matrix multiplication assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix multiplication assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix multiplication assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix multiplication assignment (symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,0) = 2;
      mat(1,1) = 2;
      mat(2,2) = 2;
      mat.insert( 1UL, 2UL, 0 );

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 2UL );

      if( sym(0,0) !=  2 || sym(0,1) != -8 || sym(0,2) != 14 ||
          sym(1,0) != -8 || sym(1,1) !=  4 || sym(1,2) !=  0 ||
          sym(2,0) != 14 || sym(2,1) !=  0 || sym(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix multiplication assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::rowMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix multiplication assignment (non-symmetric)";

      blaze::CompressedMatrix<int,blaze::columnMajor> mat( 3UL, 3UL, 4UL );
      mat(0,1) = -2;
      mat(0,2) =  6;
      mat(1,1) =  3;
      mat(2,0) =  6;

      OST sym( 3UL );
      sym(0,0) =  1;
      sym(0,1) = -4;
      sym(0,2) =  7;
      sym(1,1) =  2;
      sym(2,2) =  3;

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix multiplication assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::rowMajor> > sym1( 3UL, 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix multiplication assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<int,blaze::columnMajor> > sym1( 3UL, 3UL );
      sym1(0,0) = 2;
      sym1(1,1) = 2;
      sym1(2,2) = 2;

      OST sym2( 3UL );
      sym2(0,0) =  1;
      sym2(0,1) = -4;
      sym2(0,2) =  7;
      sym2(1,1) =  2;
      sym2(2,2) =  3;

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 2UL );

      if( sym2(0,0) !=  2 || sym2(0,1) != -8 || sym2(0,2) != 14 ||
          sym2(1,0) != -8 || sym2(1,1) !=  4 || sym2(1,2) !=  0 ||
          sym2(2,0) != 14 || sym2(2,1) !=  0 || sym2(2,2) !=  6 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n(  2 -8 14 )\n( -8  4  0 )\n( 14  0  6 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace symmetricmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running SymmetricMatrix sparse numeric test (part 1)..." << std::endl;

   try
   {
      RUN_SYMMETRICMATRIX_SPARSENUMERIC_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during SymmetricMatrix sparse numeric test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
