//=================================================================================================
/*!
//  \file src/mathtest/symmetricmatrix/SparseNonNumericTest1.cpp
//  \brief Source file for the SymmetricMatrix sparse non-numeric test (part 1)
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/StaticMatrix.h>
#include <blazetest/mathtest/symmetricmatrix/SparseNonNumericTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace symmetricmatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the SymmetricMatrix sparse non-numeric test.
//
// \exception std::runtime_error Operation error detected.
*/
SparseNonNumericTest::SparseNonNumericTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
   testSchurAssign();
   testMultAssign();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the SymmetricMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   // Default constructor (CompressedMatrix)
   {
      test_ = "Row-major SymmetricMatrix default constructor (CompressedMatrix)";

      const ST sym;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   // Size constructor (CompressedMatrix)
   {
      test_ = "Row-major SymmetricMatrix size constructor (CompressedMatrix)";

      const ST sym( 2UL );

      checkRows    ( sym, 2UL );
      checkColumns ( sym, 2UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix copy constructor (0x0)";

      const ST sym1;
      const ST sym2( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Row-major SymmetricMatrix copy constructor (3x3)";

      ST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      const ST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) ( 0 ) )\n"
                                     "( (  7 ) (  0 ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix move constructor (0x0)";

      ST sym1;
      ST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Row-major SymmetricMatrix move constructor (3x3)";

      ST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) ( 0 ) )\n"
                                     "( (  7 ) (  0 ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<VT,blaze::rowMajor> mat;
      const ST sym( mat );

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Conversion constructor (symmetric)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      const ST sym( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (non-symmetric)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         const ST sym( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-symmetric SymmetricMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (SymmetricMatrix)
   {
      test_ = "Row-major SymmetricMatrix conversion constructor (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      const ST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   // Default constructor (CompressedMatrix)
   {
      test_ = "Column-major SymmetricMatrix default constructor (CompressedMatrix)";

      const OST sym;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   // Size constructor (CompressedMatrix)
   {
      test_ = "Column-major SymmetricMatrix size constructor (CompressedMatrix)";

      const OST sym( 2UL );

      checkRows    ( sym, 2UL );
      checkColumns ( sym, 2UL );
      checkNonZeros( sym, 0UL );
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   // Copy constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix copy constructor (0x0)";

      const OST sym1;
      const OST sym2( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy constructor (3x3)
   {
      test_ = "Column-major SymmetricMatrix copy constructor (3x3)";

      OST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      const OST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   // Move constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix move constructor (0x0)";

      OST sym1;
      OST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move constructor (3x3)
   {
      test_ = "Column-major SymmetricMatrix move constructor (3x3)";

      OST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2( std::move( sym1 ) );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major conversion constructor
   //=====================================================================================

   // Conversion constructor (0x0)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (0x0)";

      const blaze::DynamicMatrix<VT,blaze::columnMajor> mat;
      const OST sym( mat );

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Conversion constructor (symmetric)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      const OST sym( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkCapacity( sym, 7UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Conversion constructor (non-symmetric)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         const OST sym( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-symmetric SymmetricMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Conversion constructor (SymmetricMatrix)
   {
      test_ = "Column-major SymmetricMatrix conversion constructor (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      const OST sym2( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkCapacity( sym2, 7UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the SymmetricMatrix specialization.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testAssignment()
{
   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix copy assignment (0x0)";

      ST sym1, sym2;

      sym2 = sym1;

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Row-major SymmetricMatrix copy assignment (3x3)";

      ST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix move assignment (0x0)";

      ST sym1, sym2;

      sym2 = std::move( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Row-major SymmetricMatrix move assignment (3x3)";

      ST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = std::move( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<VT,blaze::rowMajor> mat;

      ST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Row-major/row-major dense matrix assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Row-major SymmetricMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<VT,blaze::rowMajor> mat;

      ST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Row-major/row-major sparse matrix assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<VT,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<VT,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      ST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix computation assignment
   //=====================================================================================

   // Row-major/row-major dense matrix computation assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix computation assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major dense matrix computation assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix computation assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      ST sym;
      sym = eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major dense matrix computation assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix dense matrix computation assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major dense matrix computation assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix dense matrix computation assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         ST sym;
         sym = eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   // Copy assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix copy assignment (0x0)";

      OST sym1, sym2;

      sym2 = sym1;

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Copy assignment (3x3)
   {
      test_ = "Column-major SymmetricMatrix copy assignment (3x3)";

      OST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   // Move assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix move assignment (0x0)";

      OST sym1, sym2;

      sym2 = std::move( sym1 );

      checkRows    ( sym2, 0UL );
      checkColumns ( sym2, 0UL );
      checkNonZeros( sym2, 0UL );
   }

   // Move assignment (3x3)
   {
      test_ = "Column-major SymmetricMatrix move assignment (3x3)";

      OST sym1( 3UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = std::move( sym1 );

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix dense matrix assignment (0x0)";

      const blaze::DynamicMatrix<VT,blaze::columnMajor> mat;

      OST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Column-major/row-major dense matrix assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> > sym1;
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major sparse matrix assignment
   //=====================================================================================

   // Conversion assignment (0x0)
   {
      test_ = "Column-major SymmetricMatrix sparse matrix assignment (0x0)";

      const blaze::CompressedMatrix<VT,blaze::columnMajor> mat;

      OST sym;
      sym = mat;

      checkRows    ( sym, 0UL );
      checkColumns ( sym, 0UL );
      checkNonZeros( sym, 0UL );
   }

   // Column-major/row-major sparse matrix assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (symmetric)";

      blaze::CompressedMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major sparse matrix assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (non-symmetric)";

      blaze::CompressedMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, 7UL );
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<VT,blaze::rowMajor> > sym1( 3UL, 5UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major sparse matrix assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix sparse matrix assignment (SymmetricMatrix)";

      blaze::SymmetricMatrix< blaze::CompressedMatrix<VT,blaze::columnMajor> > sym1( 3UL, 5UL );
      sym1(0,0) = vec(  1 );
      sym1(0,1) = vec( -4 );
      sym1(0,2) = vec(  7 );
      sym1(1,1) = vec(  2 );
      sym1(2,2) = vec(  3 );

      OST sym2;
      sym2 = sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 7UL );

      if( sym2(0,0) != vec(  1 ) || sym2(0,1) != vec( -4 )  || sym2(0,2) != vec( 7 )   ||
          sym2(1,0) != vec( -4 ) || sym2(1,1) != vec(  2 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  7 ) || !isDefault( sym2(2,1) ) || sym2(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major computation assignment
   //=====================================================================================

   // Column-major/row-major dense matrix computation assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix computation assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major dense matrix computation assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix computation assignment (symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec(  7 );
      mat(2,2) = vec(  3 );

      OST sym;
      sym = eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 7UL );

      if( sym(0,0) != vec(  1 ) || sym(0,1) != vec( -4 )  || sym(0,2) != vec( 7 )   ||
          sym(1,0) != vec( -4 ) || sym(1,1) != vec(  2 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  7 ) || !isDefault( sym(2,1) ) || sym(2,2) != vec( 3 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  1 ) ( -4 ) ( 7 ) )\n"
                                     "( ( -4 ) (  2 ) (   ) )\n"
                                     "( (  7 ) (    ) ( 3 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major dense matrix computation assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix dense matrix computation assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::rowMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major dense matrix computation assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix dense matrix computation assignment (non-symmetric)";

      blaze::StaticMatrix<VT,3UL,3UL,blaze::columnMajor> mat;
      mat(0,0) = vec(  1 );
      mat(0,1) = vec( -4 );
      mat(0,2) = vec(  7 );
      mat(1,0) = vec( -4 );
      mat(1,1) = vec(  2 );
      mat(2,0) = vec( -5 );
      mat(2,2) = vec(  3 );

      try {
         OST sym;
         sym = eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testAddAssign()
{
   //=====================================================================================
   // Row-major addition assignment
   //=====================================================================================

   // Row-major/row-major addition assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major addition assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major addition assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major addition assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix addition assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major addition assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix addition assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major computation addition assignment
   //=====================================================================================

   // Row-major/row-major computation addition assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major computation addition assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major computation addition assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major computation addition assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major addition assignment
   //=====================================================================================

   // Column-major/row-major addition assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major addition assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major addition assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major addition assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix addition assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major addition assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix addition assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 += sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major computation addition assignment
   //=====================================================================================

   // Column-major/row-major computation addition assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major computation addition assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation addition assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym += eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major computation addition assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major computation addition assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation addition assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym += eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testSubAssign()
{
   //=====================================================================================
   // Row-major subtraction assignment
   //=====================================================================================

   // Row-major/row-major subtraction assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major subtraction assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix subtraction assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec(  2 );
      sym1(0,2) = vec( -6 );
      sym1(1,1) = vec( -3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major subtraction assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix subtraction assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec(  2 );
      sym1(0,2) = vec( -6 );
      sym1(1,1) = vec( -3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major computation computation subtraction assignment
   //=====================================================================================

   // Row-major/row-major computation subtraction assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major computation subtraction assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major computation subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major computation subtraction assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major subtraction assignment
   //=====================================================================================

   // Column-major/row-major subtraction assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major subtraction assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n( ( -1 ) (  3 ) (   ) )\n( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix subtraction assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec(  2 );
      sym1(0,2) = vec( -6 );
      sym1(1,1) = vec( -3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major subtraction assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix subtraction assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec(  2 );
      sym1(0,2) = vec( -6 );
      sym1(1,1) = vec( -3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 -= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -1 )  || sym2(0,2) != vec( 8 )   ||
          sym2(1,0) != vec( -1 )  || sym2(1,1) != vec(  3 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec(  8 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major computation subtraction assignment
   //=====================================================================================

   // Column-major/row-major computation subtraction assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major computation subtraction assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation subtraction assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,0) = vec(  2 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym -= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -1 )  || sym(0,2) != vec( 8 )   ||
          sym(1,0) != vec( -1 )  || sym(1,1) != vec(  3 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec(  8 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -1 ) ( 8 ) )\n"
                                     "( ( -1 ) (  3 ) (   ) )\n"
                                     "( (  8 ) (    ) (   ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major computation subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major computation subtraction assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation subtraction assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec(  2 );
      mat(0,2) = vec( -6 );
      mat(1,1) = vec( -3 );
      mat(2,0) = vec( -6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym -= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix Schur product assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the Schur product assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testSchurAssign()
{
   //=====================================================================================
   // Row-major Schur product assignment
   //=====================================================================================

   // Row-major/row-major Schur product assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major Schur product assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix Schur product assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -2 )  || sym2(0,2) != vec( 12 )  ||
          sym2(1,0) != vec( -2 )  || sym2(1,1) != vec(  0 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec( 12 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major Schur product assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix Schur product assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      ST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -2 )  || sym2(0,2) != vec( 12 )  ||
          sym2(1,0) != vec( -2 )  || sym2(1,1) != vec(  0 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec( 12 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major computation Schur product assignment
   //=====================================================================================

   // Row-major/row-major computation Schur product assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major computation Schur product assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major computation Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix computation Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major computation Schur product assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix computation Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major Schur product assignment
   //=====================================================================================

   // Column-major/row-major Schur product assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major Schur product assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix Schur product assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -2 )  || sym2(0,2) != vec( 12 )  ||
          sym2(1,0) != vec( -2 )  || sym2(1,1) != vec(  0 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec( 12 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major Schur product assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix Schur product assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,1) = vec( -2 );
      sym1(0,2) = vec(  6 );
      sym1(1,1) = vec(  3 );

      OST sym2( 3UL );
      sym2(0,1) = vec( 1 );
      sym2(0,2) = vec( 2 );
      sym2(1,1) = vec( 0 );

      sym2 %= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 5UL );
      checkNonZeros( sym2, 0UL, 2UL );
      checkNonZeros( sym2, 1UL, 2UL );
      checkNonZeros( sym2, 2UL, 1UL );

      if( !isDefault( sym2(0,0) ) || sym2(0,1) != vec( -2 )  || sym2(0,2) != vec( 12 )  ||
          sym2(1,0) != vec( -2 )  || sym2(1,1) != vec(  0 )  || !isDefault( sym2(1,2) ) ||
          sym2(2,0) != vec( 12 )  || !isDefault( sym2(2,1) ) || !isDefault( sym2(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major computation Schur product assignment
   //=====================================================================================

   // Column-major/row-major computation Schur product assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major computation Schur product assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation Schur product assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,0) = vec( -2 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      sym %= eval( mat );

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 5UL );
      checkNonZeros( sym, 0UL, 2UL );
      checkNonZeros( sym, 1UL, 2UL );
      checkNonZeros( sym, 2UL, 1UL );

      if( !isDefault( sym(0,0) ) || sym(0,1) != vec( -2 )  || sym(0,2) != vec( 12 )  ||
          sym(1,0) != vec( -2 )  || sym(1,1) != vec(  0 )  || !isDefault( sym(1,2) ) ||
          sym(2,0) != vec( 12 )  || !isDefault( sym(2,1) ) || !isDefault( sym(2,2) ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (    ) ( -2 ) ( 12 ) )\n"
                                     "( ( -2 ) (  0 ) (    ) )\n"
                                     "( ( 12 ) (    ) (    ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major computation Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix computation Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major computation Schur product assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix computation Schur product assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,1) = vec( 1 );
      sym(0,2) = vec( 2 );
      sym(1,1) = vec( 0 );

      try {
         sym %= eval( mat );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Schur product assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the SymmetricMatrix multiplication assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the multiplication assignment operators of the SymmetricMatrix
// specialization. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void SparseNonNumericTest::testMultAssign()
{
   //=====================================================================================
   // Row-major multiplication assignment
   //=====================================================================================

   // Row-major/row-major multiplication assignment (symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,0) = vec( 2 );
      mat(1,1) = vec( 2 );
      mat(2,2) = vec( 2 );

      ST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 9UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) != vec(  2 ) || sym(0,1) != vec( -8 ) || sym(0,2) != vec( 14 ) ||
          sym(1,0) != vec( -8 ) || sym(1,1) != vec(  4 ) || sym(1,2) != vec(  0 ) ||
          sym(2,0) != vec( 14 ) || sym(2,1) != vec(  0 ) || sym(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major multiplication assignment (symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,0) = vec( 2 );
      mat(1,1) = vec( 2 );
      mat(2,2) = vec( 2 );

      ST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 9UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) != vec(  2 ) || sym(0,1) != vec( -8 ) || sym(0,2) != vec( 14 ) ||
          sym(1,0) != vec( -8 ) || sym(1,1) != vec(  4 ) || sym(1,2) != vec(  0 ) ||
          sym(2,0) != vec( 14 ) || sym(2,1) != vec(  0 ) || sym(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/row-major multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/row-major SymmetricMatrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/column-major multiplication assignment (non-symmetric)
   {
      test_ = "Row-major/column-major SymmetricMatrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      ST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Row-major/row-major multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/row-major SymmetricMatrix multiplication assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,0) = vec( 2 );
      sym1(0,1) = vec( 0 );
      sym1(0,2) = vec( 0 );
      sym1(1,1) = vec( 2 );
      sym1(1,2) = vec( 0 );
      sym1(2,2) = vec( 2 );

      ST sym2( 3UL );
      sym2(0,0) = vec(  1 );
      sym2(0,1) = vec( -4 );
      sym2(0,2) = vec(  7 );
      sym2(1,1) = vec(  2 );
      sym2(1,2) = vec(  0 );
      sym2(2,2) = vec(  3 );

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 9UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 3UL );
      checkNonZeros( sym2, 2UL, 3UL );

      if( sym2(0,0) != vec(  2 ) || sym2(0,1) != vec( -8 ) || sym2(0,2) != vec( 14 ) ||
          sym2(1,0) != vec( -8 ) || sym2(1,1) != vec(  4 ) || sym2(1,2) != vec(  0 ) ||
          sym2(2,0) != vec( 14 ) || sym2(2,1) != vec(  0 ) || sym2(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Row-major/column-major multiplication assignment (SymmetricMatrix)
   {
      test_ = "Row-major/column-major SymmetricMatrix multiplication assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,0) = vec( 2 );
      sym1(0,1) = vec( 0 );
      sym1(0,2) = vec( 0 );
      sym1(1,1) = vec( 2 );
      sym1(1,2) = vec( 0 );
      sym1(2,2) = vec( 2 );

      ST sym2( 3UL );
      sym2(0,0) = vec(  1 );
      sym2(0,1) = vec( -4 );
      sym2(0,2) = vec(  7 );
      sym2(1,1) = vec(  2 );
      sym2(1,2) = vec(  0 );
      sym2(2,2) = vec(  3 );

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 9UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 3UL );
      checkNonZeros( sym2, 2UL, 3UL );

      if( sym2(0,0) != vec(  2 ) || sym2(0,1) != vec( -8 ) || sym2(0,2) != vec( 14 ) ||
          sym2(1,0) != vec( -8 ) || sym2(1,1) != vec(  4 ) || sym2(1,2) != vec(  0 ) ||
          sym2(2,0) != vec( 14 ) || sym2(2,1) != vec(  0 ) || sym2(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major multiplication assignment
   //=====================================================================================

   // Column-major/row-major multiplication assignment (symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,0) = vec( 2 );
      mat(1,1) = vec( 2 );
      mat(2,2) = vec( 2 );

      OST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 9UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) != vec(  2 ) || sym(0,1) != vec( -8 ) || sym(0,2) != vec( 14 ) ||
          sym(1,0) != vec( -8 ) || sym(1,1) != vec(  4 ) || sym(1,2) != vec(  0 ) ||
          sym(2,0) != vec( 14 ) || sym(2,1) != vec(  0 ) || sym(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major multiplication assignment (symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix multiplication assignment (symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,0) = vec( 2 );
      mat(1,1) = vec( 2 );
      mat(2,2) = vec( 2 );

      OST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      sym *= mat;

      checkRows    ( sym, 3UL );
      checkColumns ( sym, 3UL );
      checkNonZeros( sym, 9UL );
      checkNonZeros( sym, 0UL, 3UL );
      checkNonZeros( sym, 1UL, 3UL );
      checkNonZeros( sym, 2UL, 3UL );

      if( sym(0,0) != vec(  2 ) || sym(0,1) != vec( -8 ) || sym(0,2) != vec( 14 ) ||
          sym(1,0) != vec( -8 ) || sym(1,1) != vec(  4 ) || sym(1,2) != vec(  0 ) ||
          sym(2,0) != vec( 14 ) || sym(2,1) != vec(  0 ) || sym(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/row-major multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/row-major SymmetricMatrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::rowMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric row-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/column-major multiplication assignment (non-symmetric)
   {
      test_ = "Column-major/column-major SymmetricMatrix multiplication assignment (non-symmetric)";

      blaze::DynamicMatrix<VT,blaze::columnMajor> mat( 3UL, 3UL, vec( 0 ) );
      mat(0,1) = vec( -2 );
      mat(0,2) = vec(  6 );
      mat(1,1) = vec(  3 );
      mat(2,0) = vec(  6 );

      OST sym( 3UL );
      sym(0,0) = vec(  1 );
      sym(0,1) = vec( -4 );
      sym(0,2) = vec(  7 );
      sym(1,1) = vec(  2 );
      sym(1,2) = vec(  0 );
      sym(2,2) = vec(  3 );

      try {
         sym *= mat;

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment of non-symmetric column-major matrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << sym << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }

   // Column-major/row-major multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/row-major SymmetricMatrix multiplication assignment (SymmetricMatrix)";

      ST sym1( 3UL );
      sym1(0,0) = vec( 2 );
      sym1(0,1) = vec( 0 );
      sym1(0,2) = vec( 0 );
      sym1(1,1) = vec( 2 );
      sym1(1,2) = vec( 0 );
      sym1(2,2) = vec( 2 );

      OST sym2( 3UL );
      sym2(0,0) = vec(  1 );
      sym2(0,1) = vec( -4 );
      sym2(0,2) = vec(  7 );
      sym2(1,1) = vec(  2 );
      sym2(1,2) = vec(  0 );
      sym2(2,2) = vec(  3 );

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 9UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 3UL );
      checkNonZeros( sym2, 2UL, 3UL );

      if( sym2(0,0) != vec(  2 ) || sym2(0,1) != vec( -8 ) || sym2(0,2) != vec( 14 ) ||
          sym2(1,0) != vec( -8 ) || sym2(1,1) != vec(  4 ) || sym2(1,2) != vec(  0 ) ||
          sym2(2,0) != vec( 14 ) || sym2(2,1) != vec(  0 ) || sym2(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Column-major/column-major multiplication assignment (SymmetricMatrix)
   {
      test_ = "Column-major/column-major SymmetricMatrix multiplication assignment (SymmetricMatrix)";

      OST sym1( 3UL );
      sym1(0,0) = vec( 2 );
      sym1(0,1) = vec( 0 );
      sym1(0,2) = vec( 0 );
      sym1(1,1) = vec( 2 );
      sym1(1,2) = vec( 0 );
      sym1(2,2) = vec( 2 );

      OST sym2( 3UL );
      sym2(0,0) = vec(  1 );
      sym2(0,1) = vec( -4 );
      sym2(0,2) = vec(  7 );
      sym2(1,1) = vec(  2 );
      sym2(1,2) = vec(  0 );
      sym2(2,2) = vec(  3 );

      sym2 *= sym1;

      checkRows    ( sym2, 3UL );
      checkColumns ( sym2, 3UL );
      checkNonZeros( sym2, 9UL );
      checkNonZeros( sym2, 0UL, 3UL );
      checkNonZeros( sym2, 1UL, 3UL );
      checkNonZeros( sym2, 2UL, 3UL );

      if( sym2(0,0) != vec(  2 ) || sym2(0,1) != vec( -8 ) || sym2(0,2) != vec( 14 ) ||
          sym2(1,0) != vec( -8 ) || sym2(1,1) != vec(  4 ) || sym2(1,2) != vec(  0 ) ||
          sym2(2,0) != vec( 14 ) || sym2(2,1) != vec(  0 ) || sym2(2,2) != vec(  6 ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << sym2 << "\n"
             << "   Expected result:\n( (  2 ) ( -8 ) ( 14 ) )\n"
                                     "( ( -8 ) (  4 ) (  0 ) )\n"
                                     "( ( 14 ) (  0 ) (  6 ) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace symmetricmatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running SymmetricMatrix sparse non-numeric test (part 1)..." << std::endl;

   try
   {
      RUN_SYMMETRICMATRIX_SPARSENONNUMERIC_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during SymmetricMatrix sparse non-numeric test (part 1):\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
