//=================================================================================================
/*!
//  \file src/mathtest/traits/decldiagtrait/ClassTest.cpp
//  \brief Source file for the DeclDiagTrait class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <utility>
#include <blaze/math/CompressedMatrix.h>
#include <blaze/math/CustomMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/math/HermitianMatrix.h>
#include <blaze/math/IdentityMatrix.h>
#include <blaze/math/InitializerMatrix.h>
#include <blaze/math/LowerMatrix.h>
#include <blaze/math/StaticMatrix.h>
#include <blaze/math/StrictlyLowerMatrix.h>
#include <blaze/math/StrictlyUpperMatrix.h>
#include <blaze/math/SymmetricMatrix.h>
#include <blaze/math/traits/DeclDiagTrait.h>
#include <blaze/math/typetraits/StorageOrder.h>
#include <blaze/math/UniformMatrix.h>
#include <blaze/math/UniLowerMatrix.h>
#include <blaze/math/UniUpperMatrix.h>
#include <blaze/math/UpperMatrix.h>
#include <blaze/math/ZeroMatrix.h>
#include <blaze/util/Complex.h>
#include <blaze/util/typetraits/Decay.h>
#include <blaze/util/typetraits/IsSame.h>
#include <blazetest/mathtest/traits/decldiagtrait/ClassTest.h>


namespace blazetest {

namespace mathtest {

namespace traits {

namespace decldiagtrait {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the DeclDiagTrait class test.
//
// \exception std::runtime_error Error detected.
*/
ClassTest::ClassTest()
{
   testMatrixDeclDiag();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the 'DeclDiagTrait' class template for matrix operations.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a compile time test of the 'DeclDiagTrait' class template for matrix
// operations. In case an error is detected, a compilation error is created.
*/
void ClassTest::testMatrixDeclDiag()
{
   using namespace blaze;


   // StaticMatrix
   {
      {
         using MT = StaticMatrix<int,3UL,3UL,rowMajor>;
         using RT = DiagonalMatrix< StaticMatrix<int,3UL,3UL,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = StaticMatrix<int,3UL,3UL,columnMajor>;
         using RT = DiagonalMatrix< StaticMatrix<int,3UL,3UL,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // HybridMatrix
   {
      {
         using MT = HybridMatrix<int,3UL,3UL,rowMajor>;
         using RT = DiagonalMatrix< HybridMatrix<int,3UL,3UL,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = HybridMatrix<int,3UL,3UL,columnMajor>;
         using RT = DiagonalMatrix< HybridMatrix<int,3UL,3UL,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // DynamicMatrix
   {
      {
         using MT = DynamicMatrix<int,rowMajor>;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = DynamicMatrix<int,columnMajor>;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // CustomMatrix
   {
      {
         using MT = CustomMatrix<int,unaligned,unpadded,rowMajor>;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = CustomMatrix<int,unaligned,unpadded,columnMajor>;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // UniformMatrix
   {
      {
         using MT = UniformMatrix<int,rowMajor>;
         using RT = DiagonalMatrix< UniformMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = UniformMatrix<int,columnMajor>;
         using RT = DiagonalMatrix< UniformMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // InitializerMatrix
   {
      {
         using MT = InitializerMatrix<int>;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // CompressedMatrix
   {
      {
         using MT = CompressedMatrix<int,rowMajor>;
         using RT = DiagonalMatrix< CompressedMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = CompressedMatrix<int,columnMajor>;
         using RT = DiagonalMatrix< CompressedMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // IdentityMatrix
   {
      {
         using MT = IdentityMatrix<int,rowMajor>;
         using RT = IdentityMatrix<int,rowMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = IdentityMatrix<int,columnMajor>;
         using RT = IdentityMatrix<int,columnMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // ZeroMatrix
   {
      {
         using MT = ZeroMatrix<int,rowMajor>;
         using RT = DiagonalMatrix< ZeroMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = ZeroMatrix<int,columnMajor>;
         using RT = DiagonalMatrix< ZeroMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // SymmetricMatrix (real)
   {
      {
         using MT = SymmetricMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = SymmetricMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // SymmetricMatrix (complex)
   {
      {
         using MT = SymmetricMatrix< DynamicMatrix<complex<int>,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<complex<int>,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = SymmetricMatrix< DynamicMatrix<complex<int>,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<complex<int>,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // HermitianMatrix (symmetric)
   {
      {
         using MT = HermitianMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = HermitianMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // HermitianMatrix (Hermitian)
   {
      {
         using MT = HermitianMatrix< DynamicMatrix<complex<int>,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<complex<int>,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = HermitianMatrix< DynamicMatrix<complex<int>,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<complex<int>,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // LowerMatrix
   {
      {
         using MT = LowerMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = LowerMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // UniLowerMatrix
   {
      {
         using MT = UniLowerMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = IdentityMatrix<int,rowMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = UniLowerMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = IdentityMatrix<int,columnMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // StrictlyLowerMatrix
   {
      {
         using MT = StrictlyLowerMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = ZeroMatrix<int,rowMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = StrictlyLowerMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = ZeroMatrix<int,columnMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // UpperMatrix
   {
      {
         using MT = UpperMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = UpperMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // UniUpperMatrix
   {
      {
         using MT = UniUpperMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = IdentityMatrix<int,rowMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = UniUpperMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = IdentityMatrix<int,columnMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // StrictlyUpperMatrix
   {
      {
         using MT = StrictlyUpperMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = ZeroMatrix<int,rowMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = StrictlyUpperMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = ZeroMatrix<int,columnMajor>;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }

   // DiagonalMatrix
   {
      {
         using MT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,rowMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
      {
         using MT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         using RT = DiagonalMatrix< DynamicMatrix<int,columnMajor> >;
         static_assert( IsSame_v< DeclDiagTrait_t<MT>, RT >, "Non-matching type detected" );

         using Expr = Decay_t< decltype( decldiag( std::declval<MT>() ) ) >;
         static_assert( StorageOrder_v<Expr> == StorageOrder_v<RT>, "Non-matching storage order detected" );
      }
   }
}
//*************************************************************************************************

} // namespace decldiagtrait

} // namespace traits

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running DeclDiagTrait class test..." << std::endl;

   try
   {
      RUN_DECLDIAGTRAIT_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during DeclDiagTrait class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
