//=================================================================================================
/*!
//  \file src/mathtest/zeromatrix/ClassTest.cpp
//  \brief Source file for the ZeroMatrix class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blaze/math/CompressedMatrix.h>
#include <blaze/math/DynamicMatrix.h>
#include <blaze/util/Complex.h>
#include <blaze/util/Random.h>
#include <blazetest/mathtest/zeromatrix/ClassTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace zeromatrix {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the ZeroMatrix class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testConstructors();
   testAssignment();
   testFunctionCall();
   testAt();
   testIterator();
   testNonZeros();
   testReset();
   testClear();
   testResize();
   testSwap();
   testFind();
   testLowerBound();
   testUpperBound();
   testTranspose();
   testCTranspose();
   testIsDefault();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the ZeroMatrix constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the ZeroMatrix class template. In case
// an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testConstructors()
{
   //=====================================================================================
   // Row-major default constructor
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix default constructor";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z;

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }


   //=====================================================================================
   // Row-major size constructor
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix size constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 0UL, 0UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }

   {
      test_ = "Row-major ZeroMatrix size constructor (3x4)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major copy constructor
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix copy constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 0UL, 0UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 0UL );
      checkColumns ( Z2, 0UL );
      checkNonZeros( Z2, 0UL );
   }

   {
      test_ = "Row-major ZeroMatrix copy constructor (3x4)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 3UL, 4UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 4UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major move constructor
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix move constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 0UL, 0UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( std::move( Z1 ) );

      checkRows    ( Z2, 0UL );
      checkColumns ( Z2, 0UL );
      checkNonZeros( Z2, 0UL );
   }

   {
      test_ = "Row-major ZeroMatrix move constructor (3x4)";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 3UL, 4UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( std::move( Z1 ) );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 4UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Row-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major ZeroMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::rowMajor> Z1{ { 0, 0, 0 }, { 0, 0, 0 } };
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major ZeroMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::columnMajor> Z1{ { 0, 0, 0 }, { 0, 0, 0 } };
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major ZeroMatrix dense matrix constructor (non-zero)";

      blaze::DynamicMatrix<int,blaze::rowMajor> Z1{ { 0, 0, 0 }, { 0, 1, 0 } };

      try {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-zero ZeroMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Row-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Row-major/row-major ZeroMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> Z1( 2UL, 3UL, 2UL );
      Z1.insert( 0UL, 1UL, 0 );
      Z1.insert( 1UL, 2UL, 0 );

      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major/column-major ZeroMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> Z1( 2UL, 3UL, 2UL );
      Z1.insert( 0UL, 1UL, 0 );
      Z1.insert( 1UL, 2UL, 0 );

      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major ZeroMatrix sparse matrix constructor (non-zero)";

      blaze::CompressedMatrix<int,blaze::rowMajor> Z1{ { 0, 0, 0 }, { 0, 1, 0 } };

      try {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z2( Z1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-zero ZeroMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major default constructor
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix default constructor";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z;

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }


   //=====================================================================================
   // Column-major size constructor
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix size constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 0UL, 0UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }

   {
      test_ = "Column-major ZeroMatrix size constructor (4x3)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
          Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy constructor
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix copy constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 0UL, 0UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 0UL );
      checkColumns ( Z2, 0UL );
      checkNonZeros( Z2, 0UL );
   }

   {
      test_ = "Column-major ZeroMatrix copy constructor (4x3)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 4UL, 3UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 4UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ||
          Z2(3,0) != 0 || Z2(3,1) != 0 || Z2(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major move constructor
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix move constructor (0x0)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 0UL, 0UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( std::move( Z1 ) );

      checkRows    ( Z2, 0UL );
      checkColumns ( Z2, 0UL );
      checkNonZeros( Z2, 0UL );
   }

   {
      test_ = "Column-major ZeroMatrix move constructor (4x3)";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 4UL, 3UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( std::move( Z1 ) );

      checkRows    ( Z2, 4UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ||
          Z2(3,0) != 0 || Z2(3,1) != 0 || Z2(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major dense matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major ZeroMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::rowMajor> Z1{ { 0, 0 }, { 0, 0 }, { 0, 0 } };
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 2UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major ZeroMatrix dense matrix constructor";

      blaze::DynamicMatrix<int,blaze::columnMajor> Z1{ { 0, 0 }, { 0, 0 }, { 0, 0 } };
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 2UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major ZeroMatrix dense matrix constructor (non-zero)";

      blaze::DynamicMatrix<int,blaze::columnMajor> Z1{ { 0, 0 }, { 0, 1 }, { 0, 0 } };

      try {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-zero ZeroMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }


   //=====================================================================================
   // Column-major sparse matrix constructor
   //=====================================================================================

   {
      test_ = "Column-major/row-major ZeroMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::rowMajor> Z1( 3UL, 2UL, 2UL );
      Z1.insert( 1UL, 0UL, 0 );
      Z1.insert( 2UL, 1UL, 0 );

      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 2UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major/column-major ZeroMatrix sparse matrix constructor";

      blaze::CompressedMatrix<int,blaze::columnMajor> Z1( 3UL, 2UL, 2UL );
      Z1.insert( 1UL, 0UL, 0 );
      Z1.insert( 2UL, 1UL, 0 );

      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 2UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major ZeroMatrix sparse matrix constructor (non-zero)";

      blaze::CompressedMatrix<int,blaze::columnMajor> Z1{ { 0, 0 }, { 0, 1 }, { 0, 0 } };

      try {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z2( Z1 );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Setup of non-zero ZeroMatrix succeeded\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the ZeroMatrix assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the ZeroMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testAssignment()
{
   //=====================================================================================
   // Row-major copy assignment
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix copy assignment";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 3UL, 4UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2;
      Z2 = Z1;

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 4UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 || Z2(0,3) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 || Z2(1,3) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 || Z2(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major ZeroMatrix copy assignment stress test";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1;

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t m( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t n( blaze::rand<size_t>( 0UL, 10UL ) );
         const blaze::ZeroMatrix<int,blaze::rowMajor> Z2( m, n );

         Z1 = Z2;

         if( Z1 != Z2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << Z1 << "\n"
                << "   Expected result:\n" << Z2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Row-major move assignment
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix move assignment";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 3UL, 4UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2;

      Z2 = std::move( Z1 );

      checkRows    ( Z2, 3UL );
      checkColumns ( Z2, 4UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 || Z2(0,3) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 || Z2(1,3) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 || Z2(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major copy assignment
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix copy assignment";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 4UL, 3UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2;
      Z2 = Z1;

      checkRows    ( Z2, 4UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ||
          Z2(3,0) != 0 || Z2(3,1) != 0 || Z2(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major ZeroMatrix copy assignment stress test";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1;

      for( size_t i=0UL; i<100UL; ++i )
      {
         const size_t m( blaze::rand<size_t>( 0UL, 10UL ) );
         const size_t n( blaze::rand<size_t>( 0UL, 10UL ) );
         const blaze::ZeroMatrix<int,blaze::columnMajor> Z2( m, n );

         Z1 = Z2;

         if( Z1 != Z2 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Assignment failed\n"
                << " Details:\n"
                << "   Result:\n" << Z1 << "\n"
                << "   Expected result:\n" << Z2 << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major move assignment
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix move assignment";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 4UL, 3UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2;

      Z2 = std::move( Z1 );

      checkRows    ( Z2, 4UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ||
          Z2(2,0) != 0 || Z2(2,1) != 0 || Z2(2,2) != 0 ||
          Z2(3,0) != 0 || Z2(3,1) != 0 || Z2(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the ZeroMatrix function call operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the function call operator
// of the ZeroMatrix class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testFunctionCall()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::operator()";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::operator()";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
          Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Function call operator failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c at() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the \c at() member function
// of the ZeroMatrix class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void ClassTest::testAt()
{
   //=====================================================================================
   // Row-major matrix tests
   //==========≈===========================================================================

   {
      test_ = "Row-major ZeroMatrix::at()";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z.at(0,0) != 0 || Z.at(0,1) != 0 || Z.at(0,2) != 0 || Z.at(0,3) != 0 ||
          Z.at(1,0) != 0 || Z.at(1,1) != 0 || Z.at(1,2) != 0 || Z.at(1,3) != 0 ||
          Z.at(2,0) != 0 || Z.at(2,1) != 0 || Z.at(2,2) != 0 || Z.at(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Access via at() function failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::at()";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z.at(0,0) != 0 || Z.at(0,1) != 0 || Z.at(0,2) != 0 ||
          Z.at(1,0) != 0 || Z.at(1,1) != 0 || Z.at(1,2) != 0 ||
          Z.at(2,0) != 0 || Z.at(2,1) != 0 || Z.at(2,2) != 0 ||
          Z.at(3,0) != 0 || Z.at(3,1) != 0 || Z.at(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Access via at() function failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the ZeroMatrix iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the ZeroMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testIterator()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      using MatrixType    = blaze::ZeroMatrix<int,blaze::rowMajor>;
      using ConstIterator = MatrixType::ConstIterator;

      MatrixType Z( 3UL, 4UL );

      // Testing the ConstIterator default constructor
      {
         test_ = "Row-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (end-begin)
      {
         test_ = "Row-major ConstIterator subtraction";

         const ptrdiff_t number( cend( Z, 1UL ) - cbegin( Z, 1UL ) );

         if( number != 0L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing ConstIterator comparison
      {
         test_ = "Row-major ConstIterator comparison";

         ConstIterator it ( cbegin( Z, 1UL ) );
         ConstIterator end( cend( Z, 1UL ) );

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator comparison failed\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      using MatrixType    = blaze::ZeroMatrix<int,blaze::columnMajor>;
      using ConstIterator = MatrixType::ConstIterator;

      MatrixType Z( 4UL, 3UL );

      // Testing the ConstIterator default constructor
      {
         test_ = "Column-major ConstIterator default constructor";

         ConstIterator it{};

         if( it != ConstIterator() ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Failed iterator default constructor\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Counting the number of elements in 1st row via ConstIterator (end-begin)
      {
         test_ = "Column-major ConstIterator subtraction";

         const ptrdiff_t number( cend( Z, 1UL ) - cbegin( Z, 1UL ) );

         if( number != 0L ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid number of elements detected\n"
                << " Details:\n"
                << "   Number of elements         : " << number << "\n"
                << "   Expected number of elements: 0\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Testing ConstIterator comparison
      {
         test_ = "Column-major ConstIterator comparison";

         ConstIterator it ( cbegin( Z, 1UL ) );
         ConstIterator end( cend( Z, 1UL ) );

         if( it != end ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Iterator comparison failed\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testNonZeros()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::nonZeros()";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 6UL, 8UL );

      checkRows    ( Z, 6UL );
      checkColumns ( Z, 8UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );
      checkNonZeros( Z, 4UL, 0UL );
      checkNonZeros( Z, 5UL, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::nonZeros()";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 8UL, 6UL );

      checkRows    ( Z, 8UL );
      checkColumns ( Z, 6UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );
      checkNonZeros( Z, 4UL, 0UL );
      checkNonZeros( Z, 5UL, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reset() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reset() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testReset()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::reset()";

      // Resetting a default constructed matrix
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z;

         reset( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }

      // Resetting an initialized matrix
      {
         // Initialization check
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

         checkRows    ( Z, 3UL );
         checkColumns ( Z, 4UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Resetting row 1
         reset( Z, 1UL );

         checkRows    ( Z, 3UL );
         checkColumns ( Z, 4UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Reset operation failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Resetting the entire matrix
         reset( Z );

         checkRows    ( Z, 3UL );
         checkColumns ( Z, 4UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Reset operation failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::reset()";

      // Resetting a default constructed matrix
      {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z;

         reset( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }

      // Resetting an initialized matrix
      {
         // Initialization check
         blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

         checkRows    ( Z, 4UL );
         checkColumns ( Z, 3UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
             Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Resetting column 1
         reset( Z, 1UL );

         checkRows    ( Z, 4UL );
         checkColumns ( Z, 3UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
             Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Reset operation failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Resetting the entire matrix
         reset( Z );

         checkRows    ( Z, 4UL );
         checkColumns ( Z, 3UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
             Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Reset operation failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c clear() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c clear() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testClear()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::clear()";

      // Clearing a default constructed matrix
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z;

         clear( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }

      // Clearing an initialized matrix
      {
         // Initialization check
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

         checkRows    ( Z, 3UL );
         checkColumns ( Z, 4UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Initialization failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Clearing the matrix
         clear( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::clear()";

      // Clearing a default constructed matrix
      {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z;

         clear( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }

      // Clearing an initialized matrix
      {
         // Initialization check
         blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

         checkRows    ( Z, 4UL );
         checkColumns ( Z, 3UL );
         checkNonZeros( Z, 0UL );
         checkNonZeros( Z, 0UL, 0UL );
         checkNonZeros( Z, 1UL, 0UL );
         checkNonZeros( Z, 2UL, 0UL );

         if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
             Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
             Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
             Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Clear operation failed\n"
                << " Details:\n"
                << "   Result:\n" << Z << "\n"
                << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
            throw std::runtime_error( oss.str() );
         }

         // Clearing the matrix
         clear( Z );

         checkRows    ( Z, 0UL );
         checkColumns ( Z, 0UL );
         checkNonZeros( Z, 0UL );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c resize() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c resize() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testResize()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::resize()";

      // Initialization check
      blaze::ZeroMatrix<int,blaze::rowMajor> Z;

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 0x3
      Z.resize( 0UL, 3UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 5x0
      Z.resize( 5UL, 0UL );

      checkRows    ( Z, 5UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 3x4
      Z.resize( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Resizing to 2x1
      Z.resize( 2UL, 1UL );

      checkRows    ( Z, 2UL );
      checkColumns ( Z, 1UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );

      // Resizing to 3x2
      Z.resize( 3UL, 2UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 2UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Resizing to 2x2
      Z.resize( 2UL, 2UL );

      checkRows    ( Z, 2UL );
      checkColumns ( Z, 2UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );

      // Resizing to 0x0
      Z.resize( 0UL, 0UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::resize()";

      // Initialization check
      blaze::ZeroMatrix<int,blaze::columnMajor> Z;

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 0x3
      Z.resize( 0UL, 3UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 5x0
      Z.resize( 5UL, 0UL );

      checkRows    ( Z, 5UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );

      // Resizing to 4x3
      Z.resize( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Resizing to 1x2
      Z.resize( 1UL, 2UL );

      checkRows    ( Z, 1UL );
      checkColumns ( Z, 2UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );

      // Resizing to 2x3
      Z.resize( 2UL, 3UL );

      checkRows    ( Z, 2UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Resizing to 2x2
      Z.resize( 2UL, 2UL );

      checkRows    ( Z, 2UL );
      checkColumns ( Z, 2UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );

      // Resizing to 0x0
      Z.resize( 0UL, 0UL );

      checkRows    ( Z, 0UL );
      checkColumns ( Z, 0UL );
      checkNonZeros( Z, 0UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the ZeroMatrix class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testSwap()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix swap";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z1( 2UL, 3UL );
      blaze::ZeroMatrix<int,blaze::rowMajor> Z2( 3UL, 2UL );

      swap( Z1, Z2 );

      checkRows    ( Z1, 3UL );
      checkColumns ( Z1, 2UL );
      checkNonZeros( Z1, 0UL );
      checkNonZeros( Z1, 0UL, 0UL );
      checkNonZeros( Z1, 1UL, 0UL );
      checkNonZeros( Z1, 2UL, 0UL );

      if( Z1(0,0) != 0 || Z1(0,1) != 0 ||
          Z1(1,0) != 0 || Z1(1,1) != 0 ||
          Z1(2,0) != 0 || Z1(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << Z1 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix swap";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z1( 2UL, 3UL );
      blaze::ZeroMatrix<int,blaze::columnMajor> Z2( 3UL, 2UL );

      swap( Z1, Z2 );

      checkRows    ( Z1, 3UL );
      checkColumns ( Z1, 2UL );
      checkNonZeros( Z1, 0UL );
      checkNonZeros( Z1, 0UL, 0UL );
      checkNonZeros( Z1, 1UL, 0UL );

      if( Z1(0,0) != 0 || Z1(0,1) != 0 ||
          Z1(1,0) != 0 || Z1(1,1) != 0 ||
          Z1(2,0) != 0 || Z1(2,1) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the first matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << Z1 << "\n"
             << "   Expected result:\n( 0 0 )\n( 0 0 )\n( 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }

      checkRows    ( Z2, 2UL );
      checkColumns ( Z2, 3UL );
      checkNonZeros( Z2, 0UL );
      checkNonZeros( Z2, 0UL, 0UL );
      checkNonZeros( Z2, 1UL, 0UL );
      checkNonZeros( Z2, 2UL, 0UL );

      if( Z2(0,0) != 0 || Z2(0,1) != 0 || Z2(0,2) != 0 ||
          Z2(1,0) != 0 || Z2(1,1) != 0 || Z2(1,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Swapping the second matrix failed\n"
             << " Details:\n"
             << "   Result:\n" << Z2 << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c find() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c find() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testFind()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::find()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::rowMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 6UL, 8UL );

      checkRows    ( Z, 6UL );
      checkColumns ( Z, 8UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );
      checkNonZeros( Z, 4UL, 0UL );
      checkNonZeros( Z, 5UL, 0UL );

      // Searching for the first non-existing element
      {
         ConstIterator pos( Z.find( 0UL, 0UL ) );

         if( pos != Z.end( 0UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 0\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Searching for the second non-existing element
      {
         ConstIterator pos( Z.find( 2UL, 4UL ) );

         if( pos != Z.end( 2UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 4\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Searching for the third non-existing element
      {
         ConstIterator pos( Z.find( 5UL, 7UL ) );

         if( pos != Z.end( 5UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 7\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::find()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::columnMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 8UL, 6UL );

      checkRows    ( Z, 8UL );
      checkColumns ( Z, 6UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );
      checkNonZeros( Z, 4UL, 0UL );
      checkNonZeros( Z, 5UL, 0UL );

      // Searching for the first non-existing element
      {
         ConstIterator pos( Z.find( 0UL, 0UL ) );

         if( pos != Z.end( 0UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 0\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Searching for the second non-existing element
      {
         ConstIterator pos( Z.find( 4UL, 2UL ) );

         if( pos != Z.end( 2UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 4\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Searching for the third non-existing element
      {
         ConstIterator pos( Z.find( 7UL, 5UL ) );

         if( pos != Z.end( 5UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Non-existing element could be found\n"
                << " Details:\n"
                << "   Required index = 7\n"
                << "   Found index    = " << pos->index() << "\n"
                << "   Expected value = 0\n"
                << "   Value at index = " << pos->value() << "\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c lowerBound() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c lowerBound() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testLowerBound()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::lowerBound()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::rowMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Determining the lower bound for position (1,0)
      {
         ConstIterator pos( Z.lowerBound( 1UL, 0UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,0)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the lower bound for position (1,1)
      {
         ConstIterator pos( Z.lowerBound( 1UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the lower bound for position (1,2)
      {
         ConstIterator pos( Z.lowerBound( 1UL, 2UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,2)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::lowerBound()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::columnMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Determining the lower bound for position (0,1)
      {
         ConstIterator pos( Z.lowerBound( 0UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,0)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the lower bound for position (1,1)
      {
         ConstIterator pos( Z.lowerBound( 1UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the lower bound for position (2,1)
      {
         ConstIterator pos( Z.lowerBound( 2UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Lower bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (2,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c upperBound() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c upperBound() member function of the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testUpperBound()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major ZeroMatrix::upperBound()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::rowMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Determining the upper bound for position (1,0)
      {
         ConstIterator pos( Z.upperBound( 1UL, 0UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,0)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the upper bound for position (1,1)
      {
         ConstIterator pos( Z.upperBound( 1UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the upper bound for position (1,2)
      {
         ConstIterator pos( Z.upperBound( 1UL, 2UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,2)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major ZeroMatrix::upperBound()";

      using ConstIterator = blaze::ZeroMatrix<int,blaze::columnMajor>::ConstIterator;

      // Initialization check
      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      // Determining the upper bound for position (0,1)
      {
         ConstIterator pos( Z.upperBound( 0UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (0,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the upper bound for position (1,1)
      {
         ConstIterator pos( Z.upperBound( 1UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (1,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // Determining the upper bound for position (2,1)
      {
         ConstIterator pos( Z.upperBound( 2UL, 1UL ) );

         if( pos != Z.end( 1UL ) ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Upper bound could not be determined\n"
                << " Details:\n"
                << "   Required position = (2,1)\n"
                << "   Current matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c transpose() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c transpose() member function of the ZeroMatrix
// class template. Additionally, it performs a test of self-transpose via the \c trans()
// function. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testTranspose()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major self-transpose via transpose()";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      transpose( Z );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
          Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major self-transpose via trans()";

      blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

      Z = trans( Z );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 ||
          Z(3,0) != 0 || Z(3,1) != 0 || Z(3,2) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n( 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major self-transpose via transpose()";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      transpose( Z );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major self-transpose via trans()";

      blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

      Z = trans( Z );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != 0 || Z(0,1) != 0 || Z(0,2) != 0 || Z(0,3) != 0 ||
          Z(1,0) != 0 || Z(1,1) != 0 || Z(1,2) != 0 || Z(1,3) != 0 ||
          Z(2,0) != 0 || Z(2,1) != 0 || Z(2,2) != 0 || Z(2,3) != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n( 0 0 0 0 )\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c ctranspose() member function of the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c ctranspose() member function of the ZeroMatrix
// class template. Additionally, it performs a test of self-transpose via the \c ctrans()
// function. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testCTranspose()
{
   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major self-transpose via ctranspose()";

      using cplx = blaze::complex<int>;

      blaze::ZeroMatrix<cplx,blaze::rowMajor> Z( 3UL, 4UL );

      ctranspose( Z );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != cplx(0,0) || Z(0,1) != cplx(0,0) || Z(0,2) != cplx(0,0) ||
          Z(1,0) != cplx(0,0) || Z(1,1) != cplx(0,0) || Z(1,2) != cplx(0,0) ||
          Z(2,0) != cplx(0,0) || Z(2,1) != cplx(0,0) || Z(2,2) != cplx(0,0) ||
          Z(3,0) != cplx(0,0) || Z(3,1) != cplx(0,0) || Z(3,2) != cplx(0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Row-major self-transpose via ctranspose()";

      using cplx = blaze::complex<int>;

      blaze::ZeroMatrix<cplx,blaze::rowMajor> Z( 3UL, 4UL );

      Z = ctrans( Z );

      checkRows    ( Z, 4UL );
      checkColumns ( Z, 3UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 3UL, 0UL );

      if( Z(0,0) != cplx(0,0) || Z(0,1) != cplx(0,0) || Z(0,2) != cplx(0,0) ||
          Z(1,0) != cplx(0,0) || Z(1,1) != cplx(0,0) || Z(1,2) != cplx(0,0) ||
          Z(2,0) != cplx(0,0) || Z(2,1) != cplx(0,0) || Z(2,2) != cplx(0,0) ||
          Z(3,0) != cplx(0,0) || Z(3,1) != cplx(0,0) || Z(3,2) != cplx(0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( (1,0) (0,0) (0,0) )\n"
                                     "( (0,0) (1,0) (0,0) )\n"
                                     "( (0,0) (0,0) (1,0) )\n"
                                     "( (0,0) (0,0) (0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major self-transpose via ctranspose()";

      using cplx = blaze::complex<int>;

      blaze::ZeroMatrix<cplx,blaze::columnMajor> Z( 4UL, 3UL );

      ctranspose( Z );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != cplx(0,0) || Z(0,1) != cplx(0,0) || Z(0,2) != cplx(0,0) || Z(0,3) != cplx(0,0) ||
          Z(1,0) != cplx(0,0) || Z(1,1) != cplx(0,0) || Z(1,2) != cplx(0,0) || Z(1,3) != cplx(0,0) ||
          Z(2,0) != cplx(0,0) || Z(2,1) != cplx(0,0) || Z(2,2) != cplx(0,0) || Z(2,3) != cplx(0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( (0,0) (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) (0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "Column-major self-transpose via ctranspose()";

      using cplx = blaze::complex<int>;

      blaze::ZeroMatrix<cplx,blaze::columnMajor> Z( 4UL, 3UL );

      Z = ctrans( Z );

      checkRows    ( Z, 3UL );
      checkColumns ( Z, 4UL );
      checkNonZeros( Z, 0UL );
      checkNonZeros( Z, 0UL, 0UL );
      checkNonZeros( Z, 1UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );
      checkNonZeros( Z, 2UL, 0UL );

      if( Z(0,0) != cplx(0,0) || Z(0,1) != cplx(0,0) || Z(0,2) != cplx(0,0) || Z(0,3) != cplx(0,0) ||
          Z(1,0) != cplx(0,0) || Z(1,1) != cplx(0,0) || Z(1,2) != cplx(0,0) || Z(1,3) != cplx(0,0) ||
          Z(2,0) != cplx(0,0) || Z(2,1) != cplx(0,0) || Z(2,2) != cplx(0,0) || Z(2,3) != cplx(0,0) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Transposition failed\n"
             << " Details:\n"
             << "   Result:\n" << Z << "\n"
             << "   Expected result:\n( (0,0) (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) (0,0) )\n"
                                     "( (0,0) (0,0) (0,0) (0,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isDefault() function with the ZeroMatrix class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isDefault() function with the ZeroMatrix class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void ClassTest::testIsDefault()
{
   using blaze::isDefault;


   //=====================================================================================
   // Row-major matrix tests
   //=====================================================================================

   {
      test_ = "Row-major isDefault() function";

      // isDefault with 0x0 matrix (default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z;

         if( isDefault( Z ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 0x4 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 0UL, 4UL );

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 3x0 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 0UL );

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 3x4 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 3UL, 4UL );

         if( isDefault( Z(0,1) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element: " << Z(0,1) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }


   //=====================================================================================
   // Column-major matrix tests
   //=====================================================================================

   {
      test_ = "Column-major isDefault() function";

      // isDefault with 0x0 matrix (default)
      {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z;

         if( isDefault( Z ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 0x3 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 0UL, 3UL );

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 4x0 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::rowMajor> Z( 4UL, 0UL );

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }

      // isDefault with 4x3 matrix (non-default)
      {
         blaze::ZeroMatrix<int,blaze::columnMajor> Z( 4UL, 3UL );

         if( isDefault( Z(1,0) ) != true ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix element: " << Z(1,0) << "\n";
            throw std::runtime_error( oss.str() );
         }

         if( isDefault( Z ) != false ) {
            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Invalid isDefault evaluation\n"
                << " Details:\n"
                << "   Matrix:\n" << Z << "\n";
            throw std::runtime_error( oss.str() );
         }
      }
   }
}
//*************************************************************************************************

} // namespace zeromatrix

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running ZeroMatrix class test..." << std::endl;

   try
   {
      RUN_ZEROMATRIX_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during ZeroMatrix class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
