//=================================================================================================
/*!
//  \file src/mathtest/customvector/AlignedUnpaddedTest.cpp
//  \brief Source file for the aligned/unpadded CustomVector class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <memory>
#include <blaze/math/CompressedVector.h>
#include <blaze/math/shims/Equal.h>
#include <blaze/util/Complex.h>
#include <blaze/util/Memory.h>
#include <blaze/util/policies/Deallocate.h>
#include <blaze/util/Random.h>
#include <blaze/util/typetraits/AlignmentOf.h>
#include <blazetest/mathtest/customvector/AlignedUnpaddedTest.h>
#include <blazetest/mathtest/RandomMaximum.h>
#include <blazetest/mathtest/RandomMinimum.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace customvector {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aligned/unpadded CustomVector class test.
//
// \exception std::runtime_error Operation error detected.
*/
AlignedUnpaddedTest::AlignedUnpaddedTest()
{
   testConstructors();
   testAssignment();
   testAddAssign();
   testSubAssign();
   testMultAssign();
   testDivAssign();
   testCrossAssign();
   testScaling();
   testSubscript();
   testAt();
   testIterator();
   testNonZeros();
   testReset();
   testClear();
   testSwap();
   testIsDefault();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the CustomVector constructors.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all constructors of the CustomVector class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testConstructors()
{
   //=====================================================================================
   // Default constructor
   //=====================================================================================

   {
      test_ = "CustomVector default constructor";

      VT vec;

      checkSize    ( vec, 0UL );
      checkNonZeros( vec, 0UL );
   }


   //=====================================================================================
   // Constructor ( Type*, size_t )
   //=====================================================================================

   {
      test_ = "CustomVector constructor ( Type*, size_t )";

      // Constructing a custom vector of size 10
      {
         std::unique_ptr<int[],blaze::Deallocate> array( blaze::allocate<int>( 10UL ) );
         VT vec( array.get(), 10UL );

         checkSize    ( vec, 10UL );
         checkCapacity( vec, 10UL );
      }

      // Trying to construct a custom vector with invalid array of elements
      try {
         VT vec( nullptr, 0UL );

         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Constructing a custom vector with a nullptr succeeded\n";
         throw std::runtime_error( oss.str() );
      }
      catch( std::invalid_argument& ) {}

      // Trying to construct a custom vector with invalid alignment
      if( blaze::AlignmentOf<int>::value > sizeof(int) )
      {
         try {
            std::unique_ptr<int[],blaze::Deallocate> array( blaze::allocate<int>( 5UL ) );
            VT vec( array.get()+1UL, 4UL );

            std::ostringstream oss;
            oss << " Test: " << test_ << "\n"
                << " Error: Constructing a custom vector with invalid alignment succeeded\n"
                << " Details:\n"
                << "   Result:\n" << vec << "\n";
            throw std::runtime_error( oss.str() );
         }
         catch( std::invalid_argument& ) {}
      }
   }


   //=====================================================================================
   // Copy constructor
   //=====================================================================================

   {
      test_ = "CustomVector copy constructor (size 0)";

      VT vec1;
      VT vec2( vec1 );

      checkSize    ( vec2, 0UL );
      checkNonZeros( vec2, 0UL );
   }

   {
      test_ = "CustomVector copy constructor (size 5)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec1( memory.get(), 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      VT vec2( vec1 );

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec1.data() != vec2.data() ||
          vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Move constructor
   //=====================================================================================

   {
      test_ = "CustomVector move constructor (size 0)";

      VT vec1;
      VT vec2( std::move( vec1 ) );

      checkSize    ( vec2, 0UL );
      checkNonZeros( vec2, 0UL );
   }

   {
      test_ = "CustomVector move constructor (size 5)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec1( memory.get(), 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      VT vec2( std::move( vec1 ) );

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Construction failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all assignment operators of the CustomVector class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testAssignment()
{
   //=====================================================================================
   // Homogeneous assignment
   //=====================================================================================

   {
      test_ = "CustomVector homogeneous assignment";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 3UL ) );
      VT vec( memory.get(), 3UL );
      vec = 2;

      checkSize    ( vec, 3UL );
      checkCapacity( vec, 3UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 2 || vec[1] != 2 || vec[2] != 2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 2 2 2 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // List assignment
   //=====================================================================================

   {
      test_ = "CustomVector initializer list assignment (complete list)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec = { 1, 2, 3, 4 };

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector initializer list assignment (incomplete list)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec = { 1, 2 };

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 2UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Array assignment
   //=====================================================================================

   {
      test_ = "CustomVector array assignment";

      const int array[4] = { 1, 2, 3, 4 };
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec = array;

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Copy assignment
   //=====================================================================================

   {
      test_ = "CustomVector copy assignment";

      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 5UL ) );
      VT vec1( memory1.get(), 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2 = vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Move assignment
   //=====================================================================================

   {
      test_ = "CustomVector move assignment";

      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 5UL ) );
      VT vec1( memory1.get(), 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2 = std::move( vec1 );

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Dense vector assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 5UL ) );
      AlignedUnpadded vec1( memory1.get(), 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2 = vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 5UL, 16UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2 = vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[6UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 5UL );
      vec1[0] = 1;
      vec1[1] = 2;
      vec1[2] = 3;
      vec1[3] = 4;
      vec1[4] = 5;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2 = vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 5UL );

      if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 || vec2[3] != 4 || vec2[4] != 5 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 2 3 4 5 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Sparse vector assignment
   //=====================================================================================

   {
      test_ = "CustomVector sparse vector assignment";

      blaze::CompressedVector<int,blaze::rowVector> vec1( 5UL );
      vec1[0] = 1;
      vec1[2] = 2;
      vec1[3] = 3;

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec2( memory.get(), 5UL );
      vec2 = vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 3UL );

      if( vec2[0] != 1 || vec2[1] != 0 || vec2[2] != 2 || vec2[3] != 3 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 0 2 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector addition assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the addition assignment operators of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testAddAssign()
{
   //=====================================================================================
   // Dense vector addition assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector addition assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 5UL ) );
      AlignedUnpadded vec1( memory1.get(), 5UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 += vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector addition assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 5UL, 16UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 += vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector addition assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[6UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 5UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 += vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Sparse vector addition assignment
   //=====================================================================================

   {
      test_ = "CustomVector sparse vector addition assignment";

      blaze::CompressedVector<int,blaze::rowVector> vec1( 5UL, 3UL );
      vec1[0] =  1;
      vec1[2] = -2;
      vec1[3] =  3;

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec2( memory.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 += vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector subtraction assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the subtraction assignment operators of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testSubAssign()
{
   //=====================================================================================
   // Dense vector subtraction assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector subtraction assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 5UL ) );
      AlignedUnpadded vec1( memory1.get(), 5UL );
      vec1[0] = -1;
      vec1[1] =  0;
      vec1[2] =  2;
      vec1[3] = -3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 -= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector subtraction assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 5UL, 16UL );
      vec1[0] = -1;
      vec1[1] =  0;
      vec1[2] =  2;
      vec1[3] = -3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 -= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector subtraction assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[6UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 5UL );
      vec1[0] = -1;
      vec1[1] =  0;
      vec1[2] =  2;
      vec1[3] = -3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 -= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Sparse vector subtraction assignment
   //=====================================================================================

   {
      test_ = "CustomVector sparse vector subtraction assignment";

      blaze::CompressedVector<int,blaze::rowVector> vec1( 5UL, 3UL );
      vec1[0] = -1;
      vec1[2] =  2;
      vec1[3] = -3;

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec2( memory.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 -= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 4UL );

      if( vec2[0] != 1 || vec2[1] != 4 || vec2[2] != 0 || vec2[3] != -3 || vec2[4] != 7 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 1 4 0 -3 7 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector multiplication assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the multiplication assignment operators of the CustomVector
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testMultAssign()
{
   //=====================================================================================
   // Dense vector multiplication assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector multiplication assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 5UL ) );
      AlignedUnpadded vec1( memory1.get(), 5UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 *= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 2UL );

      if( vec2[0] != 0 || vec2[1] != 0 || vec2[2] != -4 || vec2[3] != -18 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 0 -4 -18 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector multiplication assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 5UL, 16UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 *= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 2UL );

      if( vec2[0] != 0 || vec2[1] != 0 || vec2[2] != -4 || vec2[3] != -18 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 0 -4 -18 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector multiplication assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[6UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 5UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;
      vec1[3] =  3;
      vec1[4] =  0;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 *= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 2UL );

      if( vec2[0] != 0 || vec2[1] != 0 || vec2[2] != -4 || vec2[3] != -18 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 0 -4 -18 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Sparse vector multiplication assignment
   //=====================================================================================

   {
      test_ = "CustomVector sparse vector multiplication assignment";

      blaze::CompressedVector<int,blaze::rowVector> vec1( 5UL, 3UL );
      vec1[0] =  1;
      vec1[2] = -2;
      vec1[3] =  3;

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec2( memory.get(), 5UL );
      vec2[0] =  0;
      vec2[1] =  4;
      vec2[2] =  2;
      vec2[3] = -6;
      vec2[4] =  7;

      vec2 *= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 2UL );

      if( vec2[0] != 0 || vec2[1] != 0 || vec2[2] != -4 || vec2[3] != -18 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 0 -4 -18 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector division assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the division assignment operators of the CustomVector
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testDivAssign()
{
   //=====================================================================================
   // Dense vector division assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector division assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 5UL ) );
      AlignedUnpadded vec1( memory1.get(), 5UL );
      vec1[0] =  1;
      vec1[1] =  2;
      vec1[2] = -3;
      vec1[3] =  4;
      vec1[4] =  1;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -3;
      vec2[3] =  8;
      vec2[4] =  0;

      vec2 /= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 3UL );

      if( vec2[0] != 2 || vec2[1] != 0 || vec2[2] != 1 || vec2[3] != 2 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Division assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 2 0 1 2 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector division assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 5UL, 16UL );
      vec1[0] =  1;
      vec1[1] =  2;
      vec1[2] = -3;
      vec1[3] =  4;
      vec1[4] =  1;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -3;
      vec2[3] =  8;
      vec2[4] =  0;

      vec2 /= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 3UL );

      if( vec2[0] != 2 || vec2[1] != 0 || vec2[2] != 1 || vec2[3] != 2 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Division assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 2 0 1 2 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector division assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[6UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 5UL );
      vec1[0] =  1;
      vec1[1] =  2;
      vec1[2] = -3;
      vec1[3] =  4;
      vec1[4] =  1;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 5UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -3;
      vec2[3] =  8;
      vec2[4] =  0;

      vec2 /= vec1;

      checkSize    ( vec2, 5UL );
      checkCapacity( vec2, 5UL );
      checkNonZeros( vec2, 3UL );

      if( vec2[0] != 2 || vec2[1] != 0 || vec2[2] != 1 || vec2[3] != 2 || vec2[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Division assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 2 0 1 2 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector cross product assignment operators.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the cross product assignment operators of the CustomVector
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testCrossAssign()
{
   //=====================================================================================
   // Dense vector cross product assignment
   //=====================================================================================

   {
      test_ = "CustomVector dense vector cross product assignment (mixed type)";

      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using AlignedUnpadded = blaze::CustomVector<short,aligned,unpadded,rowVector>;
      std::unique_ptr<short[],blaze::Deallocate> memory1( blaze::allocate<short>( 3UL ) );
      AlignedUnpadded vec1( memory1.get(), 3UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 3UL ) );
      VT vec2( memory2.get(), 3UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -1;

      vec2 %= vec1;

      checkSize    ( vec2, 3UL );
      checkCapacity( vec2, 3UL );
      checkNonZeros( vec2, 1UL );

      if( vec2[0] != 0 || vec2[1] != 3 || vec2[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Cross product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector cross product assignment (aligned/padded)";

      using blaze::aligned;
      using blaze::padded;
      using blaze::rowVector;

      using AlignedPadded = blaze::CustomVector<int,aligned,padded,rowVector>;
      std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 16UL ) );
      AlignedPadded vec1( memory1.get(), 3UL, 16UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 5UL ) );
      VT vec2( memory2.get(), 3UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -1;

      vec2 %= vec1;

      checkSize    ( vec2, 3UL );
      checkCapacity( vec2, 3UL );
      checkNonZeros( vec2, 1UL );

      if( vec2[0] != 0 || vec2[1] != 3 || vec2[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Cross product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector dense vector cross product assignment (unaligned/unpadded)";

      using blaze::unaligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using UnalignedUnpadded = blaze::CustomVector<int,unaligned,unpadded,rowVector>;
      std::unique_ptr<int[]> memory1( new int[4UL] );
      UnalignedUnpadded vec1( memory1.get()+1UL, 3UL );
      vec1[0] =  1;
      vec1[1] =  0;
      vec1[2] = -2;

      std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 3UL ) );
      VT vec2( memory2.get(), 3UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -1;

      vec2 %= vec1;

      checkSize    ( vec2, 3UL );
      checkCapacity( vec2, 3UL );
      checkNonZeros( vec2, 1UL );

      if( vec2[0] != 0 || vec2[1] != 3 || vec2[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Cross product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Sparse vector cross product assignment
   //=====================================================================================

   {
      test_ = "CustomVector sparse vector cross product assignment";

      blaze::CompressedVector<int,blaze::rowVector> vec1( 3UL, 2UL );
      vec1[0] =  1;
      vec1[2] = -2;

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 3UL ) );
      VT vec2( memory.get(), 3UL );
      vec2[0] =  2;
      vec2[1] =  0;
      vec2[2] = -1;

      vec2 %= vec1;

      checkSize    ( vec2, 3UL );
      checkCapacity( vec2, 3UL );
      checkNonZeros( vec2, 1UL );

      if( vec2[0] != 0 || vec2[1] != 3 || vec2[2] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Cross product assignment failed\n"
             << " Details:\n"
             << "   Result:\n" << vec2 << "\n"
             << "   Expected result:\n( 0 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of all CustomVector (self-)scaling operations.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of all available ways to scale an instance of the CustomVector
// class template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testScaling()
{
   //=====================================================================================
   // Self-scaling (v*=s)
   //=====================================================================================

   {
      test_ = "CustomVector self-scaling (v*=s)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec( memory.get(), 5UL );
      vec[0] =  1;
      vec[1] =  0;
      vec[2] = -2;
      vec[3] =  3;
      vec[4] =  0;

      vec *= 2;

      checkSize    ( vec, 5UL );
      checkCapacity( vec, 5UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 2 || vec[1] != 0 || vec[2] != -4 || vec[3] != 6 || vec[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 2 0 -4 6 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Self-scaling (v=v*s)
   //=====================================================================================

   {
      test_ = "CustomVector self-scaling (v=v*s)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec( memory.get(), 5UL );
      vec[0] =  1;
      vec[1] =  0;
      vec[2] = -2;
      vec[3] =  3;
      vec[4] =  0;

      vec = vec * 2;

      checkSize    ( vec, 5UL );
      checkCapacity( vec, 5UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 2 || vec[1] != 0 || vec[2] != -4 || vec[3] != 6 || vec[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 2 0 -4 6 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Self-scaling (v=s*v)
   //=====================================================================================

   {
      test_ = "CustomVector self-scaling (v=s*v)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec( memory.get(), 5UL );
      vec[0] =  1;
      vec[1] =  0;
      vec[2] = -2;
      vec[3] =  3;
      vec[4] =  0;

      vec = 2 * vec;

      checkSize    ( vec, 5UL );
      checkCapacity( vec, 5UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 2 || vec[1] != 0 || vec[2] != -4 || vec[3] != 6 || vec[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 2 0 -4 6 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Self-scaling (v/=s)
   //=====================================================================================

   {
      test_ = "CustomVector self-scaling (v/=s)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec( memory.get(), 5UL );
      vec[0] =  2;
      vec[1] =  0;
      vec[2] = -4;
      vec[3] =  6;
      vec[4] =  0;

      vec /= 2;

      checkSize    ( vec, 5UL );
      checkCapacity( vec, 5UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 0 || vec[2] != -2 || vec[3] != 3 || vec[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 0 -2 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // Self-scaling (v=v/s)
   //=====================================================================================

   {
      test_ = "CustomVector self-scaling (v=v/s)";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 5UL ) );
      VT vec( memory.get(), 5UL );
      vec[0] =  2;
      vec[1] =  0;
      vec[2] = -4;
      vec[3] =  6;
      vec[4] =  0;

      vec = vec / 2;

      checkSize    ( vec, 5UL );
      checkCapacity( vec, 5UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 0 || vec[2] != -2 || vec[3] != 3 || vec[4] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed self-scaling operation\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 0 -2 3 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // CustomVector::scale()
   //=====================================================================================

   {
      test_ = "CustomVector::scale() (int)";

      // Initialization check
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec[0] = 1;
      vec[1] = 2;
      vec[2] = 3;
      vec[3] = 4;

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Integral scaling of the vector
      vec.scale( 2 );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 2 || vec[1] != 4 || vec[2] != 6 || vec[3] != 8 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 2 4 6 8 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Floating point scaling of the vector
      vec.scale( 0.5 );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      test_ = "CustomVector::scale() (complex)";

      using blaze::complex;
      using blaze::aligned;
      using blaze::unpadded;
      using blaze::rowVector;

      using cplx = complex<float>;
      using AlignedUnpadded = blaze::CustomVector<cplx,aligned,unpadded,rowVector>;
      std::unique_ptr<cplx[],blaze::Deallocate> memory( blaze::allocate<cplx>( 2UL ) );
      AlignedUnpadded vec( memory.get(), 2UL );
      vec[0] = cplx( 1.0F, 0.0F );
      vec[1] = cplx( 2.0F, 0.0F );
      vec.scale( cplx( 3.0F, 0.0F ) );

      checkSize    ( vec, 2UL );
      checkCapacity( vec, 2UL );
      checkNonZeros( vec, 2UL );

      if( vec[0] != cplx( 3.0F, 0.0F ) || vec[1] != cplx( 6.0F, 0.0F ) ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scale operation failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( (3,0) (6,0) )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector subscript operator.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the subscript operator
// of the CustomVector class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void AlignedUnpaddedTest::testSubscript()
{
   test_ = "CustomVector::operator[]";

   // Assignment to the element at index 2
   std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 7UL ) );
   VT vec( memory.get(), 7UL );
   reset( vec );
   vec[2] = 1;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 1UL );

   if( vec[2] != 1 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 0 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 5
   vec[5] = 2;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 2UL );

   if( vec[2] != 1 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 0 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 3
   vec[3] = 3;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 3UL );

   if( vec[2] != 1 || vec[3] != 3 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 0
   vec[0] = 4;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 4UL );

   if( vec[0] != 4 || vec[2] != 1 || vec[3] != 3 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 0 1 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Addition assignment to the element at index 2
   vec[2] += vec[3];

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 4UL );

   if( vec[0] != 4 || vec[2] != 4 || vec[3] != 3 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 0 4 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Subtraction assignment to the element at index 1
   vec[1] -= vec[5];

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec[0] != 4 || vec[1] != -2 || vec[2] != 4 || vec[3] != 3 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 4 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Multiplication assignment to the element at index 3
   vec[3] *= -3;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec[0] != 4 || vec[1] != -2 || vec[2] != 4 || vec[3] != -9 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 4 -9 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Division assignment to the element at index 2
   vec[2] /= 2;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec[0] != 4 || vec[1] != -2 || vec[2] != 2 || vec[3] != -9 || vec[5] != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Subscript operator failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 2 -9 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c at() member function of the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of adding and accessing elements via the \c at() member function
// of the CustomVector class template. In case an error is detected, a \a std::runtime_error
// exception is thrown.
*/
void AlignedUnpaddedTest::testAt()
{
   test_ = "CustomVector::at()";

   // Assignment to the element at index 2
   std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 7UL ) );
   VT vec( memory.get(), 7UL );
   reset( vec );
   vec.at(2) = 1;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 1UL );

   if( vec.at(2) != 1 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 0 0 0 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 5
   vec.at(5) = 2;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 2UL );

   if( vec.at(2) != 1 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 0 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 3
   vec.at(3) = 3;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 3UL );

   if( vec.at(2) != 1 || vec.at(3) != 3 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 0 0 1 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Assignment to the element at index 0
   vec.at(0) = 4;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 4UL );

   if( vec.at(0) != 4 || vec.at(2) != 1 || vec.at(3) != 3 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 0 1 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Addition assignment to the element at index 2
   vec.at(2) += vec.at(3);

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 4UL );

   if( vec.at(0) != 4 || vec.at(2) != 4 || vec.at(3) != 3 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 0 4 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Subtraction assignment to the element at index 1
   vec.at(1) -= vec.at(5);

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec.at(0) != 4 || vec.at(1) != -2 || vec.at(2) != 4 || vec.at(3) != 3 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 4 3 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Multiplication assignment to the element at index 3
   vec.at(3) *= -3;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec.at(0) != 4 || vec.at(1) != -2 || vec.at(2) != 4 || vec.at(3) != -9 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 4 -9 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Division assignment to the element at index 2
   vec.at(2) /= 2;

   checkSize    ( vec, 7UL );
   checkCapacity( vec, 7UL );
   checkNonZeros( vec, 5UL );

   if( vec.at(0) != 4 || vec.at(1) != -2 || vec.at(2) != 2 || vec.at(3) != -9 || vec.at(5) != 2 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Access via at() function failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 2 -9 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Attempt to assign to the element at index 7
   try {
      vec.at(7) = 2;

      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Out-of-bound access succeeded\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 4 -2 2 -9 0 2 0 )\n";
      throw std::runtime_error( oss.str() );
   }
   catch( std::out_of_range& ) {}
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the CustomVector iterator implementation.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the iterator implementation of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testIterator()
{
   using Iterator      = VT::Iterator;
   using ConstIterator = VT::ConstIterator;

   std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
   VT vec( memory.get(), 4UL );
   vec[0] =  1;
   vec[1] =  0;
   vec[2] = -2;
   vec[3] = -3;

   // Testing the Iterator default constructor
   {
      test_ = "Iterator default constructor";

      Iterator it{};

      if( it != Iterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing the ConstIterator default constructor
   {
      test_ = "ConstIterator default constructor";

      ConstIterator it{};

      if( it != ConstIterator() ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator default constructor\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing conversion from Iterator to ConstIterator
   {
      test_ = "Iterator/ConstIterator conversion";

      ConstIterator it( begin( vec ) );

      if( it == end( vec ) || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Failed iterator conversion detected\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via Iterator (end-begin)
   {
      test_ = "Iterator subtraction (end-begin)";

      const ptrdiff_t number( end( vec ) - begin( vec ) );

      if( number != 4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via Iterator (begin-end)
   {
      test_ = "Iterator subtraction (begin-end)";

      const ptrdiff_t number( begin( vec ) - end( vec ) );

      if( number != -4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via ConstIterator (end-begin)
   {
      test_ = "ConstIterator subtraction (end-begin)";

      const ptrdiff_t number( cend( vec ) - cbegin( vec ) );

      if( number != 4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: 4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Counting the number of elements via ConstIterator (begin-end)
   {
      test_ = "ConstIterator subtraction (begin-end)";

      const ptrdiff_t number( cbegin( vec ) - cend( vec ) );

      if( number != -4L ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid number of elements detected\n"
             << " Details:\n"
             << "   Number of elements         : " << number << "\n"
             << "   Expected number of elements: -4\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing read-only access via ConstIterator
   {
      test_ = "Read-only access via ConstIterator";

      ConstIterator it ( cbegin( vec ) );
      ConstIterator end( cend( vec ) );

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid initial iterator detected\n";
         throw std::runtime_error( oss.str() );
      }

      ++it;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      --it;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator pre-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it++;

      if( it == end || *it != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-increment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it--;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator post-decrement failed\n";
         throw std::runtime_error( oss.str() );
      }

      it += 2UL;

      if( it == end || *it != -2 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator addition assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it -= 2UL;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator subtraction assignment failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it + 3UL;

      if( it == end || *it != -3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar addition failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = it - 3UL;

      if( it == end || *it != 1 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Iterator/scalar subtraction failed\n";
         throw std::runtime_error( oss.str() );
      }

      it = 4UL + it;

      if( it != end ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Scalar/iterator addition failed\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing assignment via Iterator
   {
      test_ = "Assignment via Iterator";

      int value = 6;

      for( Iterator it=begin( vec ); it!=end( vec ); ++it ) {
         *it = value++;
      }

      if( vec[0] != 6 || vec[1] != 7 || vec[2] != 8 || vec[3] != 9 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Assignment via iterator failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 6 7 8 9 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing addition assignment via Iterator
   {
      test_ = "Addition assignment via Iterator";

      int value = 2;

      for( Iterator it=begin( vec ); it!=end( vec ); ++it ) {
         *it += value++;
      }

      if( vec[0] != 8 || vec[1] != 10 || vec[2] != 12 || vec[3] != 14 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Addition assignment via iterator failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 8 10 12 14 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing subtraction assignment via Iterator
   {
      test_ = "Subtraction assignment via Iterator";

      int value = 2;

      for( Iterator it=begin( vec ); it!=end( vec ); ++it ) {
         *it -= value++;
      }

      if( vec[0] != 6 || vec[1] != 7 || vec[2] != 8 || vec[3] != 9 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Subtraction assignment via iterator failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 6 7 8 9 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing multiplication assignment via Iterator
   {
      test_ = "Multiplication assignment via Iterator";

      int value = 1;

      for( Iterator it=begin( vec ); it!=end( vec ); ++it ) {
         *it *= value++;
      }

      if( vec[0] != 6 || vec[1] != 14 || vec[2] != 24 || vec[3] != 36 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Multiplication assignment via iterator failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 6 14 24 36 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // Testing division assignment via Iterator
   {
      test_ = "Division assignment via Iterator";

      for( Iterator it=begin( vec ); it!=end( vec ); ++it ) {
         *it /= 2;
      }

      if( vec[0] != 3 || vec[1] != 7 || vec[2] != 12 || vec[3] != 18 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Division assignment via iterator failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 3 7 12 18 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c nonZeros() member function of the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c nonZeros() member function of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testNonZeros()
{
   test_ = "CustomVector::nonZeros()";

   {
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      reset( vec );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 0UL );

      if( vec[0] != 0 || vec[1] != 0 || vec[2] != 0 || vec[3] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }

   {
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec[0] = 1;
      vec[1] = 2;
      vec[2] = 0;
      vec[3] = 3;

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 3 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 0 3 )\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c reset() member function of the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c reset() member function of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testReset()
{
   using blaze::reset;


   //=====================================================================================
   // CustomVector::reset()
   //=====================================================================================

   {
      test_ = "CustomVector::reset()";

      // Initialization check
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec[0] = 1;
      vec[1] = 2;
      vec[2] = 3;
      vec[3] = 4;

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 4UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Initialization failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 3 4 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting a single element
      reset( vec[2] );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 3UL );

      if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 4 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 1 2 0 4 )\n";
         throw std::runtime_error( oss.str() );
      }

      // Resetting the vector
      reset( vec );

      checkSize    ( vec, 4UL );
      checkCapacity( vec, 4UL );
      checkNonZeros( vec, 0UL );

      if( vec[0] != 0 || vec[1] != 0 || vec[2] != 0 || vec[3] != 0 ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Reset operation failed\n"
             << " Details:\n"
             << "   Result:\n" << vec << "\n"
             << "   Expected result:\n( 0 0 0 0 )\n";
         throw std::runtime_error( oss.str() );
      }
   }


   //=====================================================================================
   // CustomVector::reset( Type*, size_t, size_t )
   //=====================================================================================

   {
      test_ = "CustomVector::reset( Type*, size_t, size_t )";

      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
      VT vec( memory.get(), 4UL );
      vec[0] = 1;
      vec[1] = 2;
      vec[2] = 3;
      vec[3] = 4;

      std::unique_ptr<int[],blaze::Deallocate> array( blaze::allocate<int>( 27UL ) );
      vec.reset( array.get(), 27UL );

      checkSize    ( vec, 27UL );
      checkCapacity( vec, 27UL );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c clear() member function of the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c clear() member function of the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testClear()
{
   using blaze::clear;

   test_ = "CustomVector::clear()";

   // Initialization check
   std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 4UL ) );
   VT vec( memory.get(), 4UL );
   vec[0] = 1;
   vec[1] = 2;
   vec[2] = 3;
   vec[3] = 4;

   checkSize    ( vec, 4UL );
   checkCapacity( vec, 4UL );
   checkNonZeros( vec, 4UL );

   if( vec[0] != 1 || vec[1] != 2 || vec[2] != 3 || vec[3] != 4 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Initialization failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 2 3 4 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Clearing a single element
   clear( vec[2] );

   checkSize    ( vec, 4UL );
   checkCapacity( vec, 4UL );
   checkNonZeros( vec, 3UL );

   if( vec[0] != 1 || vec[1] != 2 || vec[2] != 0 || vec[3] != 4 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Clear operation failed\n"
          << " Details:\n"
          << "   Result:\n" << vec << "\n"
          << "   Expected result:\n( 1 2 0 4 )\n";
      throw std::runtime_error( oss.str() );
   }

   // Clearing the vector
   clear( vec );

   checkSize    ( vec, 0UL );
   checkNonZeros( vec, 0UL );
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c swap() functionality of the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c swap() function of the CustomVector class template.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testSwap()
{
   test_ = "CustomVector swap";

   std::unique_ptr<int[],blaze::Deallocate> memory1( blaze::allocate<int>( 3UL ) );
   VT vec1( memory1.get(), 3UL );
   vec1[0] = 1;
   vec1[1] = 2;
   vec1[2] = 3;

   std::unique_ptr<int[],blaze::Deallocate> memory2( blaze::allocate<int>( 4UL ) );
   VT vec2( memory2.get(), 4UL );
   vec2[0] = 4;
   vec2[1] = 3;
   vec2[2] = 2;
   vec2[3] = 1;

   swap( vec1, vec2 );

   checkSize    ( vec1, 4UL );
   checkCapacity( vec1, 4UL );
   checkNonZeros( vec1, 4UL );

   if( vec1[0] != 4 || vec1[1] != 3 || vec1[2] != 2 || vec1[3] != 1 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the first vector failed\n"
          << " Details:\n"
          << "   Result:\n" << vec1 << "\n"
          << "   Expected result:\n( 4 3 2 1 )\n";
      throw std::runtime_error( oss.str() );
   }

   checkSize    ( vec2, 3UL );
   checkCapacity( vec2, 3UL );
   checkNonZeros( vec2, 3UL );

   if( vec2[0] != 1 || vec2[1] != 2 || vec2[2] != 3 ) {
      std::ostringstream oss;
      oss << " Test: " << test_ << "\n"
          << " Error: Swapping the second vector failed\n"
          << " Details:\n"
          << "   Result:\n" << vec1 << "\n"
          << "   Expected result:\n( 1 2 3 )\n";
      throw std::runtime_error( oss.str() );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the \c isDefault() function with the CustomVector class template.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs a test of the \c isDefault() function with the CustomVector class
// template. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AlignedUnpaddedTest::testIsDefault()
{
   using blaze::isDefault;

   test_ = "isDefault() function";

   // isDefault with vector of size 0
   {
      VT vec;

      if( isDefault( vec ) != true ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid isDefault evaluation\n"
             << " Details:\n"
             << "   Vector:\n" << vec << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // isDefault with default vector
   {
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 3UL ) );
      VT vec( memory.get(), 3UL );
      reset( vec );

      if( isDefault( vec[1] ) != true ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid isDefault evaluation\n"
             << " Details:\n"
             << "   Vector element: " << vec[1] << "\n";
         throw std::runtime_error( oss.str() );
      }

      if( isDefault( vec ) != false ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid isDefault evaluation\n"
             << " Details:\n"
             << "   Vector:\n" << vec << "\n";
         throw std::runtime_error( oss.str() );
      }
   }

   // isDefault with non-default vector
   {
      std::unique_ptr<int[],blaze::Deallocate> memory( blaze::allocate<int>( 3UL ) );
      VT vec( memory.get(), 3UL );
      reset( vec );
      vec[1] = 1;

      if( isDefault( vec[1] ) != false ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid isDefault evaluation\n"
             << " Details:\n"
             << "   Vector element: " << vec[1] << "\n";
         throw std::runtime_error( oss.str() );
      }

      if( isDefault( vec ) != false ) {
         std::ostringstream oss;
         oss << " Test: " << test_ << "\n"
             << " Error: Invalid isDefault evaluation\n"
             << " Details:\n"
             << "   Vector:\n" << vec << "\n";
         throw std::runtime_error( oss.str() );
      }
   }
}
//*************************************************************************************************

} // namespace customvector

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aligned/unpadded CustomVector class test..." << std::endl;

   try
   {
      RUN_CUSTOMVECTOR_ALIGNED_UNPADDED_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aligned/unpadded CustomVector class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
