//=================================================================================================
/*!
//  \file src/mathtest/tdvecdmatmult/AliasingTest.cpp
//  \brief Source file for the dense vector/dense matrix multiplication aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/tdvecdmatmult/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace tdvecdmatmult {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : dA4x3_ ( 4UL, 3UL )
   , dB3x3_ ( 3UL, 3UL )
   , tdA4x3_( 4UL, 3UL )
   , tdB3x3_( 3UL, 3UL )
   , tda4_  ( 4UL )
   , tdb4_  ( 4UL )
   , tdc3_  ( 3UL )
   , tdd3_  ( 3UL )
   , tde3_  ( 3UL )
   , tsa4_  ( 4UL )
   , tsb3_  ( 3UL )
   , result_()
   , test_  ()
{
   testTDVecDMatMult ();
   testTDVecTDMatMult();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the dense vector/dense matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense vector/dense matrix multiplication.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTDVecDMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TDVecDMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tda4_ * dA4x3_;
      tda4_   = tda4_ * dA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = tdb4_ * ( trans( tda4_ ) * tdc3_ );
      tda4_   = tdb4_ * ( trans( tda4_ ) * tdc3_ );

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Assignment to second operand of left-hand side compound";

      initialize();

      result_ = tdb4_ * ( trans( tda4_ ) * tdc3_ );
      tdc3_   = tdb4_ * ( trans( tda4_ ) * tdc3_ );

      checkResult( tdc3_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * dA4x3_;
      tda4_   = ( tda4_ + tsa4_ ) * dA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * dA4x3_;
      tsa4_   = ( tda4_ + tsa4_ ) * dA4x3_;

      checkResult( tsa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TDVecDMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ += tdc3_ * dB3x3_;
      tdc3_   += tdc3_ * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdc3_   += tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdd3_;
      result_ += tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdd3_   += tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdd3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += ( tdc3_ + tsb3_ ) * dB3x3_;
      tdc3_   += ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ += ( tdc3_ + tsb3_ ) * dB3x3_;
      tsb3_   += ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TDVecDMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ -= tdc3_ * dB3x3_;
      tdc3_   -= tdc3_ * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to first operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdc3_   -= tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdd3_;
      result_ -= tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdd3_   -= tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdd3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= ( tdc3_ + tsb3_ ) * dB3x3_;
      tdc3_   -= ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ -= ( tdc3_ + tsb3_ ) * dB3x3_;
      tsb3_   -= ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TDVecDMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ *= tdc3_ * dB3x3_;
      tdc3_   *= tdc3_ * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdc3_   *= tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "TDVecDMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tdd3_;
      result_ *= tde3_ * ( trans( tdc3_ ) * tdd3_ );
      tdd3_   *= tde3_ * ( trans( tdc3_ ) * tdd3_ );

      checkResult( tdd3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= ( tdc3_ + tsb3_ ) * dB3x3_;
      tdc3_   *= ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TDVecDMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ *= ( tdc3_ + tsb3_ ) * dB3x3_;
      tsb3_   *= ( tdc3_ + tsb3_ ) * dB3x3_;

      checkResult( tsb3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the transpose dense vector/transpose dense matrix multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the transpose dense vector/transpose dense matrix
// multiplication. In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testTDVecTDMatMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TDVecTDMatMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tda4_ * tdA4x3_;
      tda4_   = tda4_ * tdA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * tdA4x3_;
      tda4_   = ( tda4_ + tsa4_ ) * tdA4x3_;

      checkResult( tda4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = ( tda4_ + tsa4_ ) * tdA4x3_;
      tsa4_   = ( tda4_ + tsa4_ ) * tdA4x3_;

      checkResult( tsa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TDVecTDMatMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ += tdc3_ * tdB3x3_;
      tdc3_   += tdc3_ * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ += ( tdc3_ + tsb3_ ) * tdB3x3_;
      tdc3_   += ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ += ( tdc3_ + tsb3_ ) * tdB3x3_;
      tsb3_   += ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TDVecTDMatMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ -= tdc3_ * tdB3x3_;
      tdc3_   -= tdc3_ * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ -= ( tdc3_ + tsb3_ ) * tdB3x3_;
      tdc3_   -= ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ -= ( tdc3_ + tsb3_ ) * tdB3x3_;
      tsb3_   -= ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tsb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TDVecTDMatMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  tdc3_;
      result_ *= tdc3_ * tdB3x3_;
      tdc3_   *= tdc3_ * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  tdc3_;
      result_ *= ( tdc3_ + tsb3_ ) * tdB3x3_;
      tdc3_   *= ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tdc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TDVecTDMatMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  tsb3_;
      result_ *= ( tdc3_ + tsb3_ ) * tdB3x3_;
      tsb3_   *= ( tdc3_ + tsb3_ ) * tdB3x3_;

      checkResult( tsb3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the dense matrices
   //=====================================================================================

   // Initializing the first row-major dense matrix
   dA4x3_(0,0) = -1;
   dA4x3_(0,1) =  0;
   dA4x3_(0,2) = -2;
   dA4x3_(1,0) =  0;
   dA4x3_(1,1) =  2;
   dA4x3_(1,2) = -3;
   dA4x3_(2,0) =  0;
   dA4x3_(2,1) =  1;
   dA4x3_(2,2) =  2;
   dA4x3_(3,0) =  1;
   dA4x3_(3,1) =  0;
   dA4x3_(3,2) = -2;

   // Initializing the second row-major dense matrix
   dB3x3_(0,0) =  0;
   dB3x3_(0,1) = -1;
   dB3x3_(0,2) =  0;
   dB3x3_(1,0) =  1;
   dB3x3_(1,1) = -2;
   dB3x3_(1,2) =  2;
   dB3x3_(2,0) =  0;
   dB3x3_(2,1) =  0;
   dB3x3_(2,2) = -3;

   // Initializing the first column-major dense matrix
   tdA4x3_(0,0) = -1;
   tdA4x3_(0,1) =  0;
   tdA4x3_(0,2) = -2;
   tdA4x3_(1,0) =  0;
   tdA4x3_(1,1) =  2;
   tdA4x3_(1,2) = -3;
   tdA4x3_(2,0) =  0;
   tdA4x3_(2,1) =  1;
   tdA4x3_(2,2) =  2;
   tdA4x3_(3,0) =  1;
   tdA4x3_(3,1) =  0;
   tdA4x3_(3,2) = -2;

   // Initializing the second column-major dense matrix
   tdB3x3_(0,0) =  0;
   tdB3x3_(0,1) = -1;
   tdB3x3_(0,2) =  0;
   tdB3x3_(1,0) =  1;
   tdB3x3_(1,1) = -2;
   tdB3x3_(1,2) =  2;
   tdB3x3_(2,0) =  0;
   tdB3x3_(2,1) =  0;
   tdB3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense row vector
   tda4_.resize( 4UL, false );
   tda4_[0] = -1;
   tda4_[1] =  0;
   tda4_[2] = -3;
   tda4_[3] =  2;

   // Initializing the second dense row vector
   tdb4_.resize( 4UL, false );
   tdb4_[0] =  0;
   tdb4_[1] =  1;
   tdb4_[2] =  2;
   tdb4_[3] = -1;

   // Initializing the third dense row vector
   tdc3_.resize( 3UL, false );
   tdc3_[0] = 1;
   tdc3_[1] = 2;
   tdc3_[2] = 3;

   // Initializing the fourth dense row vector
   tdd3_.resize( 3UL, false );
   tdd3_[0] = 0;
   tdd3_[1] = 2;
   tdd3_[2] = 1;

   // Initializing the fifth dense row vector
   tde3_.resize( 3UL, false );
   tde3_[0] = 0;
   tde3_[1] = 1;
   tde3_[2] = 3;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse row vector
   tsa4_.resize( 4UL, false );
   tsa4_.reset();
   tsa4_[0] = -1;
   tsa4_[2] = -3;
   tsa4_[3] =  2;

   // Initializing the second sparse row vector
   tsb3_.resize( 3UL, false );
   tsb3_.reset();
   tsb3_[1] = 2;
   tsb3_[2] = 1;
}
//*************************************************************************************************

} // namespace tdvecdmatmult

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_TDVECDMATMULT_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
