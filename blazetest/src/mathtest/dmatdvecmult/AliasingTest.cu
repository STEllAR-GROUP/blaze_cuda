//=================================================================================================
/*!
//  \file src/mathtest/dmatdvecmult/AliasingTest.cpp
//  \brief Source file for the dense matrix/dense vector multiplication aliasing test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/mathtest/dmatdvecmult/AliasingTest.h>

#ifdef BLAZE_USE_HPX_THREADS
#  include <hpx/hpx_main.hpp>
#endif


namespace blazetest {

namespace mathtest {

namespace dmatdvecmult {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the aliasing test class.
//
// \exception std::runtime_error Operation error detected.
*/
AliasingTest::AliasingTest()
   : dA3x4_ ( 3UL, 4UL )
   , dB3x3_ ( 3UL, 3UL )
   , tdA3x4_( 3UL, 4UL )
   , tdB3x3_( 3UL, 3UL )
   , da4_   ( 4UL )
   , db4_   ( 4UL )
   , dc3_   ( 3UL )
   , dd3_   ( 3UL )
   , de3_   ( 3UL )
   , sa4_   ( 4UL )
   , sb3_   ( 3UL )
   , result_()
   , test_  ()
{
   testDMatDVecMult ();
   testTDMatDVecMult();
}
//*************************************************************************************************




//=================================================================================================
//
//  TEST FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Test of the dense matrix/dense vector multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the dense matrix/dense vector multiplication.
// In case an error is detected, a \a std::runtime_error exception is thrown.
*/
void AliasingTest::testDMatDVecMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "DMatDVecMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = dA3x4_ * da4_;
      da4_    = dA3x4_ * da4_;

      checkResult( da4_, result_ );
   }

   // Assignment to first operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Assignment to first operand of left-hand side compound";

      initialize();

      result_ = ( dc3_ * trans( da4_ ) ) * db4_;
      dc3_    = ( dc3_ * trans( da4_ ) ) * db4_;

      checkResult( dc3_, result_ );
   }

   // Assignment to second operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Assignment to second operand of left-hand side compound";

      initialize();

      result_ = ( dc3_ * trans( da4_ ) ) * db4_;
      da4_    = ( dc3_ * trans( da4_ ) ) * db4_;

      checkResult( da4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = dA3x4_ * ( da4_ + sa4_ );
      da4_    = dA3x4_ * ( da4_ + sa4_ );

      checkResult( da4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = dA3x4_ * ( da4_ + sa4_ );
      sa4_    = dA3x4_ * ( da4_ + sa4_ );

      checkResult( sa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "DMatDVecMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ += dB3x3_ * dc3_;
      dc3_    += dB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Addition assignment to first operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ += ( dc3_ * trans( dd3_ ) ) * de3_;
      dc3_    += ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dc3_, result_ );
   }

   // Addition assignment to second operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dd3_;
      result_ += ( dc3_ * trans( dd3_ ) ) * de3_;
      dd3_    += ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dd3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ += dB3x3_ * ( dc3_ + sb3_ );
      dc3_    += dB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ += dB3x3_ * ( dc3_ + sb3_ );
      sb3_    += dB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "DMatDVecMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ -= dB3x3_ * dc3_;
      dc3_    -= dB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to first operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ -= ( dc3_ * trans( dd3_ ) ) * de3_;
      dc3_    -= ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to second operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dd3_;
      result_ -= ( dc3_ * trans( dd3_ ) ) * de3_;
      dd3_    -= ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dd3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ -= dB3x3_ * ( dc3_ + sb3_ );
      dc3_    -= dB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ -= dB3x3_ * ( dc3_ + sb3_ );
      sb3_    -= dB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "DMatDVecMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ *= dB3x3_ * dc3_;
      dc3_    *= dB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to first operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ *= ( dc3_ * trans( dd3_ ) ) * de3_;
      dc3_    *= ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to second operand of left-hand side compound
   {
      test_ = "DMatDVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  dd3_;
      result_ *= ( dc3_ * trans( dd3_ ) ) * de3_;
      dd3_    *= ( dc3_ * trans( dd3_ ) ) * de3_;

      checkResult( dd3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ *= dB3x3_ * ( dc3_ + sb3_ );
      dc3_    *= dB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "DMatDVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ *= dB3x3_ * ( dc3_ + sb3_ );
      sb3_    *= dB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }
}
//*************************************************************************************************


//*************************************************************************************************
/*!\brief Test of the transpose dense matrix/dense vector multiplication.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function performs aliasing tests for the transpose dense matrix/dense vector
// multiplication. In case an error is detected, a \a std::runtime_error exception is
// thrown.
*/
void AliasingTest::testTDMatDVecMult()
{
   //=====================================================================================
   // Multiplication
   //=====================================================================================

   // Assignment to left-hand side operand
   {
      test_ = "TDMatDVecMult - Assignment to right-hand side vector operand";

      initialize();

      result_ = tdA3x4_ * da4_;
      da4_    = tdA3x4_ * da4_;

      checkResult( da4_, result_ );
   }

   // Assignment to first operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Assignment to first operand of right-hand side compound";

      initialize();

      result_ = tdA3x4_ * ( da4_ + sa4_ );
      da4_    = tdA3x4_ * ( da4_ + sa4_ );

      checkResult( da4_, result_ );
   }

   // Assignment to second operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Assignment to second operand of right-hand side compound";

      initialize();

      result_ = tdA3x4_ * ( da4_ + sa4_ );
      sa4_    = tdA3x4_ * ( da4_ + sa4_ );

      checkResult( sa4_, result_ );
   }


   //=====================================================================================
   // Multiplication with addition assignment
   //=====================================================================================

   // Addition assignment to left-hand side operand
   {
      test_ = "TDMatDVecMult - Addition assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ += tdB3x3_ * dc3_;
      dc3_    += tdB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Addition assignment to first operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Addition assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ += tdB3x3_ * ( dc3_ + sb3_ );
      dc3_    += tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Addition assignment to second operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Addition assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ += tdB3x3_ * ( dc3_ + sb3_ );
      sb3_    += tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with subtraction assignment
   //=====================================================================================

   // Subtraction assignment to left-hand side operand
   {
      test_ = "TDMatDVecMult - Subtraction assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ -= tdB3x3_ * dc3_;
      dc3_    -= tdB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to first operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Subtraction assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ -= tdB3x3_ * ( dc3_ + sb3_ );
      dc3_    -= tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Subtraction assignment to second operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Subtraction assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ -= tdB3x3_ * ( dc3_ + sb3_ );
      sb3_    -= tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }


   //=====================================================================================
   // Multiplication with multiplication assignment
   //=====================================================================================

   // Multiplication assignment to left-hand side operand
   {
      test_ = "TDMatDVecMult - Multiplication assignment to right-hand side vector operand";

      initialize();

      result_ =  dc3_;
      result_ *= tdB3x3_ * dc3_;
      dc3_    *= tdB3x3_ * dc3_;

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to first operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Multiplication assignment to first operand of left-hand side compound";

      initialize();

      result_ =  dc3_;
      result_ *= tdB3x3_ * ( dc3_ + sb3_ );
      dc3_    *= tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( dc3_, result_ );
   }

   // Multiplication assignment to second operand of right-hand side compound
   {
      test_ = "TDMatDVecMult - Multiplication assignment to second operand of left-hand side compound";

      initialize();

      result_ =  sb3_;
      result_ *= tdB3x3_ * ( dc3_ + sb3_ );
      sb3_    *= tdB3x3_ * ( dc3_ + sb3_ );

      checkResult( sb3_, result_ );
   }
}
//*************************************************************************************************




//=================================================================================================
//
//  UTILITY FUNCTIONS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Initialization of all member vectors and matrices.
//
// \return void
// \exception std::runtime_error Error detected.
//
// This function initializes all member vectors and matrices to specific predetermined values.
*/
void AliasingTest::initialize()
{
   //=====================================================================================
   // Initialization of the dense matrices
   //=====================================================================================

   // Initializing the first row-major dense matrix
   dA3x4_(0,0) = -1;
   dA3x4_(0,1) =  0;
   dA3x4_(0,2) = -2;
   dA3x4_(0,3) =  0;
   dA3x4_(1,0) =  0;
   dA3x4_(1,1) =  2;
   dA3x4_(1,2) = -3;
   dA3x4_(1,3) =  1;
   dA3x4_(2,0) =  0;
   dA3x4_(2,1) =  1;
   dA3x4_(2,2) =  2;
   dA3x4_(2,3) =  2;

   // Initializing the second row-major dense matrix
   dB3x3_(0,0) =  0;
   dB3x3_(0,1) = -1;
   dB3x3_(0,2) =  0;
   dB3x3_(1,0) =  1;
   dB3x3_(1,1) = -2;
   dB3x3_(1,2) =  2;
   dB3x3_(2,0) =  0;
   dB3x3_(2,1) =  0;
   dB3x3_(2,2) = -3;

   // Initializing the first column-major dense matrix
   tdA3x4_(0,0) = -1;
   tdA3x4_(0,1) =  0;
   tdA3x4_(0,2) = -2;
   tdA3x4_(0,3) =  0;
   tdA3x4_(1,0) =  0;
   tdA3x4_(1,1) =  2;
   tdA3x4_(1,2) = -3;
   tdA3x4_(1,3) =  1;
   tdA3x4_(2,0) =  0;
   tdA3x4_(2,1) =  1;
   tdA3x4_(2,2) =  2;
   tdA3x4_(2,3) =  2;

   // Initializing the second column-major dense matrix
   tdB3x3_(0,0) =  0;
   tdB3x3_(0,1) = -1;
   tdB3x3_(0,2) =  0;
   tdB3x3_(1,0) =  1;
   tdB3x3_(1,1) = -2;
   tdB3x3_(1,2) =  2;
   tdB3x3_(2,0) =  0;
   tdB3x3_(2,1) =  0;
   tdB3x3_(2,2) = -3;


   //=====================================================================================
   // Initialization of the dense vectors
   //=====================================================================================

   // Initializing the first dense column vector
   da4_.resize( 4UL, false );
   da4_[0] = -1;
   da4_[1] =  0;
   da4_[2] = -3;
   da4_[3] =  2;

   // Initializing the second dense column vector
   db4_.resize( 4UL, false );
   db4_[0] =  0;
   db4_[1] =  1;
   db4_[2] =  2;
   db4_[3] = -1;

   // Initializing the third dense column vector
   dc3_.resize( 3UL, false );
   dc3_[0] = 1;
   dc3_[1] = 2;
   dc3_[2] = 3;

   // Initializing the fourth dense column vector
   dd3_.resize( 3UL, false );
   dd3_[0] = 0;
   dd3_[1] = 2;
   dd3_[2] = 1;

   // Initializing the fifth dense column vector
   de3_.resize( 3UL, false );
   de3_[0] = 0;
   de3_[1] = 1;
   de3_[2] = 3;


   //=====================================================================================
   // Initialization of the sparse vectors
   //=====================================================================================

   // Initializing the first sparse column vector
   sa4_.resize( 4UL, false );
   sa4_.reset();
   sa4_[0] = -1;
   sa4_[2] = -3;
   sa4_[3] =  2;

   // Initializing the second sparse column vector
   sb3_.resize( 3UL, false );
   sb3_.reset();
   sb3_[1] = 2;
   sb3_[2] = 1;
}
//*************************************************************************************************

} // namespace dmatdvecmult

} // namespace mathtest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running aliasing test..." << std::endl;

   try
   {
      RUN_DMATDVECMULT_ALIASING_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during aliasing test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
