//=================================================================================================
/*!
//  \file src/utiltest/alignedallocator/ClassTest.cpp
//  \brief Source file for the AlignedAllocator class test
//
//  Copyright (C) 2012-2019 Klaus Iglberger - All Rights Reserved
//
//  This file is part of the Blaze library. You can redistribute it and/or modify it under
//  the terms of the New (Revised) BSD License. Redistribution and use in source and binary
//  forms, with or without modification, are permitted provided that the following conditions
//  are met:
//
//  1. Redistributions of source code must retain the above copyright notice, this list of
//     conditions and the following disclaimer.
//  2. Redistributions in binary form must reproduce the above copyright notice, this list
//     of conditions and the following disclaimer in the documentation and/or other materials
//     provided with the distribution.
//  3. Neither the names of the Blaze development group nor the names of its contributors
//     may be used to endorse or promote products derived from this software without specific
//     prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
//  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
//  OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
//  SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
//  TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
//  DAMAGE.
*/
//=================================================================================================


//*************************************************************************************************
// Includes
//*************************************************************************************************

#include <cstdlib>
#include <iostream>
#include <blazetest/utiltest/alignedallocator/ClassTest.h>


namespace blazetest {

namespace utiltest {

namespace alignedallocator {

//=================================================================================================
//
//  CONSTRUCTORS
//
//=================================================================================================

//*************************************************************************************************
/*!\brief Constructor for the AlignedAllocator class test.
//
// \exception std::runtime_error Operation error detected.
*/
ClassTest::ClassTest()
{
   testAllocation<char          >();
   testAllocation<signed char   >();
   testAllocation<unsigned char >();
   testAllocation<wchar_t       >();
   testAllocation<short         >();
   testAllocation<unsigned short>();
   testAllocation<int           >();
   testAllocation<unsigned int  >();
   testAllocation<long          >();
   testAllocation<unsigned long >();
   testAllocation<float         >();
   testAllocation<double        >();
   testAllocation<long double   >();
   testAllocation<Aligned16     >();
   testAllocation<Aligned32     >();
   testAllocation<Aligned64     >();
   testAllocation<Aligned128    >();
}
//*************************************************************************************************

} // namespace alignedallocator

} // namespace utiltest

} // namespace blazetest




//=================================================================================================
//
//  MAIN FUNCTION
//
//=================================================================================================

//*************************************************************************************************
int main()
{
   std::cout << "   Running AlignedAllocator class test..." << std::endl;

   try
   {
      RUN_ALIGNEDALLOCATOR_CLASS_TEST;
   }
   catch( std::exception& ex ) {
      std::cerr << "\n\n ERROR DETECTED during AlignedAllocator class test:\n"
                << ex.what() << "\n";
      return EXIT_FAILURE;
   }

   return EXIT_SUCCESS;
}
//*************************************************************************************************
